#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU Lesser General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Lesser General Public License for more details.

    You should have received a copy of the GNU Lesser General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "GenerateHologramCUDA.h"

////////////////////////////////////////////////////////////////////////////////
//Calculate Phases and return as float
//
//Possible improvements
//-compute powers of X, Y and phase2pi only once and put in registers
////////////////////////////////////////////////////////////////////////////////

__global__ void getPhases(unsigned char *g_pSLMuc, float *g_pSLM_start, hipfftComplex *g_cSLMcc, float *g_LUT_coeff, int LUT_on, int data_w)
{	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int s_idx;
	int X_shifted, Y_shifted;
	int N_pixels = data_w * data_w;
	if (idx<N_pixels)
	{
		float phase255;
		float N = data_w;
		int half_w = data_w/2;
		int logN = (int)log2(N);

		//int X = idx&(int)(N-1); works only for data_w = power of 2
 		//int Y = (idx-X)>>logN;
		float X = idx%data_w;
 		float Y = floor(idx/N); 		
 		if (X < half_w)
		{	
			X_shifted = X + half_w;
			if (Y < half_w)
			{
				Y_shifted = Y + half_w;
				s_idx = idx + (data_w * half_w) + half_w;
			}
			else
			{
				Y_shifted = Y - half_w;
				s_idx = idx - (data_w * half_w) + half_w;
			}
		}
		else
		{
			X_shifted = X - half_w;
			if (Y < half_w)
			{
				Y_shifted = Y + half_w;			
				s_idx = idx + (data_w * half_w) - half_w;
			}
			else
			{
				Y_shifted = Y - half_w;			
				s_idx = idx - (data_w * half_w) - half_w;
			}
		}
 		float phase = atan2f(g_cSLMcc[idx].y, g_cSLMcc[idx].x);
		float phase2pi = M_PI + phase;	
		
/*		if (LUT_on == 1)
		{
			__shared__ float s_c[N_LUT_coeff];
			if (threadIdx.x <N_LUT_coeff)
				s_c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
			__syncthreads();
			
			phase255 = 255.0 - (s_c[0] + s_c[1]*X_shifted + s_c[2]*Y_shifted + s_c[3]*phase2pi + s_c[4]*powf(X_shifted,2) + s_c[5]*X*Y_shifted + s_c[6]*X_shifted*phase2pi + s_c[7]*powf(Y_shifted,2) + s_c[8]*Y_shifted*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X_shifted,3) + s_c[11]*powf(X_shifted,2)*Y_shifted + s_c[12]*powf(X_shifted,2)*phase2pi + s_c[13]*X_shifted*powf(Y_shifted,2) + s_c[14]*X_shifted*Y_shifted*phase2pi + s_c[15]*X_shifted*powf(phase2pi,2) + s_c[16]*powf(Y_shifted,3) + s_c[17]*powf(Y_shifted,2)*phase2pi + s_c[18]*Y_shifted*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X_shifted,4) + s_c[21]*powf(X_shifted,3)*Y_shifted + s_c[22]*powf(X_shifted,3)*phase2pi + s_c[23]*powf(X_shifted,2)*powf(Y_shifted,2) + s_c[24]*powf(X_shifted,2)*Y_shifted*phase2pi + s_c[25]*powf(X_shifted,2)*powf(phase2pi,2) + s_c[26]*X_shifted*powf(Y_shifted,3) + s_c[27]*X_shifted*powf(Y_shifted,2)*phase2pi + s_c[28]*X_shifted*Y_shifted*powf(phase2pi,2) + s_c[29]*X_shifted*powf(phase2pi,3) + s_c[30]*powf(Y_shifted,4) + s_c[31]*powf(Y_shifted,3)*phase2pi + s_c[32]*powf(Y_shifted,2)*powf(phase2pi,2) + s_c[33]*Y_shifted*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X_shifted,5) + s_c[36]*powf(X_shifted,4)*Y_shifted + s_c[37]*powf(X_shifted,4)*phase2pi + s_c[38]*powf(X_shifted,3)*powf(Y_shifted,2) + s_c[39]*powf(X_shifted,3)*Y_shifted*phase2pi + s_c[40]*powf(X_shifted,3)*powf(phase2pi,2) + s_c[41]*powf(X_shifted,2)*powf(Y_shifted,3) + s_c[42]*powf(X_shifted,2)*powf(Y_shifted,2)*phase2pi + s_c[43]*powf(X_shifted,2)*Y_shifted*powf(phase2pi,2) + s_c[44]*powf(X_shifted,2)*powf(phase2pi,3) + s_c[45]*X_shifted*powf(Y_shifted,4) + s_c[46]*X_shifted*powf(Y_shifted,3)*phase2pi + s_c[47]*X_shifted*powf(Y_shifted,2)*powf(phase2pi,2) + s_c[48]*X_shifted*Y_shifted*powf(phase2pi,3) + s_c[49]*X_shifted*powf(phase2pi,4) + s_c[50]*powf(Y_shifted,5) + s_c[51]*powf(Y_shifted,4)*phase2pi + s_c[52]*powf(Y_shifted,3)*powf(phase2pi,2) + s_c[53]*powf(Y_shifted,2)*powf(phase2pi,3) + s_c[54]*Y_shifted*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5) + s_c[56]*powf(X_shifted,6) + s_c[57]*powf(X_shifted,5)*Y_shifted + s_c[58]*powf(X_shifted,5)*phase2pi + s_c[59]*powf(X_shifted,4)*powf(Y_shifted,2) + s_c[60]*powf(X_shifted,4)*Y_shifted*phase2pi + s_c[61]*powf(X_shifted,4)*powf(phase2pi,2) + s_c[62]*powf(X_shifted,3)*powf(Y_shifted,3) + s_c[63]*powf(X_shifted,3)*powf(Y_shifted,2)*phase2pi + s_c[64]*powf(X_shifted,3)*Y_shifted*powf(phase2pi,2) + s_c[65]*powf(X_shifted,3)*powf(phase2pi,3) + s_c[66]*powf(X_shifted,2)*powf(Y_shifted,4) + s_c[67]*powf(X_shifted,2)*powf(Y_shifted,3)*phase2pi + s_c[68]*powf(X_shifted,2)*powf(Y_shifted,2)*powf(phase2pi,2) + s_c[69]*powf(X_shifted,2)*Y_shifted*powf(phase2pi,3) + s_c[70]*powf(X_shifted,2)*powf(phase2pi,4) + s_c[71]*X_shifted*powf(Y_shifted,5) + s_c[72]*X_shifted*powf(Y_shifted,4)*phase2pi + s_c[73]*X_shifted*powf(Y_shifted,3)*powf(phase2pi,2) + s_c[74]*X_shifted*powf(Y_shifted,2)*powf(phase2pi,3) + s_c[75]*X_shifted*Y_shifted*powf(phase2pi,4) + s_c[76]*X_shifted*powf(phase2pi,5) + s_c[77]*powf(Y_shifted,6) + s_c[78]*powf(Y_shifted,5)*phase2pi + s_c[79]*powf(Y_shifted,4)*powf(phase2pi,2) + s_c[80]*powf(Y_shifted,3)*powf(phase2pi,3) + s_c[81]*powf(Y_shifted,2)*powf(phase2pi,4) + s_c[82]*Y_shifted*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6) + s_c[84]*powf(X_shifted,7) + s_c[85]*powf(X_shifted,6)*Y_shifted + s_c[86]*powf(X_shifted,6)*phase2pi + s_c[87]*powf(X_shifted,5)*powf(Y_shifted,2) + s_c[88]*powf(X_shifted,5)*Y_shifted*phase2pi + s_c[89]*powf(X_shifted,5)*powf(phase2pi,2) + s_c[90]*powf(X_shifted,4)*powf(Y_shifted,3) + s_c[91]*powf(X_shifted,4)*powf(Y_shifted,2)*phase2pi + s_c[92]*powf(X_shifted,4)*Y_shifted*powf(phase2pi,2) + s_c[93]*powf(X_shifted,4)*powf(phase2pi,3) + s_c[94]*powf(X_shifted,3)*powf(Y_shifted,4) + s_c[95]*powf(X_shifted,3)*powf(Y_shifted,3)*phase2pi + s_c[96]*powf(X_shifted,3)*powf(Y_shifted,2)*powf(phase2pi,2) + s_c[97]*powf(X_shifted,3)*Y_shifted*powf(phase2pi,3) + s_c[98]*powf(X_shifted,3)*powf(phase2pi,4) + s_c[99]*powf(X_shifted,2)*powf(Y_shifted,5) + s_c[100]*powf(X_shifted,2)*powf(Y_shifted,4)*phase2pi + s_c[101]*powf(X_shifted,2)*powf(Y_shifted,3)*powf(phase2pi,2) + s_c[102]*powf(X_shifted,2)*powf(Y_shifted,2)*powf(phase2pi,3) + s_c[103]*powf(X_shifted,2)*Y_shifted*powf(phase2pi,4) + s_c[104]*powf(X_shifted,2)*powf(phase2pi,5) + s_c[105]*X_shifted*powf(Y_shifted,6) + s_c[106]*X_shifted*powf(Y_shifted,5)*phase2pi + s_c[107]*X_shifted*powf(Y_shifted,4)*powf(phase2pi,2) + s_c[108]*X_shifted*powf(Y_shifted,3)*powf(phase2pi,3) + s_c[109]*X_shifted*powf(Y_shifted,2)*powf(phase2pi,4) + s_c[110]*X_shifted*Y_shifted*powf(phase2pi,5) + s_c[111]*X_shifted*powf(phase2pi,6) + s_c[112]*powf(Y_shifted,7) + s_c[113]*powf(Y_shifted,6)*phase2pi + s_c[114]*powf(Y_shifted,5)*powf(phase2pi,2) + s_c[115]*powf(Y_shifted,4)*powf(phase2pi,3) + s_c[116]*powf(Y_shifted,3)*powf(phase2pi,4) + s_c[117]*powf(Y_shifted,2)*powf(phase2pi,5) + s_c[118]*Y_shifted*powf(phase2pi,6) + s_c[119]*powf(phase2pi,7));
			if (phase255 < 0)
				phase255 = 0;	
		}
		else 
		{*/
			phase255 = 255.0 * phase2pi / (2.0 * M_PI);
		//}
		__syncthreads();
		
		g_pSLM_start[idx] = phase;
		g_pSLMuc[s_idx] = (unsigned char)phase255;		
	}
	__syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
//Calculate complex from phases
////////////////////////////////////////////////////////////////////////////////
__global__ void p2c(hipfftComplex *g_c, float *g_p, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		float phase = g_p[idx];
 		g_c[idx].x = cosf(phase);
		g_c[idx].y = sinf(phase);
	}
	__syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
//Calculate amplitudes from complex
////////////////////////////////////////////////////////////////////////////////
__global__ void c_cc2a_f(float *g_a, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_a[idx] = hypotf(g_c[idx].x, g_c[idx].y);
	}
	__syncthreads();
}
////////////////////////////////////////////////////////////////////////////////
//Calculate phases from complex
////////////////////////////////////////////////////////////////////////////////
__global__ void c_cc2p_cc(hipfftComplex *g_p, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_p[idx].x = atan2f(g_c[idx].y, g_c[idx].x);
		g_p[idx].y = 0;
	}
	__syncthreads();
}
////////////////////////////////////////////////////////////////////////////////
//Calculate phases from complex
////////////////////////////////////////////////////////////////////////////////
__global__ void c_cc2p_f(float *g_p, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_p[idx] = atan2f(g_c[idx].y, g_c[idx].x);
	}
	__syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
//Copy real part from complex
////////////////////////////////////////////////////////////////////////////////
__global__ void c_cc2re_f(float *g_p, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_p[idx] = g_c[idx].x;
	}
	__syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
//Copy imaginary part from complex
////////////////////////////////////////////////////////////////////////////////
__global__ void c_cc2im_f(float *g_p, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_p[idx] = g_c[idx].y;
	}
	__syncthreads();
}
////////////////////////////////////////////////////////////////////////////////
//reset amplitudes to ones PCR
////////////////////////////////////////////////////////////////////////////////

__global__ void ReplaceAmpsSLM_FFT(float *g_aLaser, hipfftComplex *g_cAmp, float *g_pSLM_start, int N_pixels, float RPC)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	

	if (idx<N_pixels)
	{
		float aLaser = 1.0f/(float)N_pixels;//g_aLaser[idx];
		float pSLMstart = g_pSLM_start[idx];
		hipfftComplex cAmp = g_cAmp[idx];
 		float phase = atan2f(cAmp.y, cAmp.x);
 		
 		if (RPC < (2.0f*M_PI))
		{	
			if (fabs(phase - pSLMstart) < RPC)
			{
				cAmp.x = aLaser*cosf(phase);
				cAmp.y = aLaser*sinf(phase);
			}
			else
			{
				cAmp.x = aLaser*cosf(pSLMstart);
				cAmp.y = aLaser*sinf(pSLMstart);
			}
		}
		else
		{
			cAmp.x = aLaser*cosf(phase);
			cAmp.y = aLaser*sinf(phase);
		}	
		
		g_cAmp[idx].x = cAmp.x;
		g_cAmp[idx].y = cAmp.y;
	}
	__syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
//Copy phases in desired spots
////////////////////////////////////////////////////////////////////////////////

__global__ void ReplaceAmpsSpots_FFT(hipfftComplex *g_cSpotAmpObtained, hipfftComplex *g_cSpotAmpDesired, int *g_spotIndex, int N_spots, int iteration, float *g_amplitude, float *g_weight, float amp_desired)
{
	int tid = threadIdx.x;
	int spotIndex;
	float phase;
	float amp_new;
	float amp_obtained;
	float weight_next;
	hipfftComplex cSpotAmp;

	__shared__ float weight_sum[256];

	if (tid < N_spots)
	{
		spotIndex = g_spotIndex[tid];
		cSpotAmp = g_cSpotAmpObtained[spotIndex];
		amp_obtained = hypotf(cSpotAmp.x, cSpotAmp.y);
		phase = atan2f(cSpotAmp.y, cSpotAmp.x);
		if (amp_obtained < 0.0000001)
		{
			amp_obtained = 1.0f/(float)N_spots;//powf(256,4);
		}
		weight_next = g_weight[N_spots * (iteration) + tid] * sqrtf(amp_desired / amp_obtained);
		weight_sum[tid] = weight_next;
	}

	__syncthreads();	
				
	if (N_spots >= 256) { if (tid < 128) { weight_sum[tid] += weight_sum[tid + 128]; } __syncthreads(); }
	if (N_spots >= 128) { if (tid < 64) { weight_sum[tid] += weight_sum[tid + 64]; } __syncthreads(); }
	if (tid < 32) 
	{
		if (N_spots >= 64) weight_sum[tid] += weight_sum[tid + 32];
		if (N_spots >= 32) weight_sum[tid] += weight_sum[tid + 16];
		if (N_spots >= 16) weight_sum[tid] += weight_sum[tid + 8];
		if (N_spots >= 8) weight_sum[tid] += weight_sum[tid + 4];
		if (N_spots >= 4) weight_sum[tid] += weight_sum[tid + 2];
		if (N_spots >= 2) weight_sum[tid] += weight_sum[tid + 1];
	}

	__syncthreads();
	
	if (tid<N_spots)												
	{
		weight_next = weight_next / weight_sum[0];
		amp_new = weight_next * amp_desired;    
		cSpotAmp.x = cosf(phase) * amp_new;
		cSpotAmp.y = sinf(phase) * amp_new;
		g_cSpotAmpDesired[spotIndex] = cSpotAmp;
		g_weight[N_spots * (iteration + 1) + tid] = weight_next;
		g_amplitude[N_spots * (iteration) + tid] = amp_obtained;
	}

	__syncthreads();
}
////////////////////////////////////////////////////////////////////////////////
//Convert from coordinates to spot indices
////////////////////////////////////////////////////////////////////////////////

__global__ void XYtoIndex(float *g_x, float *g_y, int *g_spot_index, int N_spots, int data_w)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i < N_spots)
		g_spot_index[i] = ((int)(g_x[i])&(data_w-1))  + ((int)(g_y[i])&(data_w-1))* data_w;
	
	__syncthreads();
}



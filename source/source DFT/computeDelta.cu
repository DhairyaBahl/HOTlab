#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Foobar.  If not, see <http://www.gnu.org/licenses/>.
*/
///////////////////////////////////////////////////////////////////////////////////////
//Calculate one matrix for each spot containing its distance to each pixel on the SLM
///////////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA.h"

__global__ void computeDelta(float *g_x, float *g_y, float *g_z, float *g_delta, float *g_ei_delta_re, float *g_ei_delta_im, int N_spots, int N_pixels, int data_w)
{
	float N = data_w;
	int logN = (int)log2(N);
	float d = 1.0/N;
	
	__shared__ float s_x[block_size];
	__shared__ float s_y[block_size];
	__shared__ float s_z[block_size];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadIdx.x < N_spots)
	{
		s_x[threadIdx.x] = g_x[threadIdx.x];
		s_y[threadIdx.x] = g_y[threadIdx.x];
		s_z[threadIdx.x] = g_z[threadIdx.x];
	}
	__syncthreads();
	int X_int = idx&(int)(N-1);
 	int Y_int = (idx&(int)(N_pixels-1)-X_int)>>logN;		//(idx-X)>>logN gives error for z!=0
 	float X = d * (float)(X_int - 256);
 	float Y = d * (float)(Y_int - 256);
 	
 	for (int m=0; m<N_spots; m++)
	{	
		if (idx<((m+1)*N_pixels) && idx >= m*N_pixels )
		{
			float delta = M_PI * s_z[m] * (X*X + Y*Y) + 2.0 * M_PI * (X * s_x[m] + Y * s_y[m]);
			g_delta[idx] = delta;
			g_ei_delta_re[idx] = cosf(delta);  
			g_ei_delta_im[idx] = sinf(delta); 
		}
	}
	__syncthreads();
}

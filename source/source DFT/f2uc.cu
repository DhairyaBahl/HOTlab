#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//Convert float to unsigned char
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA.h"

__global__ void f2uc(unsigned char *g_uc, float *g_f, int N_pixels, unsigned char *g_LUT, int use_LUTfile, int data_w)
{
	__shared__ unsigned char s_LUT[256];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (use_LUTfile == 1)
	{
		int tid = threadIdx.x;
		if (tid < 256)
		{
			int j = 0;
			while (j < 256)
			{
				s_LUT[tid + j] = g_LUT[tid + j];
				j += block_size;
			}
		}
		__syncthreads();
		if (idx<N_pixels)
		{
			g_uc[idx] = s_LUT[(unsigned char)(((g_f[idx] + M_PI)*255.0/(2.0*M_PI)))];

		}	
	}	
	else
	{
		if (idx<N_pixels)
		{
			g_uc[idx] = (unsigned char)(((g_f[idx] + M_PI)*255.0/(2.0*M_PI)));
		}	
	}
	__syncthreads();
}


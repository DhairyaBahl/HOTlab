#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
////////////////////////////////////////////////////////////////////////////////
//Obtain complex field in image plane using Fresnel propagation
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA.h"

__global__ void computeV(float *g_Vre, float *g_Vim, float *g_pSLM, float *g_delta, int N, int N_spots)
{
	float p;	
	float pSLM, aL;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx<N)
	{
		pSLM = g_pSLM[idx];
		for (int ii=0; ii<N_spots; ++ii)
		{
			int id = idx + ii * N;
			p = pSLM - g_delta[id];
 			g_Vre[id] = cosf(p);
			g_Vim[id] = sinf(p);
		}
	}
}

#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//Obtain complex field in image plane using Fresnel propagation
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA.h"

__global__ void computeV(float *g_Vre, float *g_Vim, float *g_pSLM, float *g_delta, int N, int N_spots)
{
	float p;	
	float pSLM, aL;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx<N)
	{
		pSLM = g_pSLM[idx];
		for (int ii=0; ii<N_spots; ++ii)
		{
			int id = idx + ii * N;
			p = pSLM - g_delta[id];
 			g_Vre[id] = cosf(p);
			g_Vim[id] = sinf(p);
		}
	}
}


/* Old version (50% slower)
__global__ void computeV(float *g_Vre, float *g_Vim, float *g_pSLM, float *g_delta, int N, int N_spots, float *g_aLaser)
{
	
	int N_tot = N*N_spots;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int id = idx&(int)(N-1);
	if (idx<N_tot)
	{
		float p;		
		p = g_pSLM[id] - g_delta[idx];
 		g_Vre[idx] = cosf(p) * g_aLaser[id];
		g_Vim[idx] = sinf(p) * g_aLaser[id];
	}
}
*/
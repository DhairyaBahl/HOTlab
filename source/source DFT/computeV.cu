#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
////////////////////////////////////////////////////////////////////////////////
//Obtain complex field in image plane using Fresnel propagation
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA.h"

__global__ void computeV(float *g_Vre, float *g_Vim, float *g_pSLM, float *g_delta, int N, int N_spots)
{
	float p;	
	float pSLM, aL;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx<N)
	{
		pSLM = g_pSLM[idx];
		for (int ii=0; ii<N_spots; ++ii)
		{
			int id = idx + ii * N;
			p = pSLM - g_delta[id];
 			g_Vre[id] = cosf(p);
			g_Vim[id] = sinf(p);
		}
	}
}


/* Old version (50% slower)
__global__ void computeV(float *g_Vre, float *g_Vim, float *g_pSLM, float *g_delta, int N, int N_spots, float *g_aLaser)
{
	
	int N_tot = N*N_spots;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int id = idx&(int)(N-1);
	if (idx<N_tot)
	{
		float p;		
		p = g_pSLM[id] - g_delta[idx];
 		g_Vre[idx] = cosf(p) * g_aLaser[id];
		g_Vim[idx] = sinf(p) * g_aLaser[id];
	}
}
*/
#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//Obtain phases in SLM plane
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA.h"

__global__ void computeWeights(float *g_Vre, 
							float *g_Vim, 
							int N_spots, 
							float *g_weights, 
							int iteration, 
							float *g_amps_o, 
							float *g_I, 
							int N_pixels)
{

	int tid = threadIdx.x;	
	__shared__ float s_aSpot_o[block_size], s_aSpot_mean;
	float Vre, Vim, wSpot;

	if (tid<N_spots)
	{										//the desited amplitude for each spot
		Vre = g_Vre[tid*N_pixels];
		Vim = g_Vim[tid*N_pixels];
		s_aSpot_o[tid] = hypotf(Vim, Vre)/sqrtf(g_I[tid]);	//the obtained amplitude for each spot divided by the desired amplitude	
		wSpot = g_weights[tid + iteration*N_spots];			//the previous weight for each spot
	}

	__syncthreads();
	if  (tid==0)
	{
		float aSpot_sum = 0;
		for (int kk=0; kk<N_spots; kk++)
		{
			aSpot_sum += s_aSpot_o[kk];
		}
		s_aSpot_mean = aSpot_sum / N_spots; //integer division!!
	}
	__syncthreads();
	
	if (tid<N_spots)
	{
		wSpot = wSpot * s_aSpot_mean / s_aSpot_o[tid];	
		g_weights[tid + N_spots*(iteration+1)] = wSpot;
		g_amps_o[tid + N_spots*iteration] = s_aSpot_o[tid];		
	}
}

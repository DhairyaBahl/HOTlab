#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU Lesser General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Lesser General Public License for more details.

    You should have received a copy of the GNU Lesser General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "GenerateHologramCUDA.h"
__device__ unsigned char applyPolLUT(float phase2pi, int X, int Y, float *s_c, int N_PolCoeff)		
{

	__syncthreads();
	float phase255;
	switch (N_PolCoeff)	{
		case 120:
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5) + s_c[56]*powf(X,6) + s_c[57]*powf(X,5)*Y + s_c[58]*powf(X,5)*phase2pi + s_c[59]*powf(X,4)*powf(Y,2) + s_c[60]*powf(X,4)*Y*phase2pi + s_c[61]*powf(X,4)*powf(phase2pi,2) + s_c[62]*powf(X,3)*powf(Y,3) + s_c[63]*powf(X,3)*powf(Y,2)*phase2pi + s_c[64]*powf(X,3)*Y*powf(phase2pi,2) + s_c[65]*powf(X,3)*powf(phase2pi,3) + s_c[66]*powf(X,2)*powf(Y,4) + s_c[67]*powf(X,2)*powf(Y,3)*phase2pi + s_c[68]*powf(X,2)*powf(Y,2)*powf(phase2pi,2) + s_c[69]*powf(X,2)*Y*powf(phase2pi,3) + s_c[70]*powf(X,2)*powf(phase2pi,4) + s_c[71]*X*powf(Y,5) + s_c[72]*X*powf(Y,4)*phase2pi + s_c[73]*X*powf(Y,3)*powf(phase2pi,2) + s_c[74]*X*powf(Y,2)*powf(phase2pi,3) + s_c[75]*X*Y*powf(phase2pi,4) + s_c[76]*X*powf(phase2pi,5) + s_c[77]*powf(Y,6) + s_c[78]*powf(Y,5)*phase2pi + s_c[79]*powf(Y,4)*powf(phase2pi,2) + s_c[80]*powf(Y,3)*powf(phase2pi,3) + s_c[81]*powf(Y,2)*powf(phase2pi,4) + s_c[82]*Y*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6) + s_c[84]*powf(X,7) + s_c[85]*powf(X,6)*Y + s_c[86]*powf(X,6)*phase2pi + s_c[87]*powf(X,5)*powf(Y,2) + s_c[88]*powf(X,5)*Y*phase2pi + s_c[89]*powf(X,5)*powf(phase2pi,2) + s_c[90]*powf(X,4)*powf(Y,3) + s_c[91]*powf(X,4)*powf(Y,2)*phase2pi + s_c[92]*powf(X,4)*Y*powf(phase2pi,2) + s_c[93]*powf(X,4)*powf(phase2pi,3) + s_c[94]*powf(X,3)*powf(Y,4) + s_c[95]*powf(X,3)*powf(Y,3)*phase2pi + s_c[96]*powf(X,3)*powf(Y,2)*powf(phase2pi,2) + s_c[97]*powf(X,3)*Y*powf(phase2pi,3) + s_c[98]*powf(X,3)*powf(phase2pi,4) + s_c[99]*powf(X,2)*powf(Y,5) + s_c[100]*powf(X,2)*powf(Y,4)*phase2pi + s_c[101]*powf(X,2)*powf(Y,3)*powf(phase2pi,2) + s_c[102]*powf(X,2)*powf(Y,2)*powf(phase2pi,3) + s_c[103]*powf(X,2)*Y*powf(phase2pi,4) + s_c[104]*powf(X,2)*powf(phase2pi,5) + s_c[105]*X*powf(Y,6) + s_c[106]*X*powf(Y,5)*phase2pi + s_c[107]*X*powf(Y,4)*powf(phase2pi,2) + s_c[108]*X*powf(Y,3)*powf(phase2pi,3) + s_c[109]*X*powf(Y,2)*powf(phase2pi,4) + s_c[110]*X*Y*powf(phase2pi,5) + s_c[111]*X*powf(phase2pi,6) + s_c[112]*powf(Y,7) + s_c[113]*powf(Y,6)*phase2pi + s_c[114]*powf(Y,5)*powf(phase2pi,2) + s_c[115]*powf(Y,4)*powf(phase2pi,3) + s_c[116]*powf(Y,3)*powf(phase2pi,4) + s_c[117]*powf(Y,2)*powf(phase2pi,5) + s_c[118]*Y*powf(phase2pi,6) + s_c[119]*powf(phase2pi,7));
			break;
		case 84:
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5) + s_c[56]*powf(X,6) + s_c[57]*powf(X,5)*Y + s_c[58]*powf(X,5)*phase2pi + s_c[59]*powf(X,4)*powf(Y,2) + s_c[60]*powf(X,4)*Y*phase2pi + s_c[61]*powf(X,4)*powf(phase2pi,2) + s_c[62]*powf(X,3)*powf(Y,3) + s_c[63]*powf(X,3)*powf(Y,2)*phase2pi + s_c[64]*powf(X,3)*Y*powf(phase2pi,2) + s_c[65]*powf(X,3)*powf(phase2pi,3) + s_c[66]*powf(X,2)*powf(Y,4) + s_c[67]*powf(X,2)*powf(Y,3)*phase2pi + s_c[68]*powf(X,2)*powf(Y,2)*powf(phase2pi,2) + s_c[69]*powf(X,2)*Y*powf(phase2pi,3) + s_c[70]*powf(X,2)*powf(phase2pi,4) + s_c[71]*X*powf(Y,5) + s_c[72]*X*powf(Y,4)*phase2pi + s_c[73]*X*powf(Y,3)*powf(phase2pi,2) + s_c[74]*X*powf(Y,2)*powf(phase2pi,3) + s_c[75]*X*Y*powf(phase2pi,4) + s_c[76]*X*powf(phase2pi,5) + s_c[77]*powf(Y,6) + s_c[78]*powf(Y,5)*phase2pi + s_c[79]*powf(Y,4)*powf(phase2pi,2) + s_c[80]*powf(Y,3)*powf(phase2pi,3) + s_c[81]*powf(Y,2)*powf(phase2pi,4) + s_c[82]*Y*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6));
			break;
		case 56:
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5));
			break;
		case 36:
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4));
			break;
		case 20:
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3));
			break;
		default:
			phase255 = 0;
			break;
	}
	if (phase255 < 0)
		phase255 = 0;	
	return (unsigned char)phase255;
}
__global__ void LensesAndPrisms(float *g_x, float *g_y, float *g_z, float *g_I, unsigned char *g_SLMuc, int N_spots, unsigned char *g_LUT, bool ApplyLUT_b, int data_w, bool UseAberrationCorr_b, float *d_AberrationCorr_f, bool UseLUTPol_b, float *d_LUTPolCoeff_f, int N_PolCoeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	
	__shared__ float s_x[BLOCK_SIZE];
	__shared__ float s_y[BLOCK_SIZE];
	__shared__ float s_z[BLOCK_SIZE];
	__shared__ float s_a[BLOCK_SIZE];
	__shared__ unsigned char s_LUT[256];
	__shared__ float s_LUTcoeff[120];
	if (tid < N_spots)
	{
		s_x[tid] = g_x[tid];
		s_y[tid] = g_y[tid];
		s_z[tid] = g_z[tid];
		s_a[tid] = sqrtf(g_I[tid]);
	}

	if (ApplyLUT_b)
	{
		if (!UseLUTPol_b)
		{
			if (tid < 256)
				s_LUT[tid] = g_LUT[tid];
		}
		else
		{
			if (tid < N_PolCoeff)
				s_LUTcoeff[tid] = d_LUTPolCoeff_f[tid];
		}

	}		
	__syncthreads();	
	if (idx < data_w*data_w)
	{
		float N = data_w;
		int logN = (int)log2(N);
		int X_int = idx&(int)(N-1);
	 	//int Y_int = (idx&(int)(N*N-1)-X_int)>>logN;
	 	int Y_int = (idx-X_int)>>logN;
	 	float X = (float)(X_int - (data_w>>1)) / data_w;
 		float Y = (float)(Y_int - (data_w>>1)) / data_w;
		
		float phase2pi;
		float phase255;
		float SLMre = 0;
		float SLMim = 0;
				
		for (int ii=0; ii<N_spots; ++ii)
		{
			phase2pi = M_PI * s_z[ii] * (X*X + Y*Y) + 2.0 * M_PI * (X * (s_x[ii]) + Y * (s_y[ii]) );
			SLMre = SLMre + s_a[ii] * cosf(phase2pi);
			SLMim = SLMim + s_a[ii] * sinf(phase2pi); 
		}
		
		phase2pi = atan2f(SLMim, SLMre) + M_PI;	
		if (UseAberrationCorr_b)
		{
			float AberrationCorr_f = d_AberrationCorr_f[idx];
			phase2pi = phase2pi + AberrationCorr_f;
			phase2pi = phase2pi - (2.0*M_PI) * floor(phase2pi / (2.0*M_PI));
		}

		if (ApplyLUT_b) 
		{
			if (!UseLUTPol_b)
			{
				phase255 = 255.0 * phase2pi / (2.0 * M_PI);
				g_SLMuc[idx] = s_LUT[(unsigned char)phase255];
			}
			else
				g_SLMuc[idx] = applyPolLUT(phase2pi, X, Y, s_LUTcoeff, N_PolCoeff);
		}
		else
		{
			phase255 = 255.0 * phase2pi / (2.0 * M_PI);
			g_SLMuc[idx] = (unsigned char)phase255;
		}
	}	
	__syncthreads();

}

__global__ void checkAmplitudes(float *g_x, float *g_y, float *g_z, unsigned char *g_pSLM_uc, float *g_amps, int N_spots, unsigned int N_pixels, int data_w)
{
	int blockSize = 512;
	int spot_number = blockIdx.x;
	unsigned int tid = threadIdx.x;
	unsigned int i = tid;
	
	__shared__ float s_Vre[512];
	__shared__ float s_Vim[512];
	__shared__ float s_xm, s_ym, s_zm;

	s_Vre[tid] = 0;
	s_Vim[tid] = 0;
		
	float N = data_w;
	int logN = (int)log2(N);
	float d = 0.001953125;	//SLM pixel size (1/512)
	
	if (tid == 0)
		s_xm = g_x[spot_number];
	if (tid == 64)
		s_ym = g_y[spot_number];	
	if (tid == 128)
		s_zm = g_z[spot_number];
	
	float X1 = d * ((float)tid - 256.0);
	float Y1 = - d * 256.0;	
	__syncthreads();
		
	while (i < N_pixels) 
	{ 
		float pSLM_1 = 2*M_PI*(float)g_pSLM_uc[i] - M_PI;
		float p = pSLM_1 - M_PI * (s_zm * (X1*X1 + Y1*Y1) + 2 * (X1 * s_xm + Y1 * s_ym));
		
		s_Vre[tid] += cosf(p);
		s_Vim[tid] += sinf(p);

		i += blockSize;
		Y1 += d; 	
	}
	__syncthreads();
 
	if (tid < 256) 
	{ 
		s_Vre[tid] += s_Vre[tid + 256]; 
		s_Vim[tid] += s_Vim[tid + 256];
	} 
	__syncthreads(); 

	if (tid < 128)
	{ 
		s_Vre[tid] += s_Vre[tid + 128];
		s_Vim[tid] += s_Vim[tid + 128];  
	} 
	__syncthreads(); 

	if (tid < 64) 
	{ 
		s_Vre[tid] += s_Vre[tid + 64];
		s_Vim[tid] += s_Vim[tid + 64]; 
	} 
	__syncthreads(); 

	
	if (tid < 32) {

		s_Vre[tid] += s_Vre[tid + 32];
		s_Vim[tid] += s_Vim[tid + 32];

		s_Vre[tid] += s_Vre[tid + 16];
		s_Vim[tid] += s_Vim[tid + 16];

		s_Vre[tid] += s_Vre[tid + 8];
		s_Vim[tid] += s_Vim[tid + 8];

		s_Vre[tid] += s_Vre[tid + 4];
		s_Vim[tid] += s_Vim[tid + 4];

		s_Vre[tid] += s_Vre[tid + 2];
		s_Vim[tid] += s_Vim[tid + 2];

		s_Vre[tid] += s_Vre[tid + 1];
		s_Vim[tid] += s_Vim[tid + 1];
		
	}
	if (tid == 0) 
	{
		float Vre = s_Vre[0] / 262144.0;
		float Vim = s_Vim[0] / 262144.0;
		g_amps[spot_number] = hypotf(Vim, Vre);
	}
}
/*
__global__ void f2uc(unsigned char *g_uc, float *g_f, int N_pixels, unsigned char *g_LUT, int use_LUT, float *g_LUT_coeff, int LUT_order, int data_w, float *wfcorrection, int use_correction)
{
	__shared__ unsigned char s_LUT[256];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float phase2pi = g_f[idx] + M_PI;
	if (use_correction == 1)
		phase2pi = phase2pi + wfcorrection[idx];
		
	if (use_LUT == 0)	
	{
		if (idx<N_pixels)
		{
			g_uc[idx] = (unsigned char)(((phase2pi)*255.0/(2.0*M_PI)));
		}	
	}
	else if (use_LUT == 1)
	{
		int tid = threadIdx.x;
		if (tid < 256)
		{
			int j = 0;
			while (j < 256)
			{
				s_LUT[tid + j] = g_LUT[tid + j];
				j += block_size;
			}
		}
		__syncthreads();
		if (idx<N_pixels)
		{
			g_uc[idx] = s_LUT[(unsigned char)(((phase2pi)*255.0/(2.0*M_PI)))];
		}	
	}
	else if (use_LUT == 2)
	{
		float N = (float)data_w;
		float X = idx%data_w;
 		float Y = floor(idx/N);

		__syncthreads();
		float phase255;
		if (LUT_order==7)
		{
			__shared__ float s_c[120];
			if (threadIdx.x < 120)
				s_c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
			__syncthreads();
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5) + s_c[56]*powf(X,6) + s_c[57]*powf(X,5)*Y + s_c[58]*powf(X,5)*phase2pi + s_c[59]*powf(X,4)*powf(Y,2) + s_c[60]*powf(X,4)*Y*phase2pi + s_c[61]*powf(X,4)*powf(phase2pi,2) + s_c[62]*powf(X,3)*powf(Y,3) + s_c[63]*powf(X,3)*powf(Y,2)*phase2pi + s_c[64]*powf(X,3)*Y*powf(phase2pi,2) + s_c[65]*powf(X,3)*powf(phase2pi,3) + s_c[66]*powf(X,2)*powf(Y,4) + s_c[67]*powf(X,2)*powf(Y,3)*phase2pi + s_c[68]*powf(X,2)*powf(Y,2)*powf(phase2pi,2) + s_c[69]*powf(X,2)*Y*powf(phase2pi,3) + s_c[70]*powf(X,2)*powf(phase2pi,4) + s_c[71]*X*powf(Y,5) + s_c[72]*X*powf(Y,4)*phase2pi + s_c[73]*X*powf(Y,3)*powf(phase2pi,2) + s_c[74]*X*powf(Y,2)*powf(phase2pi,3) + s_c[75]*X*Y*powf(phase2pi,4) + s_c[76]*X*powf(phase2pi,5) + s_c[77]*powf(Y,6) + s_c[78]*powf(Y,5)*phase2pi + s_c[79]*powf(Y,4)*powf(phase2pi,2) + s_c[80]*powf(Y,3)*powf(phase2pi,3) + s_c[81]*powf(Y,2)*powf(phase2pi,4) + s_c[82]*Y*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6) + s_c[84]*powf(X,7) + s_c[85]*powf(X,6)*Y + s_c[86]*powf(X,6)*phase2pi + s_c[87]*powf(X,5)*powf(Y,2) + s_c[88]*powf(X,5)*Y*phase2pi + s_c[89]*powf(X,5)*powf(phase2pi,2) + s_c[90]*powf(X,4)*powf(Y,3) + s_c[91]*powf(X,4)*powf(Y,2)*phase2pi + s_c[92]*powf(X,4)*Y*powf(phase2pi,2) + s_c[93]*powf(X,4)*powf(phase2pi,3) + s_c[94]*powf(X,3)*powf(Y,4) + s_c[95]*powf(X,3)*powf(Y,3)*phase2pi + s_c[96]*powf(X,3)*powf(Y,2)*powf(phase2pi,2) + s_c[97]*powf(X,3)*Y*powf(phase2pi,3) + s_c[98]*powf(X,3)*powf(phase2pi,4) + s_c[99]*powf(X,2)*powf(Y,5) + s_c[100]*powf(X,2)*powf(Y,4)*phase2pi + s_c[101]*powf(X,2)*powf(Y,3)*powf(phase2pi,2) + s_c[102]*powf(X,2)*powf(Y,2)*powf(phase2pi,3) + s_c[103]*powf(X,2)*Y*powf(phase2pi,4) + s_c[104]*powf(X,2)*powf(phase2pi,5) + s_c[105]*X*powf(Y,6) + s_c[106]*X*powf(Y,5)*phase2pi + s_c[107]*X*powf(Y,4)*powf(phase2pi,2) + s_c[108]*X*powf(Y,3)*powf(phase2pi,3) + s_c[109]*X*powf(Y,2)*powf(phase2pi,4) + s_c[110]*X*Y*powf(phase2pi,5) + s_c[111]*X*powf(phase2pi,6) + s_c[112]*powf(Y,7) + s_c[113]*powf(Y,6)*phase2pi + s_c[114]*powf(Y,5)*powf(phase2pi,2) + s_c[115]*powf(Y,4)*powf(phase2pi,3) + s_c[116]*powf(Y,3)*powf(phase2pi,4) + s_c[117]*powf(Y,2)*powf(phase2pi,5) + s_c[118]*Y*powf(phase2pi,6) + s_c[119]*powf(phase2pi,7));
		}
		else if (LUT_order==6)
		{
			__shared__ float s_c[84];
			if (threadIdx.x < 84)
				s_c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
			__syncthreads();
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5) + s_c[56]*powf(X,6) + s_c[57]*powf(X,5)*Y + s_c[58]*powf(X,5)*phase2pi + s_c[59]*powf(X,4)*powf(Y,2) + s_c[60]*powf(X,4)*Y*phase2pi + s_c[61]*powf(X,4)*powf(phase2pi,2) + s_c[62]*powf(X,3)*powf(Y,3) + s_c[63]*powf(X,3)*powf(Y,2)*phase2pi + s_c[64]*powf(X,3)*Y*powf(phase2pi,2) + s_c[65]*powf(X,3)*powf(phase2pi,3) + s_c[66]*powf(X,2)*powf(Y,4) + s_c[67]*powf(X,2)*powf(Y,3)*phase2pi + s_c[68]*powf(X,2)*powf(Y,2)*powf(phase2pi,2) + s_c[69]*powf(X,2)*Y*powf(phase2pi,3) + s_c[70]*powf(X,2)*powf(phase2pi,4) + s_c[71]*X*powf(Y,5) + s_c[72]*X*powf(Y,4)*phase2pi + s_c[73]*X*powf(Y,3)*powf(phase2pi,2) + s_c[74]*X*powf(Y,2)*powf(phase2pi,3) + s_c[75]*X*Y*powf(phase2pi,4) + s_c[76]*X*powf(phase2pi,5) + s_c[77]*powf(Y,6) + s_c[78]*powf(Y,5)*phase2pi + s_c[79]*powf(Y,4)*powf(phase2pi,2) + s_c[80]*powf(Y,3)*powf(phase2pi,3) + s_c[81]*powf(Y,2)*powf(phase2pi,4) + s_c[82]*Y*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6));
		}
		else if (LUT_order==5)
		{
			__shared__ float s_c[56];
			if (threadIdx.x < 56)
				s_c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
			__syncthreads();
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5));
		}
		else if (LUT_order==4)
		{
			__shared__ float s_c[36];
			if (threadIdx.x < 36)
				s_c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
			__syncthreads();
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4));
		}
		else if (LUT_order==3)
		{
			__shared__ float s_c[20];
			if (threadIdx.x < 20)
				s_c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
			__syncthreads();
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3));
		}
		if (phase255 < 0)
			phase255 = 0;	
		g_uc[idx] = (unsigned char)phase255;
	}
	__syncthreads();
}*/
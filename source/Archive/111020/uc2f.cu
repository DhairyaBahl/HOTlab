#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU Lesser General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Lesser General Public License for more details.

    You should have received a copy of the GNU Lesser General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
////////////////////////////////////////////////////////////////////////////////
//Convert unsigned char to float
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA.h"

__global__ void uc2f(float *g_f, unsigned char *g_uc, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<N)
	{
 		float in = g_uc[idx];
 		g_f[idx] = (float)in*(2.0f*M_PI)/255.0f - M_PI;
	}
	__syncthreads();
}
#include "hip/hip_runtime.h"
#include "GenerateHologramCUDA_PCI.h"

__global__ void LensesAndPrisms(float *g_x, float *g_y, float *g_z, unsigned char *g_SLMuc, int N_spots, float *g_LUT_coeff, int LUT_on, int data_w)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	
	__shared__ float s_x[block_size];
	__shared__ float s_y[block_size];
	__shared__ float s_z[block_size];
	
	if (tid < N_spots)
	{
		s_x[tid] = g_x[tid];
		s_y[tid] = g_y[tid];
		s_z[tid] = g_z[tid];
	}
	
	__syncthreads();
	
	if (idx < data_w*data_w)
	{
		float N = data_w;
		int logN = (int)log2(N);
		int X_int = idx&(int)(N-1);
	 	//int Y_int = (idx&(int)(N*N-1)-X_int)>>logN;
	 	int Y_int = (idx-X_int)>>logN;
	 	float X = (float)(X_int - (data_w>>1)) / data_w;
 		float Y = (float)(Y_int - (data_w>>1)) / data_w;
		
		float phase2pi;
		float phase255;
		float SLMre = 0;
		float SLMim = 0;
				
		for (int ii=0; ii<N_spots; ++ii)
		{
			phase2pi = M_PI * s_z[ii] * (X*X + Y*Y) + 2.0 * M_PI * (X * (s_x[ii]) + Y * (s_y[ii]) );
			SLMre = SLMre + cosf(phase2pi);
			SLMim = SLMim + sinf(phase2pi); 
		}
		
		phase2pi = atan2f(SLMim, SLMre) + M_PI;	
		
		if (LUT_on == 1)
		{
			__shared__ float s_c[N_LUT_coeff];
			if (threadIdx.x <N_LUT_coeff)
				s_c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
			__syncthreads();
			
			phase255 = 255.0 - (s_c[0] + s_c[1]*X_int + s_c[2]*Y_int + s_c[3]*phase2pi + s_c[4]*powf(X_int,2) + s_c[5]*X_int*Y_int + s_c[6]*X_int*phase2pi + s_c[7]*powf(Y_int,2) + s_c[8]*Y_int*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X_int,3) + s_c[11]*powf(X_int,2)*Y_int + s_c[12]*powf(X_int,2)*phase2pi + s_c[13]*X_int*powf(Y_int,2) + s_c[14]*X_int*Y_int*phase2pi + s_c[15]*X_int*powf(phase2pi,2) + s_c[16]*powf(Y_int,3) + s_c[17]*powf(Y_int,2)*phase2pi + s_c[18]*Y_int*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X_int,4) + s_c[21]*powf(X_int,3)*Y_int + s_c[22]*powf(X_int,3)*phase2pi + s_c[23]*powf(X_int,2)*powf(Y_int,2) + s_c[24]*powf(X_int,2)*Y_int*phase2pi + s_c[25]*powf(X_int,2)*powf(phase2pi,2) + s_c[26]*X_int*powf(Y_int,3) + s_c[27]*X_int*powf(Y_int,2)*phase2pi + s_c[28]*X_int*Y_int*powf(phase2pi,2) + s_c[29]*X_int*powf(phase2pi,3) + s_c[30]*powf(Y_int,4) + s_c[31]*powf(Y_int,3)*phase2pi + s_c[32]*powf(Y_int,2)*powf(phase2pi,2) + s_c[33]*Y_int*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X_int,5) + s_c[36]*powf(X_int,4)*Y_int + s_c[37]*powf(X_int,4)*phase2pi + s_c[38]*powf(X_int,3)*powf(Y_int,2) + s_c[39]*powf(X_int,3)*Y_int*phase2pi + s_c[40]*powf(X_int,3)*powf(phase2pi,2) + s_c[41]*powf(X_int,2)*powf(Y_int,3) + s_c[42]*powf(X_int,2)*powf(Y_int,2)*phase2pi + s_c[43]*powf(X_int,2)*Y_int*powf(phase2pi,2) + s_c[44]*powf(X_int,2)*powf(phase2pi,3) + s_c[45]*X_int*powf(Y_int,4) + s_c[46]*X_int*powf(Y_int,3)*phase2pi + s_c[47]*X_int*powf(Y_int,2)*powf(phase2pi,2) + s_c[48]*X_int*Y_int*powf(phase2pi,3) + s_c[49]*X_int*powf(phase2pi,4) + s_c[50]*powf(Y_int,5) + s_c[51]*powf(Y_int,4)*phase2pi + s_c[52]*powf(Y_int,3)*powf(phase2pi,2) + s_c[53]*powf(Y_int,2)*powf(phase2pi,3) + s_c[54]*Y_int*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5) + s_c[56]*powf(X_int,6) + s_c[57]*powf(X_int,5)*Y_int + s_c[58]*powf(X_int,5)*phase2pi + s_c[59]*powf(X_int,4)*powf(Y_int,2) + s_c[60]*powf(X_int,4)*Y_int*phase2pi + s_c[61]*powf(X_int,4)*powf(phase2pi,2) + s_c[62]*powf(X_int,3)*powf(Y_int,3) + s_c[63]*powf(X_int,3)*powf(Y_int,2)*phase2pi + s_c[64]*powf(X_int,3)*Y_int*powf(phase2pi,2) + s_c[65]*powf(X_int,3)*powf(phase2pi,3) + s_c[66]*powf(X_int,2)*powf(Y_int,4) + s_c[67]*powf(X_int,2)*powf(Y_int,3)*phase2pi + s_c[68]*powf(X_int,2)*powf(Y_int,2)*powf(phase2pi,2) + s_c[69]*powf(X_int,2)*Y_int*powf(phase2pi,3) + s_c[70]*powf(X_int,2)*powf(phase2pi,4) + s_c[71]*X_int*powf(Y_int,5) + s_c[72]*X_int*powf(Y_int,4)*phase2pi + s_c[73]*X_int*powf(Y_int,3)*powf(phase2pi,2) + s_c[74]*X_int*powf(Y_int,2)*powf(phase2pi,3) + s_c[75]*X_int*Y_int*powf(phase2pi,4) + s_c[76]*X_int*powf(phase2pi,5) + s_c[77]*powf(Y_int,6) + s_c[78]*powf(Y_int,5)*phase2pi + s_c[79]*powf(Y_int,4)*powf(phase2pi,2) + s_c[80]*powf(Y_int,3)*powf(phase2pi,3) + s_c[81]*powf(Y_int,2)*powf(phase2pi,4) + s_c[82]*Y_int*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6) + s_c[84]*powf(X_int,7) + s_c[85]*powf(X_int,6)*Y_int + s_c[86]*powf(X_int,6)*phase2pi + s_c[87]*powf(X_int,5)*powf(Y_int,2) + s_c[88]*powf(X_int,5)*Y_int*phase2pi + s_c[89]*powf(X_int,5)*powf(phase2pi,2) + s_c[90]*powf(X_int,4)*powf(Y_int,3) + s_c[91]*powf(X_int,4)*powf(Y_int,2)*phase2pi + s_c[92]*powf(X_int,4)*Y_int*powf(phase2pi,2) + s_c[93]*powf(X_int,4)*powf(phase2pi,3) + s_c[94]*powf(X_int,3)*powf(Y_int,4) + s_c[95]*powf(X_int,3)*powf(Y_int,3)*phase2pi + s_c[96]*powf(X_int,3)*powf(Y_int,2)*powf(phase2pi,2) + s_c[97]*powf(X_int,3)*Y_int*powf(phase2pi,3) + s_c[98]*powf(X_int,3)*powf(phase2pi,4) + s_c[99]*powf(X_int,2)*powf(Y_int,5) + s_c[100]*powf(X_int,2)*powf(Y_int,4)*phase2pi + s_c[101]*powf(X_int,2)*powf(Y_int,3)*powf(phase2pi,2) + s_c[102]*powf(X_int,2)*powf(Y_int,2)*powf(phase2pi,3) + s_c[103]*powf(X_int,2)*Y_int*powf(phase2pi,4) + s_c[104]*powf(X_int,2)*powf(phase2pi,5) + s_c[105]*X_int*powf(Y_int,6) + s_c[106]*X_int*powf(Y_int,5)*phase2pi + s_c[107]*X_int*powf(Y_int,4)*powf(phase2pi,2) + s_c[108]*X_int*powf(Y_int,3)*powf(phase2pi,3) + s_c[109]*X_int*powf(Y_int,2)*powf(phase2pi,4) + s_c[110]*X_int*Y_int*powf(phase2pi,5) + s_c[111]*X_int*powf(phase2pi,6) + s_c[112]*powf(Y_int,7) + s_c[113]*powf(Y_int,6)*phase2pi + s_c[114]*powf(Y_int,5)*powf(phase2pi,2) + s_c[115]*powf(Y_int,4)*powf(phase2pi,3) + s_c[116]*powf(Y_int,3)*powf(phase2pi,4) + s_c[117]*powf(Y_int,2)*powf(phase2pi,5) + s_c[118]*Y_int*powf(phase2pi,6) + s_c[119]*powf(phase2pi,7));
			if (phase255 >= 0)
				g_SLMuc[idx] = (unsigned char)phase255;
			else
				g_SLMuc[idx] = 0;	
		}
		else 
		{
			phase255 = 255.0 * phase2pi / (2.0 * M_PI);
			g_SLMuc[idx] = (unsigned char)phase255;
		}
	}	
	__syncthreads();

}

			
				
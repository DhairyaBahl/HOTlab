#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Hologram generating algorithms for CUDA Devices
// Written by Martin Persson 2009
// martin.persson@physics.gu.se
////////////////////////////////////////////////////////////////////////////////
//The function "GenerateHologram" contains three different algorithms for
//hologram generation. The last parameter in the function call selects which 
//one to use:
//0: Complex addition of "Lenses and Prisms", no optimization (3D)
//1: Weighted Gerchberg-Saxton algorithm using Fresnel propagation (3D)
//2: Weighted Gerchberg-Saxton algorithm using fast fourier transforms (2D)
//3: Different implementation of (1)
//4: Selects among (0), (1) and (3) to optimize speed according to the desired 
//   number of traps
//
//-(3) is written to minimize the use of global memory. The computational time of 
// (3) is only weakly dependent on the number of traps, it should be choosen if 
// more than 4 traps is desired. 
//-(0) produces optimal holograms for 1 or 2 traps and is significantly faster 
// than the others.
////////////////////////////////////////////////////////////////////////////////
//Fresnel propagation based algorithm (1 and 3) described in:
//Roberto Di Leonardo, Francesca Ianni, and Giancarlo Ruocco
//"Computer generation of optimal holograms for optical trap arrays"
//Opt. Express 15, 1913-1922 (2007) 
////////////////////////////////////////////////////////////////////////////////
//Possible improvements:
//-Put all arguments for device functions and trap positions in constant memory. 
// (Requires all functions to be moved into the same file or the use of some 
// workaround found on nVidia forum)  	
//-Put pSLM_start and aLaser in texture memory
//-Use "zero-copy" to trasfer pSLM to host. (will only work on 1.3 devices and higher)
//-Rename functions and variables for consistency and readability
////////////////////////////////////////////////////////////////////////////////


#include "GenerateHologramCUDA_PCI.h"

//////////////////////////////////////////////////
//delarations for the DFT version
//////////////////////////////////////////////////
float *d_x, *d_y, *d_z;							//trap coordinates in GPU memory
float *delta, *d_delta;							//distance between pixels on SLM and trap positions in host and GPU memory
float *d_ei_dre, *d_ei_dim;						//exp(i*re(delta)), exp(i*re(delta))
float *d_Vim, *d_Vre;							//energy flux contribution from each SLM pixel to each trap 
float *d_VimR, *d_VreR;							//sum of above for each trap
float *d_aLaserFFT, *d_aLaserDFT;				//laser amplitude distribution on SLM
float *d_pSLM;									//the optimized phase pattern, float
float *d_weights, *d_amps;						//used weights and calculated amplitudes for each spot and each iteration
float *d_pSLM_start;							//Initial phase pattern
float *d_LUT_coeff;
float *d_spotRe, *d_spotIm;

//////////////////////////////////////////////////
//delarations for the FFT version
//////////////////////////////////////////////////
hipfftComplex *d_SLM_cc;
hipfftHandle plan;
int mem_size_cC;
hipfftComplex *d_FFTo, *d_FFTd;
int *d_spot_index;


int n_blocks_V, n_blocks_Phi, memsize_SLM_f, memsize_SLM_uc, memsize_SLM_cc, memsize_V_f, memsize_spots_f, N_spots_last, N_pixels, N_iterations_last;

unsigned char *d_pSLM_uc;						//The optimized phase pattern, unsigned char, the one sent to the SLM
int blockSize = block_size;
int maxThreads = 128;
int maxBlocks = 128;
int maxThreads_device;
bool bEnableSLM;


////////////////////////////////////////////////////////////////////////////////
// Functions to talk to PCIe Hardware
////////////////////////////////////////////////////////////////////////////////
/*extern "C" void InitalizePCIe(
	bool bRAMWriteEnable
);

extern "C" void LoadImg(
	unsigned char* Img
);

extern "C" void Wait(
	int DelayMs
);

extern "C" void SetPower(
	bool bPower
);

extern "C" void ShutDownPCIe();
*/

////////////////////////////////////////////////////////////////////////////////
//Run cuGS 
////////////////////////////////////////////////////////////////////////////////

extern "C" __declspec(dllexport) float GenerateHologram(float *h_test, unsigned char *h_pSLM, float *x_spots, float *y_spots, float *z_spots, int N_spots, int data_w, int N_iterations, float *weights, float alpha, int method, int spatial_LUT)
{
	int retur = 0;
	float alpha_RPC = alpha*2.0f*M_PI;
	hipMemcpy(d_x, x_spots, memsize_spots_f, hipMemcpyHostToDevice);	
	hipMemcpy(d_y, y_spots, memsize_spots_f, hipMemcpyHostToDevice);	
	hipMemcpy(d_z, z_spots, memsize_spots_f, hipMemcpyHostToDevice);
	
	if (method == 4)
	{
		if (N_spots < 3)
			method = 0;
		else if (N_spots < 5)
			method = 1;
		else 
			method = 3;
	}
	
	if (method == 0)
	{
		LensesAndPrisms<<< n_blocks_Phi, block_size >>>(d_x, d_y, d_z, d_pSLM_uc, N_spots, d_LUT_coeff, spatial_LUT, data_w);
	}
	
	if (method == 1)				//Gerchberg-Saxton using fresnel propagation
	{
		
		if (N_spots != N_spots_last)
		{
			n_blocks_V = (N_spots*N_pixels/block_size + ((N_spots*N_pixels)%block_size == 0 ? 0:1));
			memsize_V_f = N_pixels*N_spots*sizeof(float);
			memsize_spots_f = N_spots*sizeof(float);
				
			if (N_spots > N_spots_last)
			{	
	
				for (int i=0; i < N_spots; ++i)
				{
					weights[i] = 1;
				} 
				hipFree(d_delta);
				hipFree(d_ei_dre);
				hipFree(d_ei_dim);
				hipFree(d_Vre);
				hipFree(d_Vim);
				hipFree(d_VreR);
				hipFree(d_VimR);
				hipFree(d_weights);
				hipFree(d_amps);

				hipMalloc((void**)&d_delta, memsize_V_f);
				hipMalloc((void**)&d_ei_dre, memsize_V_f);
				hipMalloc((void**)&d_ei_dim, memsize_V_f);
				hipMalloc((void**)&d_Vre, memsize_V_f);
				hipMalloc((void**)&d_Vim, memsize_V_f);
				hipMalloc((void**)&d_VreR, memsize_V_f);
				hipMalloc((void**)&d_VimR, memsize_V_f);
				hipMalloc((void**)&d_amps, N_spots_last*(N_iterations_last)*sizeof(float));
				hipMalloc((void**)&d_weights, N_spots*(N_iterations+1)*sizeof(float));
				
				hipMemcpy(d_weights, weights, N_spots*(N_iterations+1)*sizeof(float), hipMemcpyHostToDevice);	
			}
		}

		hipMemcpy(d_pSLM_start, d_pSLM, memsize_SLM_f, hipMemcpyDeviceToDevice);
		hipDeviceSynchronize();
		
		computeDelta <<< n_blocks_V, block_size >>> (d_x, d_y, d_z, d_delta, d_ei_dre, d_ei_dim, N_spots, N_pixels, data_w);
		hipDeviceSynchronize();

		for (int l=0; l<N_iterations; l++)
		{	
			// Compute complex amplitudes in spots (results will be located at spot number * N_pixels in d_Vre and d_Vim)
			computeV<<< n_blocks_Phi, block_size >>>(d_Vre, d_Vim, d_pSLM, d_delta, N_pixels, N_spots, d_aLaserDFT);
			hipDeviceSynchronize();
			
			for (int ll=0; ll<N_spots; ll++)
			{
				int offset = ll * N_pixels;
				Reduce(N_pixels, maxThreads, maxBlocks, d_Vre, d_VreR, offset);
				Reduce(N_pixels, maxThreads, maxBlocks, d_Vim, d_VimR, offset);
				hipDeviceSynchronize();
			}
		
			computeWeights<<<1, N_spots>>>(d_VreR, d_VimR, N_spots, d_weights, l, d_amps, N_pixels);
			computePhi<<< n_blocks_Phi, block_size >>>(d_VreR, d_VimR, d_pSLM, d_ei_dre, d_ei_dim, N_pixels, N_spots, d_weights, (l+1), d_pSLM_start, alpha_RPC, d_amps);	
		}
		hipDeviceSynchronize();
		f2uc<<< n_blocks_Phi, block_size >>>(d_pSLM_uc, d_pSLM, N_pixels, d_LUT_coeff, spatial_LUT, data_w);
	}
	else if (method ==2)			//Gerchberg-Saxton using fast fourier transforms 
	{
		float amp_desired = N_pixels * sqrt(1.0f/(float)N_spots);
		float weight = 1.0f/(float)N_spots;
		for (int i=0; i < N_spots; ++i)
		{
			weights[i] = weight;
		} 
		
		hipMemcpy(d_weights, weights, N_spots * sizeof(float), hipMemcpyHostToDevice);
		hipMemset(d_FFTd, 0, memsize_SLM_cc);
		
		XYtoIndex <<< 1, N_spots >>> (d_x,  d_y, d_spot_index, N_spots, data_w);
		
		for (int l=0; l<N_iterations; l++)
		{
			// Transform to FFT plane
			hipfftExecC2C(plan, d_SLM_cc, d_FFTo, HIPFFT_FORWARD);
			hipDeviceSynchronize();
		
			// Copy phases for spot indices in d_FFTo to d_FFTd
			usePhasesW <<< 1, N_spots >>> (d_FFTo, d_FFTd, d_spot_index, N_spots, l, d_amps, d_weights, amp_desired);
			hipDeviceSynchronize();

			//Transform back to SLM plane
			hipfftExecC2C(plan, d_FFTd, d_SLM_cc, HIPFFT_BACKWARD);
			hipDeviceSynchronize();
			
			// Set amplitudes in d_SLM to the laser amplitudes
			resetAmplitudesRPC <<< n_blocks_Phi, block_size >>> (d_aLaserFFT, d_SLM_cc, d_pSLM_start, N_pixels, alpha_RPC);
			hipDeviceSynchronize();
			
			retur = l;
		}	
		// Calculate phases in the FFT plane   
		getPhases<<< n_blocks_Phi, block_size >>> (d_pSLM_uc, d_pSLM_start, d_SLM_cc, d_LUT_coeff, spatial_LUT, data_w);
	}
	
	else if (method == 3)		//Alternative implementation of Gerchberg-Saxton using fresnel propagation
	{
		hipMemcpy(d_pSLM_start, d_pSLM, memsize_SLM_f, hipMemcpyDeviceToDevice);
		hipDeviceSynchronize();

		for (int l=0; l<N_iterations; l++)
		{	
			transformToFarfield<<< N_spots, 512>>>(d_x, d_y, d_z, d_pSLM, d_aLaserDFT, d_spotRe, d_spotIm, N_spots, N_pixels, data_w);
			hipDeviceSynchronize();
			
			computePhiNew<<< 512, 512 >>>(d_x, d_y, d_z, d_spotRe, d_spotIm, d_pSLM, N_pixels, N_spots, d_weights, l, d_pSLM_start, alpha_RPC, d_amps);
			hipDeviceSynchronize();
		}

		f2uc<<< n_blocks_Phi, block_size >>>(d_pSLM_uc, d_pSLM, N_pixels, d_LUT_coeff, spatial_LUT, data_w);
		
	}
	hipDeviceSynchronize();			
	hipMemcpy(h_pSLM, d_pSLM_uc, memsize_SLM_uc, hipMemcpyDeviceToHost);  //testa zero-copy ist�llet fr�n computePhi

	//load image to the PCIe hardware  SLMstuff
	//if(bEnableSLM)
	//	LoadImg(h_pSLM);
	
	//hipMemcpy(weights, d_weights, N_spots*(N_iterations+1)*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(weights, d_amps, N_spots*(N_iterations)*sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(h_test, d_aLaserDFT, memsize_SLM_f, hipMemcpyDeviceToHost);
	N_spots_last = N_spots;
	return retur;	
}

////////////////////////////////////////////////////////////////////////////////
//Enable SLM for cuGSdft
////////////////////////////////////////////////////////////////////////////////
extern "C" __declspec(dllexport) int startCUDAandSLM(int SLM_enabled, float beam_center_x, float beam_center_y, float beam_radius, float aperture_radius, int data_w,  float *test, unsigned char *h_pSLM, float *LUT_coeff)
{
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    maxThreads_device = deviceProp.maxThreadsPerBlock;
	int MaxSpots = block_size;
	int MaxIterations = 1000;
	float term1, term2, term3, Amplitude;
	N_pixels = data_w * data_w;
	N_spots_last = 8;
	N_iterations_last = 10;
	memsize_V_f = N_pixels * N_spots_last*sizeof(float);
	memsize_spots_f = MaxSpots * sizeof(float);
	memsize_SLM_f = N_pixels * sizeof(float);  
    memsize_SLM_uc = N_pixels * sizeof(unsigned char);
    memsize_SLM_cc = N_pixels * sizeof(hipfftComplex);
    n_blocks_Phi = (N_pixels/block_size + (N_pixels%block_size == 0 ? 0:1));
    n_blocks_V = (N_spots_last*N_pixels/block_size + ((N_spots_last*N_pixels)%block_size == 0 ? 0:1));
	float *h_aLaserFFT = (float *)malloc(memsize_SLM_f);
	float *h_aLaserDFT = (float *)malloc(memsize_SLM_f);	
	float weights[10000];
	for (int i=0; i < N_spots_last; ++i)
	{
		weights[i] = 1;//weight;
	} 
	hipMalloc((void**)&d_LUT_coeff, N_LUT_coeff * sizeof(float));
	hipMalloc((void**)&d_aLaserFFT, memsize_SLM_f);
	hipMalloc((void**)&d_aLaserDFT, memsize_SLM_f);
	hipMalloc((void**)&d_x, memsize_spots_f );
	hipMalloc((void**)&d_y, memsize_spots_f );
	hipMalloc((void**)&d_z, memsize_spots_f );
	hipMalloc((void**)&d_spotRe, memsize_spots_f );
	hipMalloc((void**)&d_spotIm, memsize_spots_f );
	hipMalloc((void**)&d_spot_index, MaxSpots * sizeof(int));
	hipMalloc((void**)&d_delta, memsize_V_f);
	hipMalloc((void**)&d_ei_dre, memsize_V_f);
	hipMalloc((void**)&d_ei_dim, memsize_V_f);
	hipMalloc((void**)&d_Vre, memsize_V_f);
	hipMalloc((void**)&d_Vim, memsize_V_f);
	hipMalloc((void**)&d_VreR, memsize_V_f);
	hipMalloc((void**)&d_VimR, memsize_V_f);
	hipMalloc((void**)&d_weights, MaxSpots*(MaxIterations+1)*sizeof(float));
	hipMalloc((void**)&d_amps, MaxSpots*MaxIterations*sizeof(float));
	hipMalloc((void**)&d_pSLM, memsize_SLM_f);
	hipMalloc((void**)&d_pSLM_start, memsize_SLM_f);
	hipMalloc((void**)&d_pSLM_uc, memsize_SLM_uc);
	hipMalloc((void**)&d_FFTd, memsize_SLM_cc);	
	hipMalloc((void**)&d_FFTo, memsize_SLM_cc);		
	hipMalloc((void**)&d_SLM_cc, memsize_SLM_cc);
	
	int IndexFFT, IndexDFT, row, col;
	for(row = 0; row < data_w; row++)
	{
		for(col = 0; col < data_w; col++)
		{
			//gaussian incident beam
			term1 = pow((float)row-beam_center_y,2.0f);
			term2 = pow((float)col-beam_center_x,2.0f);
			term3 = pow(beam_radius,2.0f);
			float Gauss = -(term1 + term2)/term3;
			Amplitude = exp(Gauss);
			//shift around my gaussian because we don't have an FFTShift equiv with CUDA
			if(row < (data_w/2) && col < (data_w/2))
				IndexFFT = ((row+(data_w/2))*data_w) + (col + (data_w/2));
			else if(row <(data_w/2) && col >= (data_w/2))
				IndexFFT = ((row+(data_w/2))*data_w) + (col - (data_w/2));
			else if(row >= (data_w/2) && col < (data_w/2))
				IndexFFT = ((row-(data_w/2))*data_w) + (col + (data_w/2));
			else
				IndexFFT = ((row-(data_w/2))*data_w) + (col - (data_w/2));
				
			//For the DFT version shifting is not needed
			IndexDFT = row*data_w + col;
			
			if (term1 + term2 < pow(aperture_radius, 2.0f)) 
			{
				h_aLaserFFT[IndexFFT] = Amplitude;
				h_aLaserDFT[IndexDFT] = Amplitude;
			}
			else
			{
				h_aLaserFFT[IndexFFT] = 0;
				h_aLaserDFT[IndexDFT] = 0;
			}
		}
	}

	hipMemcpy(d_pSLM_uc, h_pSLM, memsize_SLM_uc, hipMemcpyHostToDevice);
	uc2f<<< n_blocks_Phi, block_size >>>(d_pSLM, d_pSLM_uc, N_pixels);
	hipDeviceSynchronize();
	
	hipMemcpy(d_aLaserFFT, h_aLaserFFT, memsize_SLM_f, hipMemcpyHostToDevice);
	hipMemcpy(d_aLaserDFT, h_aLaserDFT, memsize_SLM_f, hipMemcpyHostToDevice);
	hipMemcpy(test, d_aLaserDFT, memsize_SLM_f, hipMemcpyDeviceToHost);
	hipMemcpy(d_weights, weights, N_spots_last*(N_iterations_last+1)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_LUT_coeff, LUT_coeff, N_LUT_coeff * sizeof(float), hipMemcpyHostToDevice);
	
	CUFFT_SAFE_CALL(hipfftPlan2d(&plan, data_w, data_w, HIPFFT_C2C));
	
	//Open up communication to the PCIe hardware
	/*bEnableSLM = SLM_enabled; //SLMstuff
	if(bEnableSLM)
	{
		bool bRAMWriteEnable = false;
		InitalizePCIe(bRAMWriteEnable);
		SetPower(true);
	}	
	*/
	free(h_aLaserDFT);
	free(h_aLaserFFT);		
	return memsize_V_f;
}

extern "C" __declspec(dllexport) int stopCUDAandSLM()
{
	hipFree(d_aLaserFFT);
	hipFree(d_aLaserDFT);	
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	hipFree(d_delta);
	hipFree(d_ei_dre);
	hipFree(d_ei_dim);
	hipFree(d_Vre);
	hipFree(d_Vim);
	hipFree(d_VreR);
	hipFree(d_VimR);
	hipFree(d_weights);
	hipFree(d_amps);
	hipFree(d_pSLM);
	hipFree(d_pSLM_start);
	hipFree(d_pSLM_uc);
	hipFree(d_FFTd);
	hipFree(d_FFTo);
	hipFree(d_SLM_cc);
	hipFree(d_LUT_coeff);
	
	hipfftDestroy(plan);
	
	//close out communication with the PCIe hardware SLMstuff
	//if(bEnableSLM)
	//	ShutDownPCIe();
	return 100;
}



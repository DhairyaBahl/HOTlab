#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//Convert unsigned char to float
////////////////////////////////////////////////////////////////////////////////

#include "cuDFTdll.h"

__global__ void uc2f(float *f, unsigned char *uc, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<N)
	{
 		float in = uc[idx];
 		f[idx] = (float)in*(2.0f*M_PI)/255.0f - M_PI;
	}
	__syncthreads();
}
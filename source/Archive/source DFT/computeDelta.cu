#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////////////
//Calculate one matrix for each spot containing its distance to each pixel on the SLM
///////////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA_PCI.h"

__global__ void computeDelta(float *g_x, float *g_y, float *g_z, float *g_delta, float *g_ei_delta_re, float *g_ei_delta_im, int N_spots, int N_pixels, int data_w)
{
	float N = data_w;
	int logN = (int)log2(N);
	float d = 1.0/N;
	
	__shared__ float s_x[block_size];
	__shared__ float s_y[block_size];
	__shared__ float s_z[block_size];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadIdx.x < N_spots)
	{
		s_x[threadIdx.x] = g_x[threadIdx.x];
		s_y[threadIdx.x] = g_y[threadIdx.x];
		s_z[threadIdx.x] = g_z[threadIdx.x];
	}
	__syncthreads();
	int X_int = idx&(int)(N-1);
 	int Y_int = (idx&(int)(N_pixels-1)-X_int)>>logN;		//(idx-X)>>logN gives error for z!=0
 	float X = d * (float)(X_int - 256);
 	float Y = d * (float)(Y_int - 256);
 	
 	for (int m=0; m<N_spots; m++)
	{	
		if (idx<((m+1)*N_pixels) && idx >= m*N_pixels )
		{
			float delta = M_PI * s_z[m] * (X*X + Y*Y) + 2.0 * M_PI * (X * s_x[m] + Y * s_y[m]);
			g_delta[idx] = delta;
			g_ei_delta_re[idx] = cosf(delta);  
			g_ei_delta_im[idx] = sinf(delta); 
		}
	}
	__syncthreads();
}

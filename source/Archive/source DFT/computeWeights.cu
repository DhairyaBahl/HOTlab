#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//Obtain phases in SLM plane
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA_PCI.h"

__global__ void computeWeights(float *g_Vre, float *g_Vim, int N_spots, float *g_weights, int iteration, float *g_amps, int N_pixels)
{

	int tid = threadIdx.x;	
	__shared__ float s_aSpot[block_size], s_aSpot_mean, s_wSpot[block_size], s_Vre[block_size], s_Vim[block_size];


	if (tid<N_spots)
	{
		s_Vre[tid] = g_Vre[tid*N_pixels];
		s_Vim[tid] = g_Vim[tid*N_pixels];
		s_aSpot[tid] = hypotf(s_Vim[tid], s_Vre[tid]);
		s_wSpot[tid] = g_weights[tid + iteration*N_spots];
	}

	__syncthreads();
	if  (tid==0)
	{
		float aSpot_sum = 0;
		for (int kk=0; kk<N_spots; kk++)
		{
			aSpot_sum += s_aSpot[kk];
		}
		s_aSpot_mean = aSpot_sum / N_spots; //integer division!!
	}
	__syncthreads();
	
	if (tid<N_spots)
	{
		s_wSpot[tid] = s_wSpot[tid] * s_aSpot_mean / s_aSpot[tid];	
		g_weights[tid + N_spots*(iteration+1)] = s_wSpot[tid];
		g_amps[tid + N_spots*iteration] = s_aSpot[tid];		
	}
}

#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//Obtain phases in SLM plane
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA_PCI.h"

// Requires computeWeights
__global__ void computePhi(float *g_Vre, float *g_Vim, float *g_Phi, float *g_ei_dre, float *g_ei_dim, int N_pixels, int N_spots, float *g_weights, int iteration, float *g_pSLM_start, float RPC, float *g_amps)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;	
	__shared__ float s_a[block_size], s_w[block_size], s_Vre[block_size], s_Vim[block_size];
	float re_SLM = 0, im_SLM = 0, Phi = 0, p_SLM_s = g_pSLM_start[idx], ei_dre, ei_dim;

	if (tid<N_spots)
	{
		s_Vre[tid] = g_Vre[tid*N_pixels];
		s_Vim[tid] = g_Vim[tid*N_pixels];
		s_a[tid] = g_amps[tid + (iteration-1)*N_spots];
		s_w[tid] = g_weights[tid + iteration*N_spots];
	}
	__syncthreads();
	for (int k=0; k<N_spots; k++)
	{
		ei_dre = g_ei_dre[idx + k*N_pixels];
		ei_dim = g_ei_dim[idx + k*N_pixels];
		re_SLM += s_w[k] * (ei_dre * s_Vre[k] - ei_dim * s_Vim[k]) / s_a[k];
		im_SLM += s_w[k] * (ei_dre * s_Vim[k] + s_Vre[k] * ei_dim) / s_a[k];
	}

	Phi = atan2f(im_SLM, re_SLM);
	
	
	if (RPC < (2.0f*M_PI))
	{	
		if ((fabs(Phi - p_SLM_s)) < RPC)
		{
			g_Phi[idx] = Phi;
		}
		else
		{
			g_Phi[idx] = p_SLM_s;
		}	
	}
	else	
	{
		g_Phi[idx] = Phi;
	}
}

/* Nice version 090909 19:12
__global__ void computePhi(float *g_Vre, float *g_Vim, float *g_Phi, float *g_ei_dre, float *g_ei_dim, int N_pixels, int N_spots, float *g_weights, int iteration, float *g_pSLM_start, float alpha, float *g_amps)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;	
	__shared__ float s_a[block_size], s_w[block_size], s_Vre[block_size], s_Vim[block_size];
	float re_SLM = 0, im_SLM = 0, Phi = 0, p_SLM_s = g_pSLM_start[idx], ei_dre, ei_dim;

	if (tid<N_spots)
	{
		s_Vre[tid] = g_Vre[tid*N_pixels];
		s_Vim[tid] = g_Vim[tid*N_pixels];
		s_a[tid] = g_amps[tid + (iteration-1)*N_spots];
		s_w[tid] = g_weights[tid + iteration*N_spots];
	}
	__syncthreads();
	for (int k=0; k<N_spots; k++)
	{
		ei_dre = g_ei_dre[idx + k*N_pixels];
		ei_dim = g_ei_dim[idx + k*N_pixels];
		re_SLM += s_w[k] * (ei_dre * s_Vre[k] - ei_dim * s_Vim[k]) / s_a[k];
		im_SLM += s_w[k] * (ei_dre * s_Vim[k] + s_Vre[k] * ei_dim) / s_a[k];
	}

	Phi = atan2f(im_SLM, re_SLM);
	
	
	if (alpha < (2.0f*M_PI))
	{	
		if ((fabs(Phi - p_SLM_s)) < alpha)
		{
			g_Phi[idx] = Phi;
		}
		else
		{
			g_Phi[idx] = p_SLM_s;
		}	
	}
	else	
	{
		g_Phi[idx] = Phi;
	}
}
*/
/*
__global__ void computePhi(float *g_Vre, float *g_Vim, float *g_Phi, float *g_ei_dre, float *g_ei_dim, int N_pixels, int N_spots, float *g_weights, int iteration, float *g_pSLM_start, float alpha, float *g_amps)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;	
	float s_Vre, s_Vim, s_w, s_a;
	float re_SLM = 0, im_SLM = 0, Phi = 0, p_SLM_s = g_pSLM_start[idx];

	for (int k=0; k<N_spots; k++)
	{
		s_Vre = g_Vre[k*N_pixels];
		s_Vim = g_Vim[k*N_pixels];
		s_w = g_weights[tid + iteration*N_spots];
		s_a = g_amps[tid + (iteration-1)*N_spots];
		re_SLM += s_w * (g_ei_dre[idx + k*N_pixels] * s_Vre - g_ei_dim[idx + k*N_pixels] * s_Vim) / s_a;
		im_SLM += s_w * (g_ei_dre[idx + k*N_pixels] * s_Vim + s_Vre * g_ei_dim[idx + k*N_pixels]) / s_a;
	}

	Phi = atan2f(im_SLM, re_SLM);
	
	
	if (alpha < (2.0f*M_PI))
	{	
		if ((fabs(Phi - p_SLM_s)) < alpha)
		{
			g_Phi[idx] = Phi;
		}
		else
		{
			g_Phi[idx] = p_SLM_s;
		}	
	}
	else	
	{
		g_Phi[idx] = Phi;
	}
}

*/



/* This version works 090909

__global__ void computePhi(float *g_Vre, float *g_Vim, float *g_Phi, float *g_ei_dre, float *g_ei_dim, int N_pixels, int N_spots, float *g_weights, int iteration, float *g_pSLM_start, float alpha)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;	
	__shared__ float s_a[block_size], a_mean, s_w[block_size], s_Vre[block_size], s_Vim[block_size];
	float re_SLM = 0, im_SLM = 0, Phi = 0, p_SLM_s = g_pSLM_start[idx];

	if (tid<N_spots)
	{
		s_Vre[tid] = g_Vre[tid*N_pixels];
		s_Vim[tid] = g_Vim[tid*N_pixels];
		s_a[tid] = hypotf(s_Vim[tid], s_Vre[tid]);
		s_w[tid] = g_weights[tid + iteration*N_spots];
	}
	__syncthreads();
	if  (tid==0)//(idx<N_pixels)
	{
		float a_sum = 0;
		for (int kk=0; kk<N_spots; kk++)
		{
			a_sum += s_a[kk];
		}
		a_mean = a_sum / N_spots; //integer division!!
	}
	__syncthreads();
	
	if (tid<N_spots)
	{
		s_w[tid] = s_w[tid] * a_mean / s_a[tid];	
		g_weights[tid + N_spots*(iteration+1)] = s_w[tid];		
	}
	__syncthreads();
	

	for (int k=0; k<N_spots; k++)
	{
		re_SLM += s_w[k] * (g_ei_dre[idx + k*N_pixels] * s_Vre[k] - g_ei_dim[idx + k*N_pixels] * s_Vim[k]) / s_a[k];
		im_SLM += s_w[k] * (g_ei_dre[idx + k*N_pixels] * s_Vim[k] + s_Vre[k] * g_ei_dim[idx + k*N_pixels]) / s_a[k];	
	}

	Phi = atan2f(im_SLM, re_SLM);
	
	
	if (alpha < (2.0f*M_PI))
	{	
		if ((fabs(Phi - p_SLM_s)) < alpha)
		{
			g_Phi[idx] = Phi;
		}
		else
		{
			g_Phi[idx] = p_SLM_s;
		}	
	}
	else	
	{
		g_Phi[idx] = Phi;
	}
}
*/
/*

Davids viktning


__global__ void computePhi(float *g_Vre, float *g_Vim, float *g_Phi, float *g_ei_dre, float *g_ei_dim, int N_pixels, int N_spots, float *g_weights, float *amp_d, int iteration)
{
	__syncthreads();
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;	
	__shared__ float s_a[block_size], a_d[block_size], w_sum, s_w[block_size], s_Vre[block_size], s_Vim[block_size];
	float re_SLM = 0, im_SLM = 0, float gamma = 0.7;

	if (tid<N_spots)
	{
		s_Vre[tid] = g_Vre[tid*N_pixels];
		s_Vim[tid] = g_Vim[tid*N_pixels];
		s_a[tid] = hypotf(s_Vim[tid], s_Vre[tid]);
		s_w[tid] = g_weights[tid + iteration*N_spots];
		a_d[tid] = amp_d[tid];
		s_w[tid] = s_w[tid] * powf((a_d[tid]/s_a[tid]), gamma);
	}
	__syncthreads();
	if (tid==0)
	{
		w_sum = 0;
		for (int kk=0; kk<N_spots; kk++)
		{
			w_sum += s_w[kk];
		}
	}
	__syncthreads();
	
	if (tid<N_spots)
	{
		s_w[tid] = s_w[tid]/w_sum;	
	}
	__syncthreads();
	

	for (int k=0; k<N_spots; k++)
	{
		re_SLM += s_w[k] *  (g_ei_dre[idx + k*N_pixels] * s_Vre[k] - g_ei_dim[idx + k*N_pixels] * s_Vim[k]) / s_a[k];// a_d[k] *
		im_SLM += s_w[k] * (g_ei_dre[idx + k*N_pixels] * s_Vim[k] - s_Vre[k] * g_ei_dim[idx + k*N_pixels]) / s_a[k];	
	}
	__syncthreads();	
	
	g_Phi[idx] = atan2f(im_SLM, re_SLM);
	
	if (tid<N_spots)
	{
		g_weights[tid + N_spots*(iteration+1)] = s_w[tid];
	}

}
*/

/*
 old version (twice as long iteration time)

__global__ void computePhi(float *g_Vre, float *g_Vim, float *g_Phi, float *g_ei_dre, float *g_ei_dim, int N_pixels, int N_spots, float *g_weights, int iteration)
{
	__syncthreads();	
	__shared__ float s_Vre[100], s_Vim[100], s_a[100], s_w[100];
	for (int k=0; k<N_spots; k++)
	{
		s_Vre[k] = g_Vre[k*N_pixels];
		s_Vim[k] = g_Vim[k*N_pixels];
	}
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx<N_pixels)
	{
		float re_SLM = 0;
		float im_SLM = 0;
		for (int kk=0; kk<N_spots; kk++)
		{
			re_SLM += (g_ei_dre[idx + kk*N_pixels] * s_Vre[kk] - g_ei_dim[idx + kk*N_pixels] * s_Vim[kk]) / hypotf(s_Vim[kk], s_Vre[kk]);
			im_SLM += (g_ei_dre[idx + kk*N_pixels] * s_Vim[kk] - s_Vre[kk] * g_ei_dim[idx + kk*N_pixels]) / hypotf(s_Vim[kk], s_Vre[kk]);		
		}
 		g_Phi[idx] = atan2f(im_SLM, re_SLM);

	}
	__syncthreads();
}
*/
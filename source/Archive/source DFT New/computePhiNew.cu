#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//Obtain phases in SLM plane
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA_PCI.h"

//works only for blocksize 512 and max 256 spots

__global__ void computePhiNew(float *g_x, float *g_y, float *g_z, float *g_SpotsRe, float *g_SpotsIm, float *g_Phi, int N_pixels, int N_spots, float *g_weights, int iteration, float *g_pSLM_start, float RPC, float *g_amps)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;	
	__shared__ float s_aSpot[256], s_aSpot_sum[256], s_a_mean, s_weight[256], s_pSpot[256];
	__shared__ float s_xm[256];
	__shared__ float s_ym[256];
	__shared__ float s_zm[256];
	float re_SLM = 0, im_SLM = 0, Phi = 0, p_SLM_s;

	if (idx<N_pixels)
	{
		//float N = 512;
		//int logN = (int)log2(N);
		float d = 0.001953125;		//(1/512)
		
		if (RPC < (2.0f*M_PI))
			p_SLM_s = g_pSLM_start[idx];

		if (N_spots <= 64)
		{
			if (tid < N_spots)
			{
				float Vre = g_SpotsRe[tid];
				float Vim = g_SpotsIm[tid];
				s_aSpot[tid] = hypotf(Vim, Vre);
				s_aSpot_sum[tid] = s_aSpot[tid];
				s_pSpot[tid] = atan2f(Vim, Vre);
			}	
			else if ((tid - 64) < N_spots)
				s_weight[tid - 64] = g_weights[tid - 64 + iteration*N_spots];
			else if ((tid - 128) < N_spots)
				s_xm[tid - 128] = g_x[tid - 128];
			else if ((tid - 192) < N_spots)
				s_ym[tid - 192] = g_y[tid - 192];
			else if ((tid - 256) < N_spots)
				s_zm[tid - 256] = g_z[tid - 256];																	
		}
		else
		{		
			if (tid<N_spots)
			{
				float Vre = g_SpotsRe[tid];
				float Vim = g_SpotsIm[tid];
				s_pSpot[tid] = atan2f(Vim, Vre);
				s_aSpot[tid] = hypotf(Vim, Vre);
				s_aSpot_sum[tid] = s_aSpot[tid];
				s_weight[tid] = g_weights[tid + iteration*N_spots];
				s_xm[tid] = g_x[tid];
				s_ym[tid] = g_y[tid];
				s_zm[tid] = g_z[tid];
			}
		}	
		__syncthreads();
				
		if (N_spots >= 256) { if (tid < 128) { s_aSpot_sum[tid] += s_aSpot_sum[tid + 128]; } __syncthreads(); }
		if (N_spots >= 128) { if (tid < 64) { s_aSpot_sum[tid] += s_aSpot_sum[tid + 64]; } __syncthreads(); }
		if (tid < 32) 
		{
			if (N_spots >= 64) s_aSpot_sum[tid] += s_aSpot_sum[tid + 32];
			if (N_spots >= 32) s_aSpot_sum[tid] += s_aSpot_sum[tid + 16];
			if (N_spots >= 16) s_aSpot_sum[tid] += s_aSpot_sum[tid + 8];
			if (N_spots >= 8) s_aSpot_sum[tid] += s_aSpot_sum[tid + 4];
			if (N_spots >= 4) s_aSpot_sum[tid] += s_aSpot_sum[tid + 2];
			if (N_spots >= 2) s_aSpot_sum[tid] += s_aSpot_sum[tid + 1];
		}
		
		if  (tid==0)
		{
			s_a_mean = s_aSpot_sum[0] / N_spots; //integer division!!
		}
		__syncthreads();
	
		if (tid<N_spots)
		{
			s_weight[tid] = s_weight[tid] * s_a_mean / s_aSpot[tid];	
			g_weights[tid + N_spots*(iteration+1)] = s_weight[tid];
			g_amps[tid + N_spots*iteration] = s_aSpot[tid];		//may be excluded, used for monitoring only
		}
		__syncthreads();		

 		float X = d * ((float)(threadIdx.x) - 256.0);
 		float Y = d * ((float)(blockIdx.x) - 256.0);
	
		for (int k=0; k<N_spots; k++)
		{
			float delta = M_PI * s_zm[k] * (X*X + Y*Y) + 2.0 * M_PI * (X * s_xm[k] + Y * s_ym[k]);

			re_SLM += s_weight[k] * cosf(s_pSpot[k] + delta);
			im_SLM += s_weight[k] * sinf(s_pSpot[k] + delta);
			
			
			//ei_dre = cosf(delta);  
			//ei_dim = sinf(delta);
			
			//re_SLM += s_weight[k] * (ei_dre * s_Vre[k] - ei_dim * s_Vim[k]) / s_aSpot[k];
			//im_SLM += s_weight[k] * (ei_dre * s_Vim[k] + s_Vre[k] * ei_dim) / s_aSpot[k];
		}

		Phi = atan2f(im_SLM, re_SLM);
		
		
		if (RPC < (2.0f*M_PI))
		{	
			if ((fabs(Phi - p_SLM_s)) < RPC)
			{
				g_Phi[idx] = Phi;
			}
			else
			{
				g_Phi[idx] = p_SLM_s;
			}	
		}
		else	
		{
			g_Phi[idx] = Phi;
		}
	}
}

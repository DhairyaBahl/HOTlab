#include "hip/hip_runtime.h"
__global__ void sum1(float *g_in, float *g_out, int offset)
{	
	extern __shared__ float s_data[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*512 + tid;

	s_data[tid] = 0;
	s_data[tid] += g_in[i + offset] + g_in[i+128 + offset] + g_in[i+256 + offset] + g_in[i+256 + 128 + offset]; 
	//s_data[tid] += g_in[i + offset] + g_in[i+256 + offset]; 
	__syncthreads();
	
	//if (tid < 128) 
	//{
	//	s_data[tid] += s_data[tid + 128];
	//	__syncthreads();
	//}
	if (tid < 64) 
	{
		s_data[tid] += s_data[tid + 64];
		__syncthreads();
	}
	if (tid < 32) 
	{
		s_data[tid] += s_data[tid + 32];
		s_data[tid] += s_data[tid + 16];
		s_data[tid] += s_data[tid + 8];
		s_data[tid] += s_data[tid + 4];
		s_data[tid] += s_data[tid + 2];
		s_data[tid] += s_data[tid + 1];
	}
	if (tid == 0) g_out[blockIdx.x + offset] = s_data[0];
}
__global__ void sum2(float *g_in, float *g_out, int offset)
{
	extern __shared__ float s_data[];
	unsigned int tid = threadIdx.x;

	s_data[tid] = 0;

	s_data[tid] += g_in[tid + offset] + g_in[tid + 256 + offset]; 
	__syncthreads();
	if (tid < 128) 
	{
		s_data[tid] += s_data[tid + 128];
	__syncthreads();
	}
	if (tid < 64) 
	{
		s_data[tid] += s_data[tid + 64];
		__syncthreads();
	}
			if (tid < 32) 
	{
		s_data[tid] += s_data[tid + 32];
		s_data[tid] += s_data[tid + 16];
		s_data[tid] += s_data[tid + 8];
		s_data[tid] += s_data[tid + 4];
		s_data[tid] += s_data[tid + 2];
		s_data[tid] += s_data[tid + 1];
	}
	if (tid == 0) g_out[offset] = s_data[0];
}
void sumV(float* d_in, float* d_out, int offset)
{
	sum1<<<512, 128, 128*sizeof(float)>>>(d_in, d_out, offset);
	hipDeviceSynchronize();
	sum2<<<1, 256, 256*sizeof(float)>>>(d_out, d_out, offset);
	hipDeviceSynchronize();
	return;
}
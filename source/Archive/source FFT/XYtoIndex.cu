#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//Convert from coordinates to spot indices
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA_PCI.h"

__global__ void XYtoIndex(float *g_x, float *g_y, int *g_spot_index, int N_spots, int data_w)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i < N_spots)
		g_spot_index[i] = ((int)(g_x[i])&(data_w-1))  + ((int)(g_y[i])&(data_w-1))* data_w;
	
	__syncthreads();
}
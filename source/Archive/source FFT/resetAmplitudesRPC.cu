#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//reset amplitudes to ones PCR
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA_PCI.h"

__global__ void resetAmplitudesRPC(float *g_aLaser, hipfftComplex *g_cAmp, float *g_pSLM_start, int N_pixels, float RPC)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	

	if (idx<N_pixels)
	{
		float aLaser = g_aLaser[idx];
		float pSLM_start = g_pSLM_start[idx];
		hipfftComplex cAmp = g_cAmp[idx];
 		float phase = atan2f(cAmp.y, cAmp.x);
 		
 		if (RPC < (2.0f*M_PI))
		{	
			if (fabs(phase - pSLM_start) < RPC)
			{
				cAmp.x = aLaser*cosf(phase);
				cAmp.y = aLaser*sinf(phase);
			}
			else
			{
				cAmp.x = aLaser*cosf(pSLM_start);
				cAmp.y = aLaser*sinf(pSLM_start);
			}
		}
		else
		{
			cAmp.x = aLaser*cosf(phase);
			cAmp.y = aLaser*sinf(phase);
		}	
		
		g_cAmp[idx].x = cAmp.x;
		g_cAmp[idx].y = cAmp.y;
	}
	__syncthreads();
}
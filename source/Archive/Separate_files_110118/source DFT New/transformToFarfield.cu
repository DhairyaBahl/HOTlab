#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "GenerateHologramCUDA.h"

//Works for 512x512 pixels only!
__global__ void transformToFarfield(float *g_x, float *g_y, float *g_z, float *g_pSLM, float *g_Vre, float *g_Vim, int N_spots, unsigned int n, int data_w)
{
	int blockSize = 512;
	int spot_number = blockIdx.x;
	unsigned int tid = threadIdx.x;
	unsigned int i = tid;
	
	__shared__ float s_Vre[512];
	__shared__ float s_Vim[512];
	__shared__ float s_xm, s_ym, s_zm;

	s_Vre[tid] = 0;
	s_Vim[tid] = 0;
		
	float N = data_w;
	int logN = (int)log2(N);
	float d = 0.001953125;	//SLM pixel size (1/512)
	
	if (tid == 0)
		s_xm = g_x[spot_number];
	if (tid == 64)
		s_ym = g_y[spot_number];	
	if (tid == 128)
		s_zm = g_z[spot_number];
	
	float X1 = d * ((float)tid - 256.0);
	float Y1 = - d * 256.0;	
	__syncthreads();
		
	while (i < n) 
	{ 
		float pSLM_1 = g_pSLM[i];
		float p = pSLM_1 - M_PI * (s_zm * (X1*X1 + Y1*Y1) + 2 * (X1 * s_xm + Y1 * s_ym));
		
		s_Vre[tid] += cosf(p);
		s_Vim[tid] += sinf(p);

		i += blockSize;
		Y1 += d; 	
	}
	__syncthreads();
 
	if (tid < 256) 
	{ 
		s_Vre[tid] += s_Vre[tid + 256]; 
		s_Vim[tid] += s_Vim[tid + 256];
	} 
	__syncthreads(); 

	if (tid < 128)
	{ 
		s_Vre[tid] += s_Vre[tid + 128];
		s_Vim[tid] += s_Vim[tid + 128];  
	} 
	__syncthreads(); 

	if (tid < 64) 
	{ 
		s_Vre[tid] += s_Vre[tid + 64];
		s_Vim[tid] += s_Vim[tid + 64]; 
	} 
	__syncthreads(); 

	
	if (tid < 32) {

		s_Vre[tid] += s_Vre[tid + 32];
		s_Vim[tid] += s_Vim[tid + 32];

		s_Vre[tid] += s_Vre[tid + 16];
		s_Vim[tid] += s_Vim[tid + 16];

		s_Vre[tid] += s_Vre[tid + 8];
		s_Vim[tid] += s_Vim[tid + 8];

		s_Vre[tid] += s_Vre[tid + 4];
		s_Vim[tid] += s_Vim[tid + 4];

		s_Vre[tid] += s_Vre[tid + 2];
		s_Vim[tid] += s_Vim[tid + 2];

		s_Vre[tid] += s_Vre[tid + 1];
		s_Vim[tid] += s_Vim[tid + 1];
		
	}
	if (tid == 0) 
	{
		g_Vre[spot_number] = s_Vre[0] / 262144.0;
		g_Vim[spot_number] = s_Vim[0] / 262144.0;
	}
}
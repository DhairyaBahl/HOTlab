#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se
   
   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "GenerateHologramCUDA.h"
//This software contains source code provided by NVIDIA Corporation.

void getNumBlocksAndThreads(int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
        if (n == 1) 
            threads = 1;
        else
            threads = (n < maxThreads*2) ? n / 2 : maxThreads;
        blocks = n / (threads * 2);
        blocks = min(maxBlocks, blocks);
}



void Reduce(int  n, int maxThreads, int maxBlocks, float* d_idata, float* d_odata, int offset)
{
    hipDeviceSynchronize();
    
    int numBlocks = 0;
    int numThreads = 0;
    getNumBlocksAndThreads(n, maxBlocks, maxThreads, numBlocks, numThreads);

    // execute the kernel
    reduce(n, numThreads, numBlocks, d_idata, d_odata, offset);

    // sum partial block sums on GPU
    int s=numBlocks;
    while(s > 1) 
    {
        int threads = 0, blocks = 0;
        getNumBlocksAndThreads(s, maxBlocks, maxThreads, blocks, threads);

        reduce(s, threads, blocks, d_odata, d_odata, offset);
        s = s / (threads*2);    
    }
    hipDeviceSynchronize();
  
    return;
}

template <unsigned int blockSize>
__global__ void reduce6(float *g_idata, float *g_odata, unsigned int n, int offset)
{
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*2*gridDim.x;
	sdata[tid] = 0;
	
	while (i < n) { sdata[tid] += g_idata[i + offset] + g_idata[i+blockSize + offset]; i += gridSize; }
	__syncthreads();
	
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	
	if (tid < 32) {
		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
	if (tid == 0) g_odata[blockIdx.x + offset] = sdata[0];
}


////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////

void reduce(int size, int threads, int blocks, float *d_idata, float *d_odata, int offset)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);
    int smemSize = threads * sizeof(float);

        switch (threads)
        {
        case 512:
            reduce6<512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, offset); break;
        case 256:
            reduce6<256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, offset); break;
        case 128:
            reduce6<128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, offset); break;
        case 64:
            reduce6<64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, offset); break;
        case 32:
            reduce6<32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, offset); break;
        case 16:
            reduce6<16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, offset); break;
        case  8:
            reduce6<8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, offset); break;
        case  4:
            reduce6<4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, offset); break;
        case  2:
            reduce6<2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, offset); break;
        case  1:
            reduce6<1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, offset); break;
        }

}

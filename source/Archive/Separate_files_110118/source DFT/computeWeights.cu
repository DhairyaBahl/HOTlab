#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
////////////////////////////////////////////////////////////////////////////////
//Compute weights for each spot based on desired and obtaied amplitude
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA.h"

__global__ void computeWeights(float *g_Vre, 
							float *g_Vim, 
							int N_spots, 
							float *g_weights, 
							int iteration, 
							float *g_amps_o, 
							float *g_I, 
							int N_pixels)
{

	int tid = threadIdx.x;	
	__shared__ float s_aSpot_o[block_size], s_aSpot_mean;
	float Vre, Vim, wSpot;

	if (tid<N_spots)
	{										//the desited amplitude for each spot
		Vre = g_Vre[tid*N_pixels];
		Vim = g_Vim[tid*N_pixels];
		s_aSpot_o[tid] = hypotf(Vim, Vre)/sqrtf(g_I[tid]);	//the obtained amplitude for each spot divided by the desired amplitude	
		wSpot = g_weights[tid + iteration*N_spots];			//the previous weight for each spot
	}

	__syncthreads();
	if  (tid==0)
	{
		float aSpot_sum = 0;
		for (int kk=0; kk<N_spots; kk++)
		{
			aSpot_sum += s_aSpot_o[kk];
		}
		s_aSpot_mean = aSpot_sum / N_spots; //integer division!!
	}
	__syncthreads();
	
	if (tid<N_spots)
	{
		wSpot = wSpot * s_aSpot_mean / s_aSpot_o[tid];	
		g_weights[tid + N_spots*(iteration+1)] = wSpot;
		g_amps_o[tid + N_spots*iteration] = s_aSpot_o[tid];		
	}
}

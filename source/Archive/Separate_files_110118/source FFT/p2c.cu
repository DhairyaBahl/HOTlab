#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//Convert polar to complex
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA.h"

__global__ void p2c(hipfftComplex *s_c, float *s_p, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
 		s_c[idx].x = cosf(s_p[idx]);
		s_c[idx].y = sinf(s_p[idx]);
	}
	__syncthreads();
}
#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//Calculate Phases and return as float
//
//Possible improvements
//-compute powers of X, Y and phase2pi only once and put in registers
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA.h"

__global__ void getPhases(unsigned char *g_pSLMuc, float *g_pSLM_start, hipfftComplex *g_cSLMcc, unsigned char *g_LUT, int use_LUTfile, int data_w)
{	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int s_idx;
	int X_shifted, Y_shifted;
	int N_pixels = data_w * data_w;
	__shared__ unsigned char s_LUT[256];
	if (use_LUTfile == 1)
	{
		int tid = threadIdx.x;
		if (tid < 256)
		{
			int j = 0;
			while (j < 256)
			{
				s_LUT[tid + j] = g_LUT[tid + j];
				j += block_size;
			}
		}
	}	
	if (idx<N_pixels)
	{
		float phase255;
		float N = data_w;
		int half_w = data_w/2;
		int logN = (int)log2(N);

		int X = idx&(int)(N-1);
 		int Y = (idx-X)>>logN;
 		
 		if (X < half_w)
		{	
			X_shifted = X + half_w;
			if (Y < half_w)
			{
				Y_shifted = Y + half_w;
				s_idx = idx + (data_w * half_w) + half_w;
			}
			else
			{
				Y_shifted = Y - half_w;
				s_idx = idx - (data_w * half_w) + half_w;
			}
		}
		else
		{
			X_shifted = X - half_w;
			if (Y < half_w)
			{
				Y_shifted = Y + half_w;			
				s_idx = idx + (data_w * half_w) - half_w;
			}
			else
			{
				Y_shifted = Y - half_w;			
				s_idx = idx - (data_w * half_w) - half_w;
			}
		}
 		float phase = atan2f(g_cSLMcc[idx].y, g_cSLMcc[idx].x);
		float phase2pi = M_PI + phase;	
		
		if (use_LUTfile == 1) 
		{
			__syncthreads();
			phase255 = 255.0 * phase2pi / (2.0 * M_PI);
			g_pSLM_start[idx] = phase;
			g_pSLMuc[s_idx] = s_LUT[(unsigned char)phase255];
		}
		else
		{
			phase255 = 255.0 * phase2pi / (2.0 * M_PI);
			g_pSLM_start[idx] = phase;
			g_pSLMuc[s_idx] = (unsigned char)phase255;
		}	
	}
}

#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//Convert unsigned char to float
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA.h"

__global__ void uc2f(float *g_f, unsigned char *g_uc, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<N)
	{
 		float in = g_uc[idx];
 		g_f[idx] = (float)in*(2.0f*M_PI)/255.0f - M_PI;
	}
	__syncthreads();
}
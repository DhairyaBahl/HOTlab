#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU Lesser General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Lesser General Public License for more details.

    You should have received a copy of the GNU Lesser General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "GenerateHologramCUDA.h"

__global__ void LensesAndPrisms(float *g_x, float *g_y, float *g_z, float *g_I, unsigned char *g_SLMuc, int N_spots, unsigned char *g_LUT, bool ApplyLUTFile_b, int data_w, bool UseAberrationCorr_b, float *d_AberrationCorr_f, bool UseLUTPol_b, float *d_LUTPolCoeff_f, int N_PolCoeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	
	__shared__ float s_x[BLOCK_SIZE];
	__shared__ float s_y[BLOCK_SIZE];
	__shared__ float s_z[BLOCK_SIZE];
	__shared__ float s_a[BLOCK_SIZE];
	__shared__ unsigned char s_LUT[256];	
	if (tid < N_spots)
	{
		s_x[tid] = g_x[tid];
		s_y[tid] = g_y[tid];
		s_z[tid] = g_z[tid];
		s_a[tid] = sqrtf(g_I[tid]);
	}
	__syncthreads();
	if (ApplyLUTFile_b)
	{
		if (tid < 256)
		{
			int j = 0;
			while (j < 256)
			{
				s_LUT[tid + j] = g_LUT[tid + j];
				j += BLOCK_SIZE;
			}
		}
	}		
	
	if (idx < data_w*data_w)
	{
		float N = data_w;
		int logN = (int)log2(N);
		int X_int = idx&(int)(N-1);
	 	//int Y_int = (idx&(int)(N*N-1)-X_int)>>logN;
	 	int Y_int = (idx-X_int)>>logN;
	 	float X = (float)(X_int - (data_w>>1)) / data_w;
 		float Y = (float)(Y_int - (data_w>>1)) / data_w;
		
		float phase2pi;
		float phase255;
		float SLMre = 0;
		float SLMim = 0;
				
		for (int ii=0; ii<N_spots; ++ii)
		{
			phase2pi = M_PI * s_z[ii] * (X*X + Y*Y) + 2.0 * M_PI * (X * (s_x[ii]) + Y * (s_y[ii]) );
			SLMre = SLMre + s_a[ii] * cosf(phase2pi);
			SLMim = SLMim + s_a[ii] * sinf(phase2pi); 
		}
		
		phase2pi = atan2f(SLMim, SLMre) + M_PI;	
		if (UseAberrationCorr_b)
		{
			float AberrationCorr_f = d_AberrationCorr_f[idx];
			phase2pi = phase2pi + AberrationCorr_f;
			//modulus!!!
		}

		if (ApplyLUTFile_b) 
		{			
			__syncthreads();
			phase255 = 255.0 * phase2pi / (2.0 * M_PI);
			g_SLMuc[idx] = s_LUT[(unsigned char)phase255];
		}
		else
		{
			phase255 = 255.0 * phase2pi / (2.0 * M_PI);
			g_SLMuc[idx] = (unsigned char)phase255;
		}
	}	
	__syncthreads();

}

__global__ void checkAmplitudes(float *g_x, float *g_y, float *g_z, unsigned char *g_pSLM_uc, float *g_amps, int N_spots, unsigned int N_pixels, int data_w)
{
	int blockSize = 512;
	int spot_number = blockIdx.x;
	unsigned int tid = threadIdx.x;
	unsigned int i = tid;
	
	__shared__ float s_Vre[512];
	__shared__ float s_Vim[512];
	__shared__ float s_xm, s_ym, s_zm;

	s_Vre[tid] = 0;
	s_Vim[tid] = 0;
		
	float N = data_w;
	int logN = (int)log2(N);
	float d = 0.001953125;	//SLM pixel size (1/512)
	
	if (tid == 0)
		s_xm = g_x[spot_number];
	if (tid == 64)
		s_ym = g_y[spot_number];	
	if (tid == 128)
		s_zm = g_z[spot_number];
	
	float X1 = d * ((float)tid - 256.0);
	float Y1 = - d * 256.0;	
	__syncthreads();
		
	while (i < N_pixels) 
	{ 
		float pSLM_1 = 2*M_PI*(float)g_pSLM_uc[i] - M_PI;
		float p = pSLM_1 - M_PI * (s_zm * (X1*X1 + Y1*Y1) + 2 * (X1 * s_xm + Y1 * s_ym));
		
		s_Vre[tid] += cosf(p);
		s_Vim[tid] += sinf(p);

		i += blockSize;
		Y1 += d; 	
	}
	__syncthreads();
 
	if (tid < 256) 
	{ 
		s_Vre[tid] += s_Vre[tid + 256]; 
		s_Vim[tid] += s_Vim[tid + 256];
	} 
	__syncthreads(); 

	if (tid < 128)
	{ 
		s_Vre[tid] += s_Vre[tid + 128];
		s_Vim[tid] += s_Vim[tid + 128];  
	} 
	__syncthreads(); 

	if (tid < 64) 
	{ 
		s_Vre[tid] += s_Vre[tid + 64];
		s_Vim[tid] += s_Vim[tid + 64]; 
	} 
	__syncthreads(); 

	
	if (tid < 32) {

		s_Vre[tid] += s_Vre[tid + 32];
		s_Vim[tid] += s_Vim[tid + 32];

		s_Vre[tid] += s_Vre[tid + 16];
		s_Vim[tid] += s_Vim[tid + 16];

		s_Vre[tid] += s_Vre[tid + 8];
		s_Vim[tid] += s_Vim[tid + 8];

		s_Vre[tid] += s_Vre[tid + 4];
		s_Vim[tid] += s_Vim[tid + 4];

		s_Vre[tid] += s_Vre[tid + 2];
		s_Vim[tid] += s_Vim[tid + 2];

		s_Vre[tid] += s_Vre[tid + 1];
		s_Vim[tid] += s_Vim[tid + 1];
		
	}
	if (tid == 0) 
	{
		float Vre = s_Vre[0] / 262144.0;
		float Vim = s_Vim[0] / 262144.0;
		g_amps[spot_number] = hypotf(Vim, Vre);
	}
}

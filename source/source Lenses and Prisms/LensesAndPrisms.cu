#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Foobar.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "GenerateHologramCUDA.h"

__global__ void LensesAndPrisms(float *g_x, float *g_y, float *g_z, float *g_I, unsigned char *g_SLMuc, int N_spots, unsigned char *g_LUT, int use_LUTfile, int data_w)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	
	__shared__ float s_x[block_size];
	__shared__ float s_y[block_size];
	__shared__ float s_z[block_size];
	__shared__ float s_a[block_size];
	__shared__ unsigned char s_LUT[256];	
	if (tid < N_spots)
	{
		s_x[tid] = g_x[tid];
		s_y[tid] = g_y[tid];
		s_z[tid] = g_z[tid];
		s_a[tid] = sqrtf(g_I[tid]);
	}
	__syncthreads();
	if (use_LUTfile == 1)
	{
		if (tid < 256)
		{
			int j = 0;
			while (j < 256)
			{
				s_LUT[tid + j] = g_LUT[tid + j];
				j += block_size;
			}
		}
	}		
	
	if (idx < data_w*data_w)
	{
		float N = data_w;
		int logN = (int)log2(N);
		int X_int = idx&(int)(N-1);
	 	//int Y_int = (idx&(int)(N*N-1)-X_int)>>logN;
	 	int Y_int = (idx-X_int)>>logN;
	 	float X = (float)(X_int - (data_w>>1)) / data_w;
 		float Y = (float)(Y_int - (data_w>>1)) / data_w;
		
		float phase2pi;
		float phase255;
		float SLMre = 0;
		float SLMim = 0;
				
		for (int ii=0; ii<N_spots; ++ii)
		{
			phase2pi = M_PI * s_z[ii] * (X*X + Y*Y) + 2.0 * M_PI * (X * (s_x[ii]) + Y * (s_y[ii]) );
			SLMre = SLMre + s_a[ii] * cosf(phase2pi);
			SLMim = SLMim + s_a[ii] * sinf(phase2pi); 
		}
		
		phase2pi = atan2f(SLMim, SLMre) + M_PI;	
		

		if (use_LUTfile == 1) 
		{			
			__syncthreads();
			phase255 = 255.0 * phase2pi / (2.0 * M_PI);
			g_SLMuc[idx] = s_LUT[(unsigned char)phase255];
		}
		else
		{
			phase255 = 255.0 * phase2pi / (2.0 * M_PI);
			g_SLMuc[idx] = (unsigned char)phase255;
		}
	}	
	__syncthreads();

}

			
				
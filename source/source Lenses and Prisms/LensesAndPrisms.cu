#include "hip/hip_runtime.h"
#include "GenerateHologramCUDA.h"

__global__ void LensesAndPrisms(float *g_x, float *g_y, float *g_z, float *g_I, unsigned char *g_SLMuc, int N_spots, unsigned char *g_LUT, int use_LUTfile, int data_w)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	
	__shared__ float s_x[block_size];
	__shared__ float s_y[block_size];
	__shared__ float s_z[block_size];
	__shared__ float s_a[block_size];
	__shared__ unsigned char s_LUT[256];	
	if (tid < N_spots)
	{
		s_x[tid] = g_x[tid];
		s_y[tid] = g_y[tid];
		s_z[tid] = g_z[tid];
		s_a[tid] = sqrtf(g_I[tid]);
	}
	__syncthreads();
	if (use_LUTfile == 1)
	{
		if (tid < 256)
		{
			int j = 0;
			while (j < 256)
			{
				s_LUT[tid + j] = g_LUT[tid + j];
				j += block_size;
			}
		}
	}		
	
	if (idx < data_w*data_w)
	{
		float N = data_w;
		int logN = (int)log2(N);
		int X_int = idx&(int)(N-1);
	 	//int Y_int = (idx&(int)(N*N-1)-X_int)>>logN;
	 	int Y_int = (idx-X_int)>>logN;
	 	float X = (float)(X_int - (data_w>>1)) / data_w;
 		float Y = (float)(Y_int - (data_w>>1)) / data_w;
		
		float phase2pi;
		float phase255;
		float SLMre = 0;
		float SLMim = 0;
				
		for (int ii=0; ii<N_spots; ++ii)
		{
			phase2pi = M_PI * s_z[ii] * (X*X + Y*Y) + 2.0 * M_PI * (X * (s_x[ii]) + Y * (s_y[ii]) );
			SLMre = SLMre + s_a[ii] * cosf(phase2pi);
			SLMim = SLMim + s_a[ii] * sinf(phase2pi); 
		}
		
		phase2pi = atan2f(SLMim, SLMre) + M_PI;	
		

		if (use_LUTfile == 1) 
		{			
			__syncthreads();
			phase255 = 255.0 * phase2pi / (2.0 * M_PI);
			g_SLMuc[idx] = s_LUT[(unsigned char)phase255];
		}
		else
		{
			phase255 = 255.0 * phase2pi / (2.0 * M_PI);
			g_SLMuc[idx] = (unsigned char)phase255;
		}
	}	
	__syncthreads();

}

			
				
#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU Lesser General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Lesser General Public License for more details.

    You should have received a copy of the GNU Lesser General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
////////////////////////////////////////////////////////////////////////////////
//Convert float to unsigned char
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA.h"

__global__ void f2uc(unsigned char *g_uc, float *g_f, int N_pixels, unsigned char *g_LUT, int use_LUTfile, int data_w)
{
	__shared__ unsigned char s_LUT[256];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (use_LUTfile == 1)
	{
		int tid = threadIdx.x;
		if (tid < 256)
		{
			int j = 0;
			while (j < 256)
			{
				s_LUT[tid + j] = g_LUT[tid + j];
				j += block_size;
			}
		}
		__syncthreads();
		if (idx<N_pixels)
		{
			g_uc[idx] = s_LUT[(unsigned char)(((g_f[idx] + M_PI)*255.0/(2.0*M_PI)))];

		}	
	}	
	else
	{
		if (idx<N_pixels)
		{
			g_uc[idx] = (unsigned char)(((g_f[idx] + M_PI)*255.0/(2.0*M_PI)));
		}	
	}
	__syncthreads();
}


#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU Lesser General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Lesser General Public License for more details.

    You should have received a copy of the GNU Lesser General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
////////////////////////////////////////////////////////////////////////////////
//Convert float to unsigned char
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA.h"

__global__ void f2uc(unsigned char *g_uc, float *g_f, int N_pixels, unsigned char *g_LUT, int ApplyLUT_b, int data_w)
{
	__shared__ unsigned char s_LUT[256];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (ApplyLUT_b)
	{
		int tid = threadIdx.x;
		if (tid < 256)
		{
			int j = 0;
			while (j < 256)
			{
				s_LUT[tid + j] = g_LUT[tid + j];
				j += BLOCK_SIZE;
			}
		}
		__syncthreads();
		if (idx<N_pixels)
		{
			g_uc[idx] = s_LUT[(unsigned char)(((g_f[idx] + M_PI)*255.0/(2.0*M_PI)))];
		}	
	}	
	else
	{
		if (idx<N_pixels)
		{
			g_uc[idx] = (unsigned char)(((g_f[idx] + M_PI)*255.0/(2.0*M_PI)));
		}	
	}
	__syncthreads();
}
/*
__global__ void f2uc(unsigned char *g_uc, float *g_f, int N_pixels, unsigned char *g_LUT, int use_LUT, float *g_LUT_coeff, int LUT_order, int data_w, float *wfcorrection, int use_correction)
{
	__shared__ unsigned char s_LUT[256];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float phase2pi = g_f[idx] + M_PI;
	if (use_correction == 1)
		phase2pi = phase2pi + wfcorrection[idx];
		
	if (use_LUT == 0)	
	{
		if (idx<N_pixels)
		{
			g_uc[idx] = (unsigned char)(((phase2pi)*255.0/(2.0*M_PI)));
		}	
	}
	else if (use_LUT == 1)
	{
		int tid = threadIdx.x;
		if (tid < 256)
		{
			int j = 0;
			while (j < 256)
			{
				s_LUT[tid + j] = g_LUT[tid + j];
				j += block_size;
			}
		}
		__syncthreads();
		if (idx<N_pixels)
		{
			g_uc[idx] = s_LUT[(unsigned char)(((phase2pi)*255.0/(2.0*M_PI)))];
		}	
	}
	else if (use_LUT == 2)
	{
		float N = (float)data_w;
		float X = idx%data_w;
 		float Y = floor(idx/N);

		__syncthreads();
		float phase255;
		if (LUT_order==7)
		{
			__shared__ float s_c[120];
			if (threadIdx.x < 120)
				s_c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
			__syncthreads();
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5) + s_c[56]*powf(X,6) + s_c[57]*powf(X,5)*Y + s_c[58]*powf(X,5)*phase2pi + s_c[59]*powf(X,4)*powf(Y,2) + s_c[60]*powf(X,4)*Y*phase2pi + s_c[61]*powf(X,4)*powf(phase2pi,2) + s_c[62]*powf(X,3)*powf(Y,3) + s_c[63]*powf(X,3)*powf(Y,2)*phase2pi + s_c[64]*powf(X,3)*Y*powf(phase2pi,2) + s_c[65]*powf(X,3)*powf(phase2pi,3) + s_c[66]*powf(X,2)*powf(Y,4) + s_c[67]*powf(X,2)*powf(Y,3)*phase2pi + s_c[68]*powf(X,2)*powf(Y,2)*powf(phase2pi,2) + s_c[69]*powf(X,2)*Y*powf(phase2pi,3) + s_c[70]*powf(X,2)*powf(phase2pi,4) + s_c[71]*X*powf(Y,5) + s_c[72]*X*powf(Y,4)*phase2pi + s_c[73]*X*powf(Y,3)*powf(phase2pi,2) + s_c[74]*X*powf(Y,2)*powf(phase2pi,3) + s_c[75]*X*Y*powf(phase2pi,4) + s_c[76]*X*powf(phase2pi,5) + s_c[77]*powf(Y,6) + s_c[78]*powf(Y,5)*phase2pi + s_c[79]*powf(Y,4)*powf(phase2pi,2) + s_c[80]*powf(Y,3)*powf(phase2pi,3) + s_c[81]*powf(Y,2)*powf(phase2pi,4) + s_c[82]*Y*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6) + s_c[84]*powf(X,7) + s_c[85]*powf(X,6)*Y + s_c[86]*powf(X,6)*phase2pi + s_c[87]*powf(X,5)*powf(Y,2) + s_c[88]*powf(X,5)*Y*phase2pi + s_c[89]*powf(X,5)*powf(phase2pi,2) + s_c[90]*powf(X,4)*powf(Y,3) + s_c[91]*powf(X,4)*powf(Y,2)*phase2pi + s_c[92]*powf(X,4)*Y*powf(phase2pi,2) + s_c[93]*powf(X,4)*powf(phase2pi,3) + s_c[94]*powf(X,3)*powf(Y,4) + s_c[95]*powf(X,3)*powf(Y,3)*phase2pi + s_c[96]*powf(X,3)*powf(Y,2)*powf(phase2pi,2) + s_c[97]*powf(X,3)*Y*powf(phase2pi,3) + s_c[98]*powf(X,3)*powf(phase2pi,4) + s_c[99]*powf(X,2)*powf(Y,5) + s_c[100]*powf(X,2)*powf(Y,4)*phase2pi + s_c[101]*powf(X,2)*powf(Y,3)*powf(phase2pi,2) + s_c[102]*powf(X,2)*powf(Y,2)*powf(phase2pi,3) + s_c[103]*powf(X,2)*Y*powf(phase2pi,4) + s_c[104]*powf(X,2)*powf(phase2pi,5) + s_c[105]*X*powf(Y,6) + s_c[106]*X*powf(Y,5)*phase2pi + s_c[107]*X*powf(Y,4)*powf(phase2pi,2) + s_c[108]*X*powf(Y,3)*powf(phase2pi,3) + s_c[109]*X*powf(Y,2)*powf(phase2pi,4) + s_c[110]*X*Y*powf(phase2pi,5) + s_c[111]*X*powf(phase2pi,6) + s_c[112]*powf(Y,7) + s_c[113]*powf(Y,6)*phase2pi + s_c[114]*powf(Y,5)*powf(phase2pi,2) + s_c[115]*powf(Y,4)*powf(phase2pi,3) + s_c[116]*powf(Y,3)*powf(phase2pi,4) + s_c[117]*powf(Y,2)*powf(phase2pi,5) + s_c[118]*Y*powf(phase2pi,6) + s_c[119]*powf(phase2pi,7));
		}
		else if (LUT_order==6)
		{
			__shared__ float s_c[84];
			if (threadIdx.x < 84)
				s_c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
			__syncthreads();
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5) + s_c[56]*powf(X,6) + s_c[57]*powf(X,5)*Y + s_c[58]*powf(X,5)*phase2pi + s_c[59]*powf(X,4)*powf(Y,2) + s_c[60]*powf(X,4)*Y*phase2pi + s_c[61]*powf(X,4)*powf(phase2pi,2) + s_c[62]*powf(X,3)*powf(Y,3) + s_c[63]*powf(X,3)*powf(Y,2)*phase2pi + s_c[64]*powf(X,3)*Y*powf(phase2pi,2) + s_c[65]*powf(X,3)*powf(phase2pi,3) + s_c[66]*powf(X,2)*powf(Y,4) + s_c[67]*powf(X,2)*powf(Y,3)*phase2pi + s_c[68]*powf(X,2)*powf(Y,2)*powf(phase2pi,2) + s_c[69]*powf(X,2)*Y*powf(phase2pi,3) + s_c[70]*powf(X,2)*powf(phase2pi,4) + s_c[71]*X*powf(Y,5) + s_c[72]*X*powf(Y,4)*phase2pi + s_c[73]*X*powf(Y,3)*powf(phase2pi,2) + s_c[74]*X*powf(Y,2)*powf(phase2pi,3) + s_c[75]*X*Y*powf(phase2pi,4) + s_c[76]*X*powf(phase2pi,5) + s_c[77]*powf(Y,6) + s_c[78]*powf(Y,5)*phase2pi + s_c[79]*powf(Y,4)*powf(phase2pi,2) + s_c[80]*powf(Y,3)*powf(phase2pi,3) + s_c[81]*powf(Y,2)*powf(phase2pi,4) + s_c[82]*Y*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6));
		}
		else if (LUT_order==5)
		{
			__shared__ float s_c[56];
			if (threadIdx.x < 56)
				s_c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
			__syncthreads();
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5));
		}
		else if (LUT_order==4)
		{
			__shared__ float s_c[36];
			if (threadIdx.x < 36)
				s_c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
			__syncthreads();
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4));
		}
		else if (LUT_order==3)
		{
			__shared__ float s_c[20];
			if (threadIdx.x < 20)
				s_c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
			__syncthreads();
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3));
		}
		if (phase255 < 0)
			phase255 = 0;	
		g_uc[idx] = (unsigned char)phase255;
	}
	__syncthreads();
}*/
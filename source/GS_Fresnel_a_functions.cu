#include "hip/hip_runtime.h"
/*
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "GenerateHologramCUDA.h"

//////////////////////////////////////////////////////////////////////////
//Compute the phase difference between each pixel in the SLM and each spot
//////////////////////////////////////////////////////////////////////////
__global__ void computeDelta(float *g_x, 
								float *g_y, 
								float *g_z, 
								float *g_delta, 
								float *g_ei_delta_re, 
								float *g_ei_delta_im, 
								int N_spots, 
								int N_pixels, 
								int data_w)
{
	float N = data_w;
	int logN = (int)log2(N);
	float d = 1.0/N;
	
	__shared__ float s_x[block_size];
	__shared__ float s_y[block_size];
	__shared__ float s_z[block_size];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadIdx.x < N_spots)
	{
		s_x[threadIdx.x] = g_x[threadIdx.x];
		s_y[threadIdx.x] = g_y[threadIdx.x];
		s_z[threadIdx.x] = g_z[threadIdx.x];
	}
	__syncthreads();
	int X_int = idx&(int)(N-1);
 	int Y_int = (idx&(int)(N_pixels-1)-X_int)>>logN;		//(idx-X)>>logN gives error for z!=0
 	float X = d * (float)(X_int - 256);
 	float Y = d * (float)(Y_int - 256);
 	
 	for (int m=0; m<N_spots; m++)
	{	
		if (idx<((m+1)*N_pixels) && idx >= m*N_pixels )
		{
			float delta = M_PI * s_z[m] * (X*X + Y*Y) + 2.0 * M_PI * (X * s_x[m] + Y * s_y[m]);
			g_delta[idx] = delta;
			g_ei_delta_re[idx] = cosf(delta);  
			g_ei_delta_im[idx] = sinf(delta); 
		}
	}
	__syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
//Obtain complex field in image plane using Fresnel propagation
////////////////////////////////////////////////////////////////////////////////
__global__ void computeV(float *g_Vre, 
							float *g_Vim, 
							float *g_pSLM, 
							float *g_delta, 
							int N, 
							int N_spots)
{
	float p, pSLM;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx<N)
	{
		pSLM = g_pSLM[idx];
		for (int ii=0; ii<N_spots; ++ii)
		{
			int id = idx + ii * N;
			p = pSLM - g_delta[id];
 			g_Vre[id] = cosf(p);
			g_Vim[id] = sinf(p);
		}
	}
}

////////////////////////////////////////////////////////////////////////////////
//Compute weights for each spot based on desired and obtaied amplitude
////////////////////////////////////////////////////////////////////////////////
__global__ void computeWeights(float *g_Vre, 
							float *g_Vim, 
							int N_spots, 
							float *g_weights, 
							int iteration, 
							float *g_amps_o, 
							float *g_I, 
							int N_pixels)
{

	int tid = threadIdx.x;	
	__shared__ float s_aSpot_o[block_size], s_aSpot_mean;
	float Vre, Vim, wSpot;

	if (tid<N_spots)
	{										//the desited amplitude for each spot
		Vre = g_Vre[tid*N_pixels];
		Vim = g_Vim[tid*N_pixels];
		s_aSpot_o[tid] = hypotf(Vim, Vre)/sqrtf(g_I[tid]);	//the obtained amplitude for each spot divided by the desired amplitude	
		wSpot = g_weights[tid + iteration*N_spots];			//the previous weight for each spot
	}

	__syncthreads();
	if  (tid==0)
	{
		float aSpot_sum = 0;
		for (int kk=0; kk<N_spots; kk++)
		{
			aSpot_sum += s_aSpot_o[kk];
		}
		s_aSpot_mean = aSpot_sum / N_spots; //integer division!!
	}
	__syncthreads();
	
	if (tid<N_spots)
	{
		wSpot = wSpot * s_aSpot_mean / s_aSpot_o[tid];	
		g_weights[tid + N_spots*(iteration+1)] = wSpot;
		g_amps_o[tid + N_spots*iteration] = s_aSpot_o[tid];		
	}
}


////////////////////////////////////////////////////////////////////////////////
//Obtain phases in SLM plane
////////////////////////////////////////////////////////////////////////////////
__global__ void computePhi(float *g_Vre, 
							float *g_Vim, 
							float *g_Phi, 
							float *g_ei_dre, 
							float *g_ei_dim, 
							int N_pixels, 
							int N_spots, 
							float *g_weights, 
							int iteration, 
							float *g_pSLM_start, 
							float RPC, 
							float *g_amps_o)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;	
	__shared__ float s_a[block_size], s_w[block_size], s_Vre[block_size], s_Vim[block_size];
	float re_SLM = 0, im_SLM = 0, Phi = 0, p_SLM_s = g_pSLM_start[idx], ei_dre, ei_dim;

	if (tid<N_spots)
	{
		s_Vre[tid] = g_Vre[tid*N_pixels];
		s_Vim[tid] = g_Vim[tid*N_pixels];
		s_a[tid] = g_amps_o[tid + (iteration-1)*N_spots];

		s_w[tid] = g_weights[tid + iteration*N_spots];
	}
	__syncthreads();
	for (int k=0; k<N_spots; k++)
	{
		ei_dre = g_ei_dre[idx + k*N_pixels];
		ei_dim = g_ei_dim[idx + k*N_pixels];
		re_SLM += s_w[k] * (ei_dre * s_Vre[k] - ei_dim * s_Vim[k]) / s_a[k];
		im_SLM += s_w[k] * (ei_dre * s_Vim[k] + s_Vre[k] * ei_dim) / s_a[k];
	}

	Phi = atan2f(im_SLM, re_SLM);
	
	
	if (RPC < (2.0f*M_PI))
	{	
		if ((fabs(Phi - p_SLM_s)) < RPC)
		{
			g_Phi[idx] = Phi;
		}
		else
		{
			g_Phi[idx] = p_SLM_s;
		}	
	}
	else	
	{
		g_Phi[idx] = Phi;
	}
}
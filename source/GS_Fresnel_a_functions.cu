#include "hip/hip_runtime.h"
/*
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU Lesser General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Lesser General Public License for more details.

    You should have received a copy of the GNU Lesser General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "GenerateHologramCUDA.h"

//////////////////////////////////////////////////////////////////////////
//Compute the phase difference between each pixel in the SLM and each spot
//////////////////////////////////////////////////////////////////////////
__global__ void computeDelta(float *g_x, 
								float *g_y, 
								float *g_z, 
								float *g_delta, 
								float *g_ei_delta_re, 
								float *g_ei_delta_im, 
								int N_spots, 
								int N_pixels, 
								int data_w)
{
	float N = data_w;
	int logN = (int)log2(N);
	float d = 1.0/N;
	
	__shared__ float s_x[block_size];
	__shared__ float s_y[block_size];
	__shared__ float s_z[block_size];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadIdx.x < N_spots)
	{
		s_x[threadIdx.x] = g_x[threadIdx.x];
		s_y[threadIdx.x] = g_y[threadIdx.x];
		s_z[threadIdx.x] = g_z[threadIdx.x];
	}
	__syncthreads();
	int X_int = idx&(int)(N-1);
 	int Y_int = (idx&(int)(N_pixels-1)-X_int)>>logN;		//(idx-X)>>logN gives error for z!=0
 	float X = d * (float)(X_int - 256);
 	float Y = d * (float)(Y_int - 256);
 	
 	for (int m=0; m<N_spots; m++)
	{	
		if (idx<((m+1)*N_pixels) && idx >= m*N_pixels )
		{
			float delta = M_PI * s_z[m] * (X*X + Y*Y) + 2.0 * M_PI * (X * s_x[m] + Y * s_y[m]);
			g_delta[idx] = delta;
			g_ei_delta_re[idx] = cosf(delta);  
			g_ei_delta_im[idx] = sinf(delta); 
		}
	}
	__syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
//Obtain complex field in image plane using Fresnel propagation
////////////////////////////////////////////////////////////////////////////////
__global__ void computeV(float *g_Vre, 
							float *g_Vim, 
							float *g_pSLM, 
							float *g_delta, 
							int N, 
							int N_spots)
{
	float p, pSLM;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx<N)
	{
		pSLM = g_pSLM[idx];
		for (int ii=0; ii<N_spots; ++ii)
		{
			int id = idx + ii * N;
			p = pSLM - g_delta[id];
 			g_Vre[id] = cosf(p);
			g_Vim[id] = sinf(p);
		}
	}
}

////////////////////////////////////////////////////////////////////////////////
//Compute weights for each spot based on desired and obtaied amplitude
////////////////////////////////////////////////////////////////////////////////
__global__ void computeWeights(float *g_Vre, 
							float *g_Vim, 
							int N_spots, 
							float *g_weights, 
							int iteration, 
							float *g_amps_o, 
							float *g_I, 
							int N_pixels)
{

	int tid = threadIdx.x;	
	__shared__ float s_aSpot_o[block_size], s_aSpot_mean;
	float Vre, Vim, wSpot;

	if (tid<N_spots)
	{										//the desited amplitude for each spot
		Vre = g_Vre[tid*N_pixels];
		Vim = g_Vim[tid*N_pixels];
		s_aSpot_o[tid] = hypotf(Vim, Vre)/sqrtf(g_I[tid]);	//the obtained amplitude for each spot divided by the desired amplitude	
		wSpot = g_weights[tid + iteration*N_spots];			//the previous weight for each spot
	}

	__syncthreads();
	if  (tid==0)
	{
		float aSpot_sum = 0;
		for (int kk=0; kk<N_spots; kk++)
		{
			aSpot_sum += s_aSpot_o[kk];
		}
		s_aSpot_mean = aSpot_sum / N_spots; //integer division!!
	}
	__syncthreads();
	
	if (tid<N_spots)
	{
		wSpot = wSpot * s_aSpot_mean / s_aSpot_o[tid];	
		g_weights[tid + N_spots*(iteration+1)] = wSpot;
		g_amps_o[tid + N_spots*iteration] = s_aSpot_o[tid];		
	}
}


////////////////////////////////////////////////////////////////////////////////
//Obtain phases in SLM plane
////////////////////////////////////////////////////////////////////////////////
__global__ void computePhi(float *g_Vre, 
							float *g_Vim, 
							float *g_Phi, 
							float *g_ei_dre, 
							float *g_ei_dim, 
							int N_pixels, 
							int N_spots, 
							float *g_weights, 
							int iteration, 
							float *g_pSLM_start, 
							float RPC, 
							float *g_amps_o)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;	
	__shared__ float s_a[block_size], s_w[block_size], s_Vre[block_size], s_Vim[block_size];
	float re_SLM = 0, im_SLM = 0, Phi = 0, p_SLM_s = g_pSLM_start[idx], ei_dre, ei_dim;

	if (tid<N_spots)
	{
		s_Vre[tid] = g_Vre[tid*N_pixels];
		s_Vim[tid] = g_Vim[tid*N_pixels];
		s_a[tid] = g_amps_o[tid + (iteration-1)*N_spots];

		s_w[tid] = g_weights[tid + iteration*N_spots];
	}
	__syncthreads();
	for (int k=0; k<N_spots; k++)
	{
		ei_dre = g_ei_dre[idx + k*N_pixels];
		ei_dim = g_ei_dim[idx + k*N_pixels];
		re_SLM += s_w[k] * (ei_dre * s_Vre[k] - ei_dim * s_Vim[k]) / s_a[k];
		im_SLM += s_w[k] * (ei_dre * s_Vim[k] + s_Vre[k] * ei_dim) / s_a[k];
	}

	Phi = atan2f(im_SLM, re_SLM);
	
	
	if (RPC < (2.0f*M_PI))
	{	
		if ((fabs(Phi - p_SLM_s)) < RPC)
		{
			g_Phi[idx] = Phi;
		}
		else
		{
			g_Phi[idx] = p_SLM_s;
		}	
	}
	else	
	{
		g_Phi[idx] = Phi;
	}
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////
//Simplified version of the parallel reduction example in the Nvidia SDK, hardcoded to 512x512 pixels
/////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void sum1(float *g_in, float *g_out, int offset)
{	
	extern __shared__ float s_data[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*512 + tid;

	s_data[tid] = 0;
	s_data[tid] += g_in[i + offset] + g_in[i+128 + offset] + g_in[i+256 + offset] + g_in[i+256 + 128 + offset]; 
	//s_data[tid] += g_in[i + offset] + g_in[i+256 + offset]; 
	__syncthreads();
	
	//if (tid < 128) 
	//{
	//	s_data[tid] += s_data[tid + 128];
	//	__syncthreads();
	//}
	if (tid < 64) 
	{
		s_data[tid] += s_data[tid + 64];
		__syncthreads();
	}
	if (tid < 32) 
	{
		s_data[tid] += s_data[tid + 32];
		s_data[tid] += s_data[tid + 16];
		s_data[tid] += s_data[tid + 8];
		s_data[tid] += s_data[tid + 4];
		s_data[tid] += s_data[tid + 2];
		s_data[tid] += s_data[tid + 1];
	}
	if (tid == 0) g_out[blockIdx.x + offset] = s_data[0];
}
__global__ void sum2(float *g_in, float *g_out, int offset)
{
	extern __shared__ float s_data[];
	unsigned int tid = threadIdx.x;

	s_data[tid] = 0;

	s_data[tid] += g_in[tid + offset] + g_in[tid + 256 + offset]; 
	__syncthreads();
	if (tid < 128) 
	{
		s_data[tid] += s_data[tid + 128];
	__syncthreads();
	}
	if (tid < 64) 
	{
		s_data[tid] += s_data[tid + 64];
		__syncthreads();
	}
			if (tid < 32) 
	{
		s_data[tid] += s_data[tid + 32];
		s_data[tid] += s_data[tid + 16];
		s_data[tid] += s_data[tid + 8];
		s_data[tid] += s_data[tid + 4];
		s_data[tid] += s_data[tid + 2];
		s_data[tid] += s_data[tid + 1];
	}
	if (tid == 0) g_out[offset] = s_data[0];
}
void sumV(float* d_in, float* d_out, int offset)
{
	sum1<<<512, 128, 128*sizeof(float)>>>(d_in, d_out, offset);
	hipDeviceSynchronize();
	sum2<<<1, 256, 256*sizeof(float)>>>(d_out, d_out, offset);
	hipDeviceSynchronize();
	return;
}
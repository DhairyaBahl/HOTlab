#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU Lesser General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Lesser General Public License for more details.

    You should have received a copy of the GNU Lesser General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "GenerateHologramCUDA.h"

//Works for 512x512 pixels only!
__global__ void transformToFarfield(float *g_x, float *g_y, float *g_z, float *g_pSLM, float *g_Vre, float *g_Vim, int N_spots, unsigned int n, int data_w)
{
	int blockSize = 512;
	int spot_number = blockIdx.x;
	unsigned int tid = threadIdx.x;
	unsigned int i = tid;
	
	__shared__ float s_Vre[512];
	__shared__ float s_Vim[512];
	__shared__ float s_xm, s_ym, s_zm;

	s_Vre[tid] = 0;
	s_Vim[tid] = 0;
		
	float N = data_w;
	int logN = (int)log2(N);
	float d = 0.001953125;	//SLM pixel size (1/512)
	
	if (tid == 0)
		s_xm = g_x[spot_number];
	if (tid == 64)
		s_ym = g_y[spot_number];	
	if (tid == 128)
		s_zm = g_z[spot_number];
	
	float X1 = d * ((float)tid - 256.0);
	float Y1 = - d * 256.0;	
	__syncthreads();
		
	while (i < n) 
	{ 
		float pSLM_1 = g_pSLM[i];
		float p = pSLM_1 - M_PI * (s_zm * (X1*X1 + Y1*Y1) + 2 * (X1 * s_xm + Y1 * s_ym));
		
		s_Vre[tid] += cosf(p);
		s_Vim[tid] += sinf(p);

		i += blockSize;
		Y1 += d; 	
	}
	__syncthreads();
 
	if (tid < 256) 
	{ 
		s_Vre[tid] += s_Vre[tid + 256]; 
		s_Vim[tid] += s_Vim[tid + 256];
	} 
	__syncthreads(); 

	if (tid < 128)
	{ 
		s_Vre[tid] += s_Vre[tid + 128];
		s_Vim[tid] += s_Vim[tid + 128];  
	} 
	__syncthreads(); 

	if (tid < 64) 
	{ 
		s_Vre[tid] += s_Vre[tid + 64];
		s_Vim[tid] += s_Vim[tid + 64]; 
	} 
	__syncthreads(); 

	
	if (tid < 32) {

		s_Vre[tid] += s_Vre[tid + 32];
		s_Vim[tid] += s_Vim[tid + 32];

		s_Vre[tid] += s_Vre[tid + 16];
		s_Vim[tid] += s_Vim[tid + 16];

		s_Vre[tid] += s_Vre[tid + 8];
		s_Vim[tid] += s_Vim[tid + 8];

		s_Vre[tid] += s_Vre[tid + 4];
		s_Vim[tid] += s_Vim[tid + 4];

		s_Vre[tid] += s_Vre[tid + 2];
		s_Vim[tid] += s_Vim[tid + 2];

		s_Vre[tid] += s_Vre[tid + 1];
		s_Vim[tid] += s_Vim[tid + 1];
		
	}
	if (tid == 0) 
	{
		g_Vre[spot_number] = s_Vre[0] / 262144.0;
		g_Vim[spot_number] = s_Vim[0] / 262144.0;
	}
}

////////////////////////////////////////////////////////////////////////////////
//Obtain phases in SLM plane
////////////////////////////////////////////////////////////////////////////////
//works only for blocksize 512 and max 256 spots

__global__ void computePhiNew(float *g_x, 
							float *g_y, 
							float *g_z, 
							float *g_I, 
							float *g_SpotsRe, 
							float *g_SpotsIm, 
							float *g_Phi, 
							int N_pixels, 
							int N_spots, 
							float *g_weights, 
							int iteration, 
							float *g_pSLM_start, 
							float RPC, 
							float *g_amps)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;	
	__shared__ float s_aSpot[256], s_a_mean, s_weight[256], s_pSpot[256];
	__shared__ float s_xm[256];
	__shared__ float s_ym[256];
	__shared__ float s_zm[256];
	float re_SLM = 0, im_SLM = 0, Phi = 0, p_SLM_s;

	if (idx<N_pixels)
	{
		//float N = 512;
		//int logN = (int)log2(N);
		float d = 0.001953125;		//Normalized pixel pitch (1/512)
		
		if (RPC < (2.0f*M_PI))
			p_SLM_s = g_pSLM_start[idx];

		if (N_spots <= 64)
		{
			if (tid < N_spots)
			{
				
				float Vre = g_SpotsRe[tid];
				float Vim = g_SpotsIm[tid];
				if (iteration == 0)
					s_aSpot[tid] = 1/sqrtf(g_I[tid]);
				else
					s_aSpot[tid] = hypotf(Vim, Vre)/sqrtf(g_I[tid]);		//divide by the desired amplitude for spot m
				//s_aSpot_sum[tid] = s_aSpot[tid];
				s_pSpot[tid] = atan2f(Vim, Vre);
			}	
			else if ((tid - 64) < N_spots)
				s_weight[tid - 64] = g_weights[tid - 64 + iteration*N_spots];
			else if ((tid - 128) < N_spots)
				s_xm[tid - 128] = g_x[tid - 128];
			else if ((tid - 192) < N_spots)
				s_ym[tid - 192] = g_y[tid - 192];
			else if ((tid - 256) < N_spots)
				s_zm[tid - 256] = g_z[tid - 256];																	
		}
		else
		{		
			if (tid<N_spots)
			{
				float Vre = g_SpotsRe[tid];
				float Vim = g_SpotsIm[tid];
				s_pSpot[tid] = atan2f(Vim, Vre);
				if (iteration == 0)
					s_aSpot[tid] = 1/sqrtf(g_I[tid]);
				else
					s_aSpot[tid] = hypotf(Vim, Vre)/sqrtf(g_I[tid]);		//divide by the desired amplitude for spot m
				//s_aSpot_sum[tid] = s_aSpot[tid];
				s_weight[tid] = g_weights[tid + iteration*N_spots];
				s_xm[tid] = g_x[tid];
				s_ym[tid] = g_y[tid];
				s_zm[tid] = g_z[tid];
				
			}
		}	
		__syncthreads();
		/*	//works only for power of 2 number of spots	
		if (N_spots >= 256) { if (tid < 128) { s_aSpot_sum[tid] += s_aSpot_sum[tid + 128]; } __syncthreads(); }
		if (N_spots >= 128) { if (tid < 64) { s_aSpot_sum[tid] += s_aSpot_sum[tid + 64]; } __syncthreads(); }
		if (tid < 32) 
		{
			if (N_spots >= 64) s_aSpot_sum[tid] += s_aSpot_sum[tid + 32];
			if (N_spots >= 32) s_aSpot_sum[tid] += s_aSpot_sum[tid + 16];
			if (N_spots >= 16) s_aSpot_sum[tid] += s_aSpot_sum[tid + 8];
			if (N_spots >= 8) s_aSpot_sum[tid] += s_aSpot_sum[tid + 4];
			if (N_spots >= 4) s_aSpot_sum[tid] += s_aSpot_sum[tid + 2];
			if (N_spots >= 2) s_aSpot_sum[tid] += s_aSpot_sum[tid + 1];
		}
		*/
		if  (tid==0)
		{
			float s_aSpot_sum = 0;
			for (int jj=0; jj<N_spots;jj++)
			{	
				s_aSpot_sum += s_aSpot[jj];		
			}
			s_a_mean = s_aSpot_sum / N_spots; //integer division!!
		}
		__syncthreads();
	
		if (tid<N_spots)
		{
			s_weight[tid] = s_weight[tid] * s_a_mean / s_aSpot[tid];	
			g_weights[tid + N_spots*(iteration+1)] = s_weight[tid];
			//g_amps[tid + N_spots*iteration] = s_aSpot[tid];		//may be excluded, used for monitoring only
		}
		__syncthreads();		

 		float X = d * ((float)(threadIdx.x) - 256.0);
 		float Y = d * ((float)(blockIdx.x) - 256.0);
	
		for (int k=0; k<N_spots; k++)
		{
			float delta = M_PI * s_zm[k] * (X*X + Y*Y) + 2.0 * M_PI * (X * s_xm[k] + Y * s_ym[k]);

			re_SLM += s_weight[k] * cosf(s_pSpot[k] + delta);
			im_SLM += s_weight[k] * sinf(s_pSpot[k] + delta);
			
			
			//ei_dre = cosf(delta);  
			//ei_dim = sinf(delta);
			
			//re_SLM += s_weight[k] * (ei_dre * s_Vre[k] - ei_dim * s_Vim[k]) / s_aSpot[k];
			//im_SLM += s_weight[k] * (ei_dre * s_Vim[k] + s_Vre[k] * ei_dim) / s_aSpot[k];
		}

		Phi = atan2f(im_SLM, re_SLM);
		
		
		if (RPC < (2.0f*M_PI))
		{	
			if ((fabs(Phi - p_SLM_s)) < RPC)
			{
				g_Phi[idx] = Phi;
			}
			else
			{
				g_Phi[idx] = p_SLM_s;
			}	
		}
		else	
		{
			g_Phi[idx] = Phi;
		}
	}
}

#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU Lesser General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Lesser General Public License for more details.

    You should have received a copy of the GNU Lesser General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "GenerateHologramCUDA.h"
////////////////////////////////////////////////////////////////////////////////////
//Common functions
///////////////////////////////////////////////////////////////////////////////////
__device__ unsigned char phase2uc(float phase2pi)
{
	return (unsigned char)floor((phase2pi + M_PI)*256.0f / (2.0f * M_PI));
}
__device__ int phase2int32(float phase2pi)
{
	return (int)floor((phase2pi + M_PI)*256.0f / (2.0f * M_PI));
}
__device__ float ApplyAberrationCorrection(float phase, float correction)
{
		phase = phase + correction;		//apply correction
		return (phase - (2.0f*M_PI) * floor((phase+M_PI) / (2.0f*M_PI))); //apply mod([-pi, pi], phase) 
}

/*__device__ unsigned char applyPolLUT(float phase2pi, float X, float Y, float *s_c, int N_PolCoeff)		
{
	float phase255 = 0.0f;
	phase2pi += M_PI;
	switch (N_PolCoeff)	{
		case 120:
			phase255 += s_c[84]*powf(X,7);
			phase255 += s_c[85]*powf(X,6)*Y;
			phase255 += s_c[86]*powf(X,6)*phase2pi;
			phase255 += s_c[87]*powf(X,5)*powf(Y,2);
			phase255 += s_c[88]*powf(X,5)*Y*phase2pi;
			phase255 += s_c[89]*powf(X,5)*powf(phase2pi,2);
			phase255 += s_c[90]*powf(X,4)*powf(Y,3);
			phase255 += s_c[91]*powf(X,4)*powf(Y,2)*phase2pi;
			phase255 += s_c[92]*powf(X,4)*Y*powf(phase2pi,2);
			phase255 += s_c[93]*powf(X,4)*powf(phase2pi,3);
			phase255 += s_c[94]*powf(X,3)*powf(Y,4);
			phase255 += s_c[95]*powf(X,3)*powf(Y,3)*phase2pi;
			phase255 += s_c[96]*powf(X,3)*powf(Y,2)*powf(phase2pi,2);
			phase255 += s_c[97]*powf(X,3)*Y*powf(phase2pi,3);
			phase255 += s_c[98]*powf(X,3)*powf(phase2pi,4);
			phase255 += s_c[99]*powf(X,2)*powf(Y,5);
			phase255 += s_c[100]*powf(X,2)*powf(Y,4)*phase2pi;
			phase255 += s_c[101]*powf(X,2)*powf(Y,3)*powf(phase2pi,2);
			phase255 += s_c[102]*powf(X,2)*powf(Y,2)*powf(phase2pi,3);
			phase255 += s_c[103]*powf(X,2)*Y*powf(phase2pi,4);
			phase255 += s_c[104]*powf(X,2)*powf(phase2pi,5);
			phase255 += s_c[105]*X*powf(Y,6);
			phase255 += s_c[106]*X*powf(Y,5)*phase2pi;
			phase255 += s_c[107]*X*powf(Y,4)*powf(phase2pi,2);
			phase255 += s_c[108]*X*powf(Y,3)*powf(phase2pi,3);
			phase255 += s_c[109]*X*powf(Y,2)*powf(phase2pi,4);
			phase255 += s_c[110]*X*Y*powf(phase2pi,5);
			phase255 += s_c[111]*X*powf(phase2pi,6);
			phase255 += s_c[112]*powf(Y,7);
			phase255 += s_c[113]*powf(Y,6)*phase2pi;
			phase255 += s_c[114]*powf(Y,5)*powf(phase2pi,2);
			phase255 += s_c[115]*powf(Y,4)*powf(phase2pi,3);
			phase255 += s_c[116]*powf(Y,3)*powf(phase2pi,4);
			phase255 += s_c[117]*powf(Y,2)*powf(phase2pi,5);
			phase255 += s_c[118]*Y*powf(phase2pi,6);
			phase255 += s_c[119]*powf(phase2pi,7);
		case 84:
			phase255 += s_c[56]*powf(X,6);
			phase255 += s_c[57]*powf(X,5)*Y;
			phase255 += s_c[58]*powf(X,5)*phase2pi;
			phase255 += s_c[59]*powf(X,4)*powf(Y,2);
			phase255 += s_c[60]*powf(X,4)*Y*phase2pi;
			phase255 += s_c[61]*powf(X,4)*powf(phase2pi,2);
			phase255 += s_c[62]*powf(X,3)*powf(Y,3);
			phase255 += s_c[63]*powf(X,3)*powf(Y,2)*phase2pi;
			phase255 += s_c[64]*powf(X,3)*Y*powf(phase2pi,2);
			phase255 += s_c[65]*powf(X,3)*powf(phase2pi,3);
			phase255 += s_c[66]*powf(X,2)*powf(Y,4);
			phase255 += s_c[67]*powf(X,2)*powf(Y,3)*phase2pi;
			phase255 += s_c[68]*powf(X,2)*powf(Y,2)*powf(phase2pi,2);
			phase255 += s_c[69]*powf(X,2)*Y*powf(phase2pi,3);
			phase255 += s_c[70]*powf(X,2)*powf(phase2pi,4);
			phase255 += s_c[71]*X*powf(Y,5);
			phase255 += s_c[72]*X*powf(Y,4)*phase2pi;
			phase255 += s_c[73]*X*powf(Y,3)*powf(phase2pi,2);
			phase255 += s_c[74]*X*powf(Y,2)*powf(phase2pi,3);
			phase255 += s_c[75]*X*Y*powf(phase2pi,4);
			phase255 += s_c[76]*X*powf(phase2pi,5);
			phase255 += s_c[77]*powf(Y,6);
			phase255 += s_c[78]*powf(Y,5)*phase2pi;
			phase255 += s_c[79]*powf(Y,4)*powf(phase2pi,2);
			phase255 += s_c[80]*powf(Y,3)*powf(phase2pi,3);
			phase255 += s_c[81]*powf(Y,2)*powf(phase2pi,4);
			phase255 += s_c[82]*Y*powf(phase2pi,5);
			phase255 += s_c[83]*powf(phase2pi,6);
		case 56:
			phase255 += s_c[35]*powf(X,5);
			phase255 += s_c[36]*powf(X,4)*Y;
			phase255 += s_c[37]*powf(X,4)*phase2pi;
			phase255 += s_c[38]*powf(X,3)*powf(Y,2);
			phase255 += s_c[39]*powf(X,3)*Y*phase2pi;
			phase255 += s_c[40]*powf(X,3)*powf(phase2pi,2);
			phase255 += s_c[41]*powf(X,2)*powf(Y,3);
			phase255 += s_c[42]*powf(X,2)*powf(Y,2)*phase2pi;
			phase255 += s_c[43]*powf(X,2)*Y*powf(phase2pi,2);
			phase255 += s_c[44]*powf(X,2)*powf(phase2pi,3);
			phase255 += s_c[45]*X*powf(Y,4);
			phase255 += s_c[46]*X*powf(Y,3)*phase2pi;
			phase255 += s_c[47]*X*powf(Y,2)*powf(phase2pi,2);
			phase255 += s_c[48]*X*Y*powf(phase2pi,3);
			phase255 += s_c[49]*X*powf(phase2pi,4);
			phase255 += s_c[50]*powf(Y,5);
			phase255 += s_c[51]*powf(Y,4)*phase2pi;
			phase255 += s_c[52]*powf(Y,3)*powf(phase2pi,2);
			phase255 += s_c[53]*powf(Y,2)*powf(phase2pi,3);
			phase255 += s_c[54]*Y*powf(phase2pi,4);
			phase255 += s_c[55]*powf(phase2pi,5);
		case 35:
			phase255 += s_c[20]*powf(X,4);
			phase255 += s_c[21]*powf(X,3)*Y;
			phase255 += s_c[22]*powf(X,3)*phase2pi;
			phase255 += s_c[23]*powf(X,2)*powf(Y,2);
			phase255 += s_c[24]*powf(X,2)*Y*phase2pi;
			phase255 += s_c[25]*powf(X,2)*powf(phase2pi,2);
			phase255 += s_c[26]*X*powf(Y,3);
			phase255 += s_c[27]*X*powf(Y,2)*phase2pi;
			phase255 += s_c[28]*X*Y*powf(phase2pi,2);
			phase255 += s_c[29]*X*powf(phase2pi,3);
			phase255 += s_c[30]*powf(Y,4);
			phase255 += s_c[31]*powf(Y,3)*phase2pi;
			phase255 += s_c[32]*powf(Y,2)*powf(phase2pi,2);
			phase255 += s_c[33]*Y*powf(phase2pi,3);
			phase255 += s_c[34]*powf(phase2pi,4);
		case 20:
			phase255 += s_c[0];
			phase255 += s_c[1]*X;
			phase255 += s_c[2]*Y;
			phase255 += s_c[3]*phase2pi;
			phase255 += s_c[4]*powf(X,2);
			phase255 += s_c[5]*X*Y;
			phase255 += s_c[6]*X*phase2pi;
			phase255 += s_c[7]*powf(Y,2);
			phase255 += s_c[8]*Y*phase2pi;
			phase255 += s_c[9]*powf(phase2pi,2);
			phase255 += s_c[10]*powf(X,3);
			phase255 += s_c[11]*powf(X,2)*Y;
			phase255 += s_c[12]*powf(X,2)*phase2pi;
			phase255 += s_c[13]*X*powf(Y,2);
			phase255 += s_c[14]*X*Y*phase2pi;
			phase255 += s_c[15]*X*powf(phase2pi,2);
			phase255 += s_c[16]*powf(Y,3);
			phase255 += s_c[17]*powf(Y,2)*phase2pi;
			phase255 += s_c[18]*Y*powf(phase2pi,2);
			phase255 += s_c[19]*powf(phase2pi,3);
			break;
		default:
			phase255 = 0;
			break;
	}
	if (phase255 < 0)
		phase255 = 0;
	return (unsigned char)(phase255);
}*/
__device__ unsigned char applyPolLUT(float phase2pi, float X, float Y, float *s_c, int N_PolCoeff)		
{
	phase2pi += M_PI;
	switch (N_PolCoeff)	{
		case 120:
			return (unsigned char)(s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5) + s_c[56]*powf(X,6) + s_c[57]*powf(X,5)*Y + s_c[58]*powf(X,5)*phase2pi + s_c[59]*powf(X,4)*powf(Y,2) + s_c[60]*powf(X,4)*Y*phase2pi + s_c[61]*powf(X,4)*powf(phase2pi,2) + s_c[62]*powf(X,3)*powf(Y,3) + s_c[63]*powf(X,3)*powf(Y,2)*phase2pi + s_c[64]*powf(X,3)*Y*powf(phase2pi,2) + s_c[65]*powf(X,3)*powf(phase2pi,3) + s_c[66]*powf(X,2)*powf(Y,4) + s_c[67]*powf(X,2)*powf(Y,3)*phase2pi + s_c[68]*powf(X,2)*powf(Y,2)*powf(phase2pi,2) + s_c[69]*powf(X,2)*Y*powf(phase2pi,3) + s_c[70]*powf(X,2)*powf(phase2pi,4) + s_c[71]*X*powf(Y,5) + s_c[72]*X*powf(Y,4)*phase2pi + s_c[73]*X*powf(Y,3)*powf(phase2pi,2) + s_c[74]*X*powf(Y,2)*powf(phase2pi,3) + s_c[75]*X*Y*powf(phase2pi,4) + s_c[76]*X*powf(phase2pi,5) + s_c[77]*powf(Y,6) + s_c[78]*powf(Y,5)*phase2pi + s_c[79]*powf(Y,4)*powf(phase2pi,2) + s_c[80]*powf(Y,3)*powf(phase2pi,3) + s_c[81]*powf(Y,2)*powf(phase2pi,4) + s_c[82]*Y*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6) + s_c[84]*powf(X,7) + s_c[85]*powf(X,6)*Y + s_c[86]*powf(X,6)*phase2pi + s_c[87]*powf(X,5)*powf(Y,2) + s_c[88]*powf(X,5)*Y*phase2pi + s_c[89]*powf(X,5)*powf(phase2pi,2) + s_c[90]*powf(X,4)*powf(Y,3) + s_c[91]*powf(X,4)*powf(Y,2)*phase2pi + s_c[92]*powf(X,4)*Y*powf(phase2pi,2) + s_c[93]*powf(X,4)*powf(phase2pi,3) + s_c[94]*powf(X,3)*powf(Y,4) + s_c[95]*powf(X,3)*powf(Y,3)*phase2pi + s_c[96]*powf(X,3)*powf(Y,2)*powf(phase2pi,2) + s_c[97]*powf(X,3)*Y*powf(phase2pi,3) + s_c[98]*powf(X,3)*powf(phase2pi,4) + s_c[99]*powf(X,2)*powf(Y,5) + s_c[100]*powf(X,2)*powf(Y,4)*phase2pi + s_c[101]*powf(X,2)*powf(Y,3)*powf(phase2pi,2) + s_c[102]*powf(X,2)*powf(Y,2)*powf(phase2pi,3) + s_c[103]*powf(X,2)*Y*powf(phase2pi,4) + s_c[104]*powf(X,2)*powf(phase2pi,5) + s_c[105]*X*powf(Y,6) + s_c[106]*X*powf(Y,5)*phase2pi + s_c[107]*X*powf(Y,4)*powf(phase2pi,2) + s_c[108]*X*powf(Y,3)*powf(phase2pi,3) + s_c[109]*X*powf(Y,2)*powf(phase2pi,4) + s_c[110]*X*Y*powf(phase2pi,5) + s_c[111]*X*powf(phase2pi,6) + s_c[112]*powf(Y,7) + s_c[113]*powf(Y,6)*phase2pi + s_c[114]*powf(Y,5)*powf(phase2pi,2) + s_c[115]*powf(Y,4)*powf(phase2pi,3) + s_c[116]*powf(Y,3)*powf(phase2pi,4) + s_c[117]*powf(Y,2)*powf(phase2pi,5) + s_c[118]*Y*powf(phase2pi,6) + s_c[119]*powf(phase2pi,7));
		case 84:
			return (unsigned char)(s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5) + s_c[56]*powf(X,6) + s_c[57]*powf(X,5)*Y + s_c[58]*powf(X,5)*phase2pi + s_c[59]*powf(X,4)*powf(Y,2) + s_c[60]*powf(X,4)*Y*phase2pi + s_c[61]*powf(X,4)*powf(phase2pi,2) + s_c[62]*powf(X,3)*powf(Y,3) + s_c[63]*powf(X,3)*powf(Y,2)*phase2pi + s_c[64]*powf(X,3)*Y*powf(phase2pi,2) + s_c[65]*powf(X,3)*powf(phase2pi,3) + s_c[66]*powf(X,2)*powf(Y,4) + s_c[67]*powf(X,2)*powf(Y,3)*phase2pi + s_c[68]*powf(X,2)*powf(Y,2)*powf(phase2pi,2) + s_c[69]*powf(X,2)*Y*powf(phase2pi,3) + s_c[70]*powf(X,2)*powf(phase2pi,4) + s_c[71]*X*powf(Y,5) + s_c[72]*X*powf(Y,4)*phase2pi + s_c[73]*X*powf(Y,3)*powf(phase2pi,2) + s_c[74]*X*powf(Y,2)*powf(phase2pi,3) + s_c[75]*X*Y*powf(phase2pi,4) + s_c[76]*X*powf(phase2pi,5) + s_c[77]*powf(Y,6) + s_c[78]*powf(Y,5)*phase2pi + s_c[79]*powf(Y,4)*powf(phase2pi,2) + s_c[80]*powf(Y,3)*powf(phase2pi,3) + s_c[81]*powf(Y,2)*powf(phase2pi,4) + s_c[82]*Y*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6));
		case 56:
			return (unsigned char)(s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5));
		case 35:
			return (unsigned char)(s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4));
		case 20:
			return (unsigned char)(s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3));
		default:
			return 0;
	}
}
/*__device__ unsigned char applyPolLUT(float phase2pi, float X, float Y, float *s_c, int N_PolCoeff)		
{
	phase2pi += M_PI;
	float phase255 = 0.0f;
	switch (N_PolCoeff)	{
		case 120:
			phase255 += (s_c[84]*powf(X,7) + s_c[85]*powf(X,6)*Y + s_c[86]*powf(X,6)*phase2pi + s_c[87]*powf(X,5)*powf(Y,2) + s_c[88]*powf(X,5)*Y*phase2pi + s_c[89]*powf(X,5)*powf(phase2pi,2) + s_c[90]*powf(X,4)*powf(Y,3) + s_c[91]*powf(X,4)*powf(Y,2)*phase2pi + s_c[92]*powf(X,4)*Y*powf(phase2pi,2) + s_c[93]*powf(X,4)*powf(phase2pi,3) + s_c[94]*powf(X,3)*powf(Y,4) + s_c[95]*powf(X,3)*powf(Y,3)*phase2pi + s_c[96]*powf(X,3)*powf(Y,2)*powf(phase2pi,2) + s_c[97]*powf(X,3)*Y*powf(phase2pi,3) + s_c[98]*powf(X,3)*powf(phase2pi,4) + s_c[99]*powf(X,2)*powf(Y,5) + s_c[100]*powf(X,2)*powf(Y,4)*phase2pi + s_c[101]*powf(X,2)*powf(Y,3)*powf(phase2pi,2) + s_c[102]*powf(X,2)*powf(Y,2)*powf(phase2pi,3) + s_c[103]*powf(X,2)*Y*powf(phase2pi,4) + s_c[104]*powf(X,2)*powf(phase2pi,5) + s_c[105]*X*powf(Y,6) + s_c[106]*X*powf(Y,5)*phase2pi + s_c[107]*X*powf(Y,4)*powf(phase2pi,2) + s_c[108]*X*powf(Y,3)*powf(phase2pi,3) + s_c[109]*X*powf(Y,2)*powf(phase2pi,4) + s_c[110]*X*Y*powf(phase2pi,5) + s_c[111]*X*powf(phase2pi,6) + s_c[112]*powf(Y,7) + s_c[113]*powf(Y,6)*phase2pi + s_c[114]*powf(Y,5)*powf(phase2pi,2) + s_c[115]*powf(Y,4)*powf(phase2pi,3) + s_c[116]*powf(Y,3)*powf(phase2pi,4) + s_c[117]*powf(Y,2)*powf(phase2pi,5) + s_c[118]*Y*powf(phase2pi,6) + s_c[119]*powf(phase2pi,7));
		case 84:
			phase255 += (s_c[56]*powf(X,6) + s_c[57]*powf(X,5)*Y + s_c[58]*powf(X,5)*phase2pi + s_c[59]*powf(X,4)*powf(Y,2) + s_c[60]*powf(X,4)*Y*phase2pi + s_c[61]*powf(X,4)*powf(phase2pi,2) + s_c[62]*powf(X,3)*powf(Y,3) + s_c[63]*powf(X,3)*powf(Y,2)*phase2pi + s_c[64]*powf(X,3)*Y*powf(phase2pi,2) + s_c[65]*powf(X,3)*powf(phase2pi,3) + s_c[66]*powf(X,2)*powf(Y,4) + s_c[67]*powf(X,2)*powf(Y,3)*phase2pi + s_c[68]*powf(X,2)*powf(Y,2)*powf(phase2pi,2) + s_c[69]*powf(X,2)*Y*powf(phase2pi,3) + s_c[70]*powf(X,2)*powf(phase2pi,4) + s_c[71]*X*powf(Y,5) + s_c[72]*X*powf(Y,4)*phase2pi + s_c[73]*X*powf(Y,3)*powf(phase2pi,2) + s_c[74]*X*powf(Y,2)*powf(phase2pi,3) + s_c[75]*X*Y*powf(phase2pi,4) + s_c[76]*X*powf(phase2pi,5) + s_c[77]*powf(Y,6) + s_c[78]*powf(Y,5)*phase2pi + s_c[79]*powf(Y,4)*powf(phase2pi,2) + s_c[80]*powf(Y,3)*powf(phase2pi,3) + s_c[81]*powf(Y,2)*powf(phase2pi,4) + s_c[82]*Y*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6));
		case 56:
			phase255 += (s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5));
		case 35:
			phase255 += (s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4));
		case 20:
			phase255 += (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2));// + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3));
			break;
		default:
			phase255 = 0;
			break;
	}
	if (phase255 < 0)
		phase255 = 0;
	return (unsigned char)phase255;
}*/
__device__ void warpReduceC(volatile float *s_Vre, volatile float *s_Vim, int tid)
{
	s_Vre[tid] += s_Vre[tid + 32];
	s_Vim[tid] += s_Vim[tid + 32];

	s_Vre[tid] += s_Vre[tid + 16];
	s_Vim[tid] += s_Vim[tid + 16];

	s_Vre[tid] += s_Vre[tid + 8];
	s_Vim[tid] += s_Vim[tid + 8];

	s_Vre[tid] += s_Vre[tid + 4];
	s_Vim[tid] += s_Vim[tid + 4];

	s_Vre[tid] += s_Vre[tid + 2];
	s_Vim[tid] += s_Vim[tid + 2];

	s_Vre[tid] += s_Vre[tid + 1];
	s_Vim[tid] += s_Vim[tid + 1];
}

/////////////////////////////////////////////////////////////////////////////////////////////////
//Calculate hologram using "Lenses and Prisms"
/////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void LensesAndPrisms(float *g_x, float *g_y, float *g_z, float *g_I, unsigned char *g_SLMuc, int N_spots, unsigned char *g_LUT, bool ApplyLUT_b, int data_w, bool UseAberrationCorr_b, float *d_AberrationCorr_f, bool UseLUTPol_b, float *d_LUTPolCoeff_f, int N_PolCoeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	
	__shared__ float s_x[BLOCK_SIZE];
	__shared__ float s_y[BLOCK_SIZE];
	__shared__ float s_z[BLOCK_SIZE];
	__shared__ float s_a[BLOCK_SIZE];
	
	if (tid < N_spots)
	{
		s_x[tid] = g_x[tid];
		s_y[tid] = g_y[tid];
		s_z[tid] = g_z[tid];
		s_a[tid] = sqrtf(g_I[tid]);
	}
	__syncthreads();	
	
	if (idx < data_w*data_w)
	{
		//get pixel coordinates
		float d = 0.001953125; //pixel pitch (1/512)
 		float X = d * ((float)threadIdx.x - 256.0f);	//512!
 		float Y = d * ((float)blockIdx.x - 256.0f);
		
		//change this to allow data_w!=512
		//float X = d*((float)(idx%data_w) - (float)data_w/2.0f);
		//float Y = d*((float)(floor((float)idx/(float)data_w)) - (float)data_w/2.0f);float N = data_w;
		
		/*...or to this (data_w must be a power of 2)
		int logN = (int)log2(N);
		int X_int = idx&(int)(N-1);
	 	int Y_int = (idx-X_int)>>logN;
	 	float X = (float)(X_int - (data_w>>1)) / N; //512
 		float Y = (float)(Y_int - (data_w>>1)) / N;*/
		
		float phase2pi;  // [-pi,pi]
		float SLMre = 0;
		float SLMim = 0;
				
		for (int ii=0; ii<N_spots; ++ii)
		{
			phase2pi = M_PI * s_z[ii] * (X*X + Y*Y) + 2.0 * M_PI * (X * (s_x[ii]) + Y * (s_y[ii]) );
			SLMre = SLMre + s_a[ii] * cosf(phase2pi);
			SLMim = SLMim + s_a[ii] * sinf(phase2pi); 
		}
		phase2pi = atan2f(SLMim, SLMre);	
		
		if (UseAberrationCorr_b)
			phase2pi = ApplyAberrationCorrection(phase2pi, d_AberrationCorr_f[idx]);

		if (ApplyLUT_b) 
		{
			if (!UseLUTPol_b)
			{
				__shared__ unsigned char s_LUT[256];
				if (tid < 256)
					s_LUT[tid] = g_LUT[tid];
				__syncthreads();
				g_SLMuc[idx] = s_LUT[phase2int32(phase2pi)];
			}
			else
			{
				__shared__ float s_LUTcoeff[120];
				if (tid < N_PolCoeff)
					s_LUTcoeff[tid] = d_LUTPolCoeff_f[tid];
				__syncthreads();
				g_SLMuc[idx] = applyPolLUT(phase2pi, X, Y, s_LUTcoeff, N_PolCoeff);
			}
		}
		else
			g_SLMuc[idx] = phase2uc(phase2pi);
	}	
	__syncthreads();

}

__global__ void checkAmplitudes(float *g_x, float *g_y, float *g_z, unsigned char *g_pSLM_uc, float *g_amps, int N_spots, int N_pixels, int data_w)
{
	int blockSize = 512;
	int spot_number = blockIdx.x;
	int tid = threadIdx.x;
	int i = tid;
	
	__shared__ float s_Vre[BLOCK_SIZE];
	__shared__ float s_Vim[BLOCK_SIZE];
	float s_xm, s_ym, s_zm;

	s_Vre[tid] = 0;
	s_Vim[tid] = 0;
		
	float N = data_w;
	int logN = (int)log2(N);
	float d = 0.001953125;	//SLM pixel size (1/512)	512!
	

	s_xm = g_x[spot_number];

	s_ym = g_y[spot_number];	

	s_zm = g_z[spot_number];
	
	float X1 = d * ((float)tid - 256.0);				//512!
	float Y1 = - d * 256.0;	

	float pSLM_1;
	float p;
	while (i < N_pixels) 
	{ 
		pSLM_1 = 2.0f*M_PI*(float)g_pSLM_uc[i]/255.0f - M_PI;
		p = pSLM_1 - M_PI * (s_zm * (X1*X1 + Y1*Y1) + 2 * (X1 * s_xm + Y1 * s_ym));
		
		s_Vre[tid] += cosf(p);
		s_Vim[tid] += sinf(p);

		i += blockSize;
		Y1 += d; 	
	}
	__syncthreads();
 
	if (tid < 256) 
	{ 
		s_Vre[tid] += s_Vre[tid + 256]; 
		s_Vim[tid] += s_Vim[tid + 256];
	} 
	__syncthreads(); 

	if (tid < 128)
	{ 
		s_Vre[tid] += s_Vre[tid + 128];
		s_Vim[tid] += s_Vim[tid + 128];  
	} 
	__syncthreads(); 

	if (tid < 64) 
	{ 
		s_Vre[tid] += s_Vre[tid + 64];
		s_Vim[tid] += s_Vim[tid + 64]; 
	} 
	__syncthreads(); 

	if (tid < 32) 
		warpReduceC(s_Vre, s_Vim, tid);

	if (tid == 0) 
	{
		float Vre = s_Vre[0] / 262144.0;			//512!
		float Vim = s_Vim[0] / 262144.0;
		g_amps[spot_number] = hypotf(Vim, Vre);
	}
}
////////////////////////////////////////////////////////////////////////////////
//Functions for GS with Fresnel propagation
////////////////////////////////////////////////////////////////////////////////
//Propagate from the SLM to the spot positions using Fresnel summation
//(Works for 512x512 pixels only!)
////////////////////////////////////////////////////////////////////////////////
__global__ void PropagateToSpotPositions_Fresnel(float *g_x, float *g_y, float *g_z, float *g_pSLM2pi, float *g_Vre, float *g_Vim, int N_spots, int n, int data_w)
{
	int blockSize = 512;
	int spot_number = blockIdx.x;
	int tid = threadIdx.x;
	int i = tid;
	
	__shared__ float s_Vre[BLOCK_SIZE];		
	__shared__ float s_Vim[BLOCK_SIZE];
	//__shared__ float s_xm, s_ym, s_zm;
	float s_xm, s_ym, s_zm;

	s_Vre[tid] = 0;
	s_Vim[tid] = 0;

	s_xm = g_x[spot_number];
	s_ym = g_y[spot_number];	
	s_zm = g_z[spot_number];

	float d = 0.001953125;	//Normalized pixel pitch (1/512) 512!	
	float X1 = d * (float)(tid - 256);	//512!
	float Y1 = - d * 256.0f;
	//float Y2 = - d * 255.0f;
	float p;
	//__syncthreads();
	while (i < n) 
	{ 
		p = g_pSLM2pi[i] - M_PI * (s_zm * (X1*X1 + Y1*Y1) + 2.0f * (X1 * s_xm + Y1 * s_ym));
		
		s_Vre[tid] += cosf(p);
		s_Vim[tid] += sinf(p);

		i += blockSize;
		Y1 += d; 	
	}
	__syncthreads();
 
	if (tid < 256) 
	{ 
		s_Vre[tid] += s_Vre[tid + 256]; 
		s_Vim[tid] += s_Vim[tid + 256];
	} 
	__syncthreads(); 

	if (tid < 128)
	{ 
		s_Vre[tid] += s_Vre[tid + 128];
		s_Vim[tid] += s_Vim[tid + 128];  
	} 
	__syncthreads(); 

	if (tid < 64) 
	{ 
		s_Vre[tid] += s_Vre[tid + 64];
		s_Vim[tid] += s_Vim[tid + 64]; 
	} 
	__syncthreads(); 

	
	if (tid < 32)
		warpReduceC(s_Vre, s_Vim, tid);

	if (tid == 0) 
	{
		g_Vre[spot_number] = s_Vre[0] / 262144.0f;
		g_Vim[spot_number] = s_Vim[0] / 262144.0f;
	}
}


////////////////////////////////////////////////////////////////////////////////
//Obtain phases in SLM plane
////////////////////////////////////////////////////////////////////////////////
//works only for blocksize 512 and max 512 spots
////////////////////////////////////////////////////////////////////////////////
__global__ void PropagateToSLM_Fresnel(float *g_x, 
								float *g_y, 
								float *g_z, 
								float *g_I, 
								float *g_SpotsRe, 
								float *g_SpotsIm, 
								float *g_pSLM2pi, 
								int N_pixels, 
								int N_spots, 
								float *g_weights, 
								int iteration, 
								float *g_pSLMstart, 
								float RPC, 
								float *g_amps,
								bool getpSLM255,
								unsigned char *g_pSLM255_uc,
								unsigned char *g_LUT, 
								bool ApplyLUT_b, 
								bool UseAberrationCorr_b, 
								float *g_AberrationCorr_f, 
								bool UseLUTPol_b, 
								float *g_LUTPolCoeff_f, 
								int N_PolCoeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;	
	__shared__ float s_aSpot[BLOCK_SIZE], s_a_mean, s_weight[BLOCK_SIZE], s_pSpot[BLOCK_SIZE];
	__shared__ float s_xm[BLOCK_SIZE];
	__shared__ float s_ym[BLOCK_SIZE];
	__shared__ float s_zm[BLOCK_SIZE];
	float reSLM = 0, imSLM = 0, pSLM2pi_f = 0;

	if (idx<N_pixels)
	{
		//float N = 512;
		//int logN = (int)log2(N);

		
		//load data to shared memory
		if (N_spots <= 64)
		{
			if (tid < N_spots)
			{
				float Vre = g_SpotsRe[tid];
				float Vim = g_SpotsIm[tid];
				if (iteration == 0)
					s_aSpot[tid] = 1.0f/sqrtf(g_I[tid]);
				else
					s_aSpot[tid] = hypotf(Vim, Vre)/sqrtf(g_I[tid]);		//divide by the desired amplitude for spot m
				//s_aSpot_sum[tid] = s_aSpot[tid];
				s_pSpot[tid] = atan2f(Vim, Vre);
			}	
			else if ((tid - 64) < N_spots)
				s_weight[tid - 64] = g_weights[tid - 64 + iteration*N_spots];
			else if ((tid - 128) < N_spots)
				s_xm[tid - 128] = g_x[tid - 128];
			else if ((tid - 192) < N_spots)
				s_ym[tid - 192] = g_y[tid - 192];
			else if ((tid - 256) < N_spots)
				s_zm[tid - 256] = g_z[tid - 256];																	
		}
		else
		{	
			if (tid<N_spots)
			{
				float Vre = g_SpotsRe[tid];
				float Vim = g_SpotsIm[tid];
				s_pSpot[tid] = atan2f(Vim, Vre);
				if (iteration == 0)
					s_aSpot[tid] = 1/sqrtf(g_I[tid]);
				else
					s_aSpot[tid] = hypotf(Vim, Vre)/sqrtf(g_I[tid]);		//divide by the desired amplitude for spot m
				//s_aSpot_sum[tid] = s_aSpot[tid];
				s_weight[tid] = g_weights[tid + iteration*N_spots];
				s_xm[tid] = g_x[tid];
				s_ym[tid] = g_y[tid];
				s_zm[tid] = g_z[tid];
				
			}
		}	
		__syncthreads();		

		//compute weights 
		if  (tid==0)
		{
			float s_aSpot_sum = 0;
			for (int jj=0; jj<N_spots;jj++)
			{	
				s_aSpot_sum += s_aSpot[jj];		
			}
			s_a_mean = s_aSpot_sum / (float)N_spots;				//integer division!!
		}
		__syncthreads();
	
		if (tid<N_spots)
		{
			s_weight[tid] = s_weight[tid] * s_a_mean / s_aSpot[tid];	
			g_weights[tid + N_spots*(iteration+1)] = s_weight[tid];
			g_amps[tid + N_spots*iteration] = s_aSpot[tid];			//may be excluded, used for monitoring only
		}
		__syncthreads();				
		//get pixel coordinates
		float d = 0.001953125;										//Normalized pixel pitch (1/512) 512!
 		float X = d * ((float)threadIdx.x - 256.0f);				//512!
 		float Y = d * ((float)blockIdx.x - 256.0f);
		
		//change this to allow data_w!=512
		//float X = d*((float)(idx%data_w) - (float)data_w/2.0f);
		//float Y = d*((float)(floor((float)idx/(float)data_w)) - (float)data_w/2.0f);
		
		//compute SLM phase by summing contribution from all spots
		for (int k=0; k<N_spots; k++)
		{
			float delta = M_PI * s_zm[k] * (X*X + Y*Y) + 2.0f * M_PI * (X * s_xm[k] + Y * s_ym[k]);
			reSLM += s_weight[k] * cosf(s_pSpot[k] + delta);
			imSLM += s_weight[k] * sinf(s_pSpot[k] + delta);
		}
		pSLM2pi_f = atan2f(imSLM, reSLM);		
		
	
		if (RPC < (2.0f*M_PI))			//Apply RPC (restricted Phase Change)
		{	
			float pSLMstart = g_pSLMstart[idx];
			if (fabs(pSLM2pi_f - pSLMstart) > RPC)
				pSLM2pi_f = pSLMstart;
			if (getpSLM255)
				g_pSLMstart[idx] = pSLM2pi_f;
		}		

		if (getpSLM255)					//Compute final SLM phases and write to global memory... 
		{								
			if (UseAberrationCorr_b)
				pSLM2pi_f = ApplyAberrationCorrection(pSLM2pi_f, g_AberrationCorr_f[idx]);
			if (ApplyLUT_b)
			{
				if (!UseLUTPol_b)
				{
					__shared__ unsigned char s_LUT[256];
					if (tid < 256)
						s_LUT[tid] = g_LUT[tid];
					__syncthreads();
					g_pSLM255_uc[idx] = s_LUT[phase2int32(pSLM2pi_f)];
				}
				else
				{
					__shared__ float s_LUTcoeff[120];
					if (tid < N_PolCoeff)
						s_LUTcoeff[tid] = g_LUTPolCoeff_f[tid];
					__syncthreads();
					g_pSLM255_uc[idx] = applyPolLUT(pSLM2pi_f, X, Y, s_LUTcoeff, N_PolCoeff);
				}
			}
			else
				g_pSLM255_uc[idx] = phase2uc(pSLM2pi_f);
		}
		else
		{
			g_pSLM2pi[idx] = pSLM2pi_f;	//...or write intermediate phase to global memory
		}
	}
}

////////////////////////////////////////////////////////////////////////////////
//Calculate complex from phases
////////////////////////////////////////////////////////////////////////////////
__global__ void p2c(hipfftComplex *g_c, float *g_p, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		float phase = g_p[idx];
 		g_c[idx].x = cosf(phase);
		g_c[idx].y = sinf(phase);
	}
	__syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
//Calculate amplitudes from complex
////////////////////////////////////////////////////////////////////////////////
__global__ void c_cc2a_f(float *g_a, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_a[idx] = hypotf(g_c[idx].x, g_c[idx].y);
	}
	__syncthreads();
}
////////////////////////////////////////////////////////////////////////////////
//Calculate phases from complex
////////////////////////////////////////////////////////////////////////////////
__global__ void c_cc2p_cc(hipfftComplex *g_p, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_p[idx].x = atan2f(g_c[idx].y, g_c[idx].x);
		g_p[idx].y = 0;
	}
	__syncthreads();
}
////////////////////////////////////////////////////////////////////////////////
//Calculate phases from complex
////////////////////////////////////////////////////////////////////////////////
__global__ void c_cc2p_f(float *g_p, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_p[idx] = atan2f(g_c[idx].y, g_c[idx].x);
	}
	__syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
//Copy real part from complex
////////////////////////////////////////////////////////////////////////////////
__global__ void c_cc2re_f(float *g_p, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_p[idx] = g_c[idx].x;
	}
	__syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
//Copy imaginary part from complex
////////////////////////////////////////////////////////////////////////////////
__global__ void c_cc2im_f(float *g_p, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_p[idx] = g_c[idx].y;
	}
	__syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
//reset amplitudes to ones PCR
////////////////////////////////////////////////////////////////////////////////
__global__ void ReplaceAmpsSLM_FFT(float *g_aLaser, hipfftComplex *g_cAmp, float *g_pSLMstart, int N_pixels, float RPC, 
								bool getpSLM255,
								unsigned char *g_pSLM255_uc,
								unsigned char *g_LUT, 
								bool ApplyLUT_b, 
								bool UseAberrationCorr_b, 
								float *g_AberrationCorr_f, 
								bool UseLUTPol_b, 
								float *g_LUTPolCoeff_f, 
								int N_PolCoeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx<N_pixels)
	{
		float aLaser = 1.0f/(float)N_pixels;//g_aLaser[idx];
		
		hipfftComplex cAmp = g_cAmp[idx];
 		float pSLM2pi_f = atan2f(cAmp.y, cAmp.x);

 		if (RPC < (2.0f*M_PI))
		{	
			float pSLMstart = g_pSLMstart[idx];
			if (fabs(pSLM2pi_f - pSLMstart) > RPC)
				pSLM2pi_f = pSLMstart;
		}
		
		if (getpSLM255)
		{
			if (RPC < (2.0f*M_PI))
				g_pSLMstart[idx] = pSLM2pi_f;

			int idxShifted;
			int X_shifted, Y_shifted;
			int data_w = 512; //512!
			//int N_pixels = data_w * data_w;

			//float phase255;
			float N = (float)data_w;
			int half_w = data_w>>1;
			int logN = (int)log2(N);

			int X = idx&(int)(N-1); //works only for data_w = power of 2
			int Y = (idx-X)>>logN;
			float d = 0.001953125;				
			//float X = (idx%data_w);
			//float Y = floor(idx/N); 		
			
			if (X < half_w)
			{	
				X_shifted = X + half_w;
				if (Y < half_w)
				{
					Y_shifted = Y + half_w;
					idxShifted = idx + (data_w * half_w) + half_w;
				}
				else
				{
					Y_shifted = Y - half_w;
					idxShifted = idx - (data_w * half_w) + half_w;
				}
			}
			else
			{
				X_shifted = X - half_w;
				if (Y < half_w)
				{
					Y_shifted = Y + half_w;			
					idxShifted = idx + (data_w * half_w) - half_w;
				}
				else
				{
					Y_shifted = Y - half_w;			
					idxShifted = idx - (data_w * half_w) - half_w;
				}
			}

			if (UseAberrationCorr_b)
				pSLM2pi_f = ApplyAberrationCorrection(pSLM2pi_f, g_AberrationCorr_f[idxShifted]);
			if (ApplyLUT_b)
			{
				int tid = threadIdx.x;
				if (!UseLUTPol_b)
				{
					__shared__ unsigned char s_LUT[256];
					if (tid < 256)
						s_LUT[tid] = g_LUT[tid];
					__syncthreads();
					g_pSLM255_uc[idxShifted] = s_LUT[phase2int32(pSLM2pi_f)];
				}
				else
				{
					__shared__ float s_LUTcoeff[120];
					if (tid < N_PolCoeff)
						s_LUTcoeff[tid] = g_LUTPolCoeff_f[tid];
					__syncthreads();
					g_pSLM255_uc[idxShifted] = applyPolLUT(pSLM2pi_f, d*(float)X_shifted, d*(float)Y_shifted, s_LUTcoeff, N_PolCoeff);
				}
			}
			else
				g_pSLM255_uc[idxShifted] = phase2uc(pSLM2pi_f);
		}
		else
		{
			g_cAmp[idx].x = aLaser*cosf(pSLM2pi_f);
			g_cAmp[idx].y = aLaser*sinf(pSLM2pi_f);
		}
	}
	__syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
//Copy phases in desired spots
////////////////////////////////////////////////////////////////////////////////

__global__ void ReplaceAmpsSpots_FFT(hipfftComplex *g_cSpotAmpObtained, hipfftComplex *g_cSpotAmpDesired, int *g_spotIndex, int N_spots, int iteration, float *g_amplitude, float *g_weight, float amp_desired, bool last_iteration, bool save_amps)
{
	int tid = threadIdx.x;
	int spotIndex;
	float phase;
	float amp_new;
	float amp_obtained;
	float weight_next;
	hipfftComplex cSpotAmp;

	__shared__ float weight_sum[256];
	if ((tid>=N_spots)&&(tid<=(N_spots*2)))
		weight_sum[tid] = 0.0f;

	if (tid < N_spots)
	{
		spotIndex = g_spotIndex[tid];
		cSpotAmp = g_cSpotAmpObtained[spotIndex];
		amp_obtained = hypotf(cSpotAmp.x, cSpotAmp.y);
		phase = atan2f(cSpotAmp.y, cSpotAmp.x);
		if (amp_obtained < 0.0000001f)
		{
			amp_obtained = 1.0f/(float)N_spots;//powf(256,4);
		}
		weight_next = g_weight[N_spots * (iteration) + tid] * sqrtf(amp_desired / amp_obtained);
		weight_sum[tid] = weight_next;
	}

	__syncthreads();	
				
	if (N_spots >= 256) { if (tid < 128) { weight_sum[tid] += weight_sum[tid + 128]; } __syncthreads(); }
	if (N_spots >= 128) { if (tid < 64) { weight_sum[tid] += weight_sum[tid + 64]; } __syncthreads(); }
	volatile float *s_w_sum = weight_sum;
	if (tid < 32) 
	{
		if (N_spots >= 64) s_w_sum[tid] += s_w_sum[tid + 32];
		if (N_spots >= 32) s_w_sum[tid] += s_w_sum[tid + 16];
		if (N_spots >= 16) s_w_sum[tid] += s_w_sum[tid + 8];
		if (N_spots >= 8) s_w_sum[tid] += s_w_sum[tid + 4];
		if (N_spots >= 4) s_w_sum[tid] += s_w_sum[tid + 2];
		if (N_spots >= 2) s_w_sum[tid] += s_w_sum[tid + 1];
	}

	__syncthreads();
	
	if (tid<N_spots)												
	{
		weight_next = weight_next / weight_sum[0];
		amp_new = weight_next * amp_desired;    
		cSpotAmp.x = cosf(phase) * amp_new;
		cSpotAmp.y = sinf(phase) * amp_new;
		g_cSpotAmpDesired[spotIndex] = cSpotAmp;
		g_weight[N_spots * (iteration + 1) + tid] = weight_next;
		if (last_iteration)
			g_weight[tid] = weight_next;
		if (save_amps)
			g_amplitude[N_spots * (iteration) + tid] = amp_obtained;
	}

	__syncthreads();
}
////////////////////////////////////////////////////////////////////////////////
//Convert from coordinates to spot indices
////////////////////////////////////////////////////////////////////////////////

__global__ void XYtoIndex(float *g_x, float *g_y, int *g_spot_index, int N_spots, int data_w)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i < N_spots)
		g_spot_index[i] = ((int)(g_x[i])&(data_w-1))  + ((int)(g_y[i])&(data_w-1))* data_w;
	
	__syncthreads();
}

#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//Copy phases in desired spots
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA_PCI.h"
__global__ void usePhasesW(hipfftComplex *g_cSpotAmpObtained, hipfftComplex *g_cSpotAmpDesired, int *g_spotIndex, int N_spots, int iteration, float *g_amplitude, float *g_weight, float amp_desired)
{
	int tid = threadIdx.x;
	int spotIndex;
	float phase;
	float amp_new;
	float abs;
	float weight_next;
	hipfftComplex cSpotAmp;

	__shared__ float weight_sum[256];

	if (tid < N_spots)
	{
		spotIndex = g_spotIndex[tid];
		cSpotAmp = g_cSpotAmpObtained[spotIndex];
		abs = hypotf(cSpotAmp.x, cSpotAmp.y);
		phase = atan2f(cSpotAmp.y, cSpotAmp.x);
		if (abs < 256)
		{
			abs = powf(256,4);
		}
		weight_next = g_weight[N_spots * (iteration) + tid] * sqrtf(amp_desired / abs);
		weight_sum[tid] = weight_next;
	}

	__syncthreads();	
				
	if (N_spots >= 256) { if (tid < 128) { weight_sum[tid] += weight_sum[tid + 128]; } __syncthreads(); }
	if (N_spots >= 128) { if (tid < 64) { weight_sum[tid] += weight_sum[tid + 64]; } __syncthreads(); }
	if (tid < 32) 
	{
		if (N_spots >= 64) weight_sum[tid] += weight_sum[tid + 32];
		if (N_spots >= 32) weight_sum[tid] += weight_sum[tid + 16];
		if (N_spots >= 16) weight_sum[tid] += weight_sum[tid + 8];
		if (N_spots >= 8) weight_sum[tid] += weight_sum[tid + 4];
		if (N_spots >= 4) weight_sum[tid] += weight_sum[tid + 2];
		if (N_spots >= 2) weight_sum[tid] += weight_sum[tid + 1];
	}

	__syncthreads();
	
	if (tid<N_spots)												
	{
		weight_next = weight_next / weight_sum[0];
		amp_new = weight_next * amp_desired;    
		cSpotAmp.x = cosf(phase) * amp_new;
		cSpotAmp.y = sinf(phase) * amp_new;
		g_cSpotAmpDesired[spotIndex] = cSpotAmp;
		g_weight[N_spots * (iteration + 1) + tid] = weight_next;
		g_amplitude[N_spots * (iteration) + tid] = abs;
	}

	__syncthreads();
}
#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//Calculate Phases and return as float
//
//Possible improvements
//-compute powers of X, Y and phase2pi only once and put in registers
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA_PCI.h"

__global__ void getPhases(unsigned char *g_pSLMuc, float *g_pSLM_start, hipfftComplex *g_cSLMcc, float *g_LUT_coeff, int LUT_on, int data_w)
{	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int s_idx;
	int X_shifted, Y_shifted;
	int N_pixels = data_w * data_w;
	if (idx<N_pixels)
	{
		float phase255;
		float N = data_w;
		int half_w = data_w/2;
		int logN = (int)log2(N);

		int X = idx&(int)(N-1);
 		int Y = (idx-X)>>logN;
 		
 		if (X < half_w)
		{	
			X_shifted = X + half_w;
			if (Y < half_w)
			{
				Y_shifted = Y + half_w;
				s_idx = idx + (data_w * half_w) + half_w;
			}
			else
			{
				Y_shifted = Y - half_w;
				s_idx = idx - (data_w * half_w) + half_w;
			}
		}
		else
		{
			X_shifted = X - half_w;
			if (Y < half_w)
			{
				Y_shifted = Y + half_w;			
				s_idx = idx + (data_w * half_w) - half_w;
			}
			else
			{
				Y_shifted = Y - half_w;			
				s_idx = idx - (data_w * half_w) - half_w;
			}
		}
 		float phase = atan2f(g_cSLMcc[idx].y, g_cSLMcc[idx].x);
		float phase2pi = M_PI + phase;	
		
		if (LUT_on == 1)
		{
			__shared__ float s_c[N_LUT_coeff];
			if (threadIdx.x <N_LUT_coeff)
				s_c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
			__syncthreads();
			
			phase255 = 255.0 - (s_c[0] + s_c[1]*X_shifted + s_c[2]*Y_shifted + s_c[3]*phase2pi + s_c[4]*powf(X_shifted,2) + s_c[5]*X*Y_shifted + s_c[6]*X_shifted*phase2pi + s_c[7]*powf(Y_shifted,2) + s_c[8]*Y_shifted*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X_shifted,3) + s_c[11]*powf(X_shifted,2)*Y_shifted + s_c[12]*powf(X_shifted,2)*phase2pi + s_c[13]*X_shifted*powf(Y_shifted,2) + s_c[14]*X_shifted*Y_shifted*phase2pi + s_c[15]*X_shifted*powf(phase2pi,2) + s_c[16]*powf(Y_shifted,3) + s_c[17]*powf(Y_shifted,2)*phase2pi + s_c[18]*Y_shifted*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X_shifted,4) + s_c[21]*powf(X_shifted,3)*Y_shifted + s_c[22]*powf(X_shifted,3)*phase2pi + s_c[23]*powf(X_shifted,2)*powf(Y_shifted,2) + s_c[24]*powf(X_shifted,2)*Y_shifted*phase2pi + s_c[25]*powf(X_shifted,2)*powf(phase2pi,2) + s_c[26]*X_shifted*powf(Y_shifted,3) + s_c[27]*X_shifted*powf(Y_shifted,2)*phase2pi + s_c[28]*X_shifted*Y_shifted*powf(phase2pi,2) + s_c[29]*X_shifted*powf(phase2pi,3) + s_c[30]*powf(Y_shifted,4) + s_c[31]*powf(Y_shifted,3)*phase2pi + s_c[32]*powf(Y_shifted,2)*powf(phase2pi,2) + s_c[33]*Y_shifted*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X_shifted,5) + s_c[36]*powf(X_shifted,4)*Y_shifted + s_c[37]*powf(X_shifted,4)*phase2pi + s_c[38]*powf(X_shifted,3)*powf(Y_shifted,2) + s_c[39]*powf(X_shifted,3)*Y_shifted*phase2pi + s_c[40]*powf(X_shifted,3)*powf(phase2pi,2) + s_c[41]*powf(X_shifted,2)*powf(Y_shifted,3) + s_c[42]*powf(X_shifted,2)*powf(Y_shifted,2)*phase2pi + s_c[43]*powf(X_shifted,2)*Y_shifted*powf(phase2pi,2) + s_c[44]*powf(X_shifted,2)*powf(phase2pi,3) + s_c[45]*X_shifted*powf(Y_shifted,4) + s_c[46]*X_shifted*powf(Y_shifted,3)*phase2pi + s_c[47]*X_shifted*powf(Y_shifted,2)*powf(phase2pi,2) + s_c[48]*X_shifted*Y_shifted*powf(phase2pi,3) + s_c[49]*X_shifted*powf(phase2pi,4) + s_c[50]*powf(Y_shifted,5) + s_c[51]*powf(Y_shifted,4)*phase2pi + s_c[52]*powf(Y_shifted,3)*powf(phase2pi,2) + s_c[53]*powf(Y_shifted,2)*powf(phase2pi,3) + s_c[54]*Y_shifted*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5) + s_c[56]*powf(X_shifted,6) + s_c[57]*powf(X_shifted,5)*Y_shifted + s_c[58]*powf(X_shifted,5)*phase2pi + s_c[59]*powf(X_shifted,4)*powf(Y_shifted,2) + s_c[60]*powf(X_shifted,4)*Y_shifted*phase2pi + s_c[61]*powf(X_shifted,4)*powf(phase2pi,2) + s_c[62]*powf(X_shifted,3)*powf(Y_shifted,3) + s_c[63]*powf(X_shifted,3)*powf(Y_shifted,2)*phase2pi + s_c[64]*powf(X_shifted,3)*Y_shifted*powf(phase2pi,2) + s_c[65]*powf(X_shifted,3)*powf(phase2pi,3) + s_c[66]*powf(X_shifted,2)*powf(Y_shifted,4) + s_c[67]*powf(X_shifted,2)*powf(Y_shifted,3)*phase2pi + s_c[68]*powf(X_shifted,2)*powf(Y_shifted,2)*powf(phase2pi,2) + s_c[69]*powf(X_shifted,2)*Y_shifted*powf(phase2pi,3) + s_c[70]*powf(X_shifted,2)*powf(phase2pi,4) + s_c[71]*X_shifted*powf(Y_shifted,5) + s_c[72]*X_shifted*powf(Y_shifted,4)*phase2pi + s_c[73]*X_shifted*powf(Y_shifted,3)*powf(phase2pi,2) + s_c[74]*X_shifted*powf(Y_shifted,2)*powf(phase2pi,3) + s_c[75]*X_shifted*Y_shifted*powf(phase2pi,4) + s_c[76]*X_shifted*powf(phase2pi,5) + s_c[77]*powf(Y_shifted,6) + s_c[78]*powf(Y_shifted,5)*phase2pi + s_c[79]*powf(Y_shifted,4)*powf(phase2pi,2) + s_c[80]*powf(Y_shifted,3)*powf(phase2pi,3) + s_c[81]*powf(Y_shifted,2)*powf(phase2pi,4) + s_c[82]*Y_shifted*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6) + s_c[84]*powf(X_shifted,7) + s_c[85]*powf(X_shifted,6)*Y_shifted + s_c[86]*powf(X_shifted,6)*phase2pi + s_c[87]*powf(X_shifted,5)*powf(Y_shifted,2) + s_c[88]*powf(X_shifted,5)*Y_shifted*phase2pi + s_c[89]*powf(X_shifted,5)*powf(phase2pi,2) + s_c[90]*powf(X_shifted,4)*powf(Y_shifted,3) + s_c[91]*powf(X_shifted,4)*powf(Y_shifted,2)*phase2pi + s_c[92]*powf(X_shifted,4)*Y_shifted*powf(phase2pi,2) + s_c[93]*powf(X_shifted,4)*powf(phase2pi,3) + s_c[94]*powf(X_shifted,3)*powf(Y_shifted,4) + s_c[95]*powf(X_shifted,3)*powf(Y_shifted,3)*phase2pi + s_c[96]*powf(X_shifted,3)*powf(Y_shifted,2)*powf(phase2pi,2) + s_c[97]*powf(X_shifted,3)*Y_shifted*powf(phase2pi,3) + s_c[98]*powf(X_shifted,3)*powf(phase2pi,4) + s_c[99]*powf(X_shifted,2)*powf(Y_shifted,5) + s_c[100]*powf(X_shifted,2)*powf(Y_shifted,4)*phase2pi + s_c[101]*powf(X_shifted,2)*powf(Y_shifted,3)*powf(phase2pi,2) + s_c[102]*powf(X_shifted,2)*powf(Y_shifted,2)*powf(phase2pi,3) + s_c[103]*powf(X_shifted,2)*Y_shifted*powf(phase2pi,4) + s_c[104]*powf(X_shifted,2)*powf(phase2pi,5) + s_c[105]*X_shifted*powf(Y_shifted,6) + s_c[106]*X_shifted*powf(Y_shifted,5)*phase2pi + s_c[107]*X_shifted*powf(Y_shifted,4)*powf(phase2pi,2) + s_c[108]*X_shifted*powf(Y_shifted,3)*powf(phase2pi,3) + s_c[109]*X_shifted*powf(Y_shifted,2)*powf(phase2pi,4) + s_c[110]*X_shifted*Y_shifted*powf(phase2pi,5) + s_c[111]*X_shifted*powf(phase2pi,6) + s_c[112]*powf(Y_shifted,7) + s_c[113]*powf(Y_shifted,6)*phase2pi + s_c[114]*powf(Y_shifted,5)*powf(phase2pi,2) + s_c[115]*powf(Y_shifted,4)*powf(phase2pi,3) + s_c[116]*powf(Y_shifted,3)*powf(phase2pi,4) + s_c[117]*powf(Y_shifted,2)*powf(phase2pi,5) + s_c[118]*Y_shifted*powf(phase2pi,6) + s_c[119]*powf(phase2pi,7));
			if (phase255 < 0)
				phase255 = 0;	
		}
		else 
		{
			phase255 = 255.0 * phase2pi / (2.0 * M_PI);
		}
		__syncthreads();
		
		g_pSLM_start[idx] = phase;
		g_pSLMuc[s_idx] = (unsigned char)phase255;
		
		
		
	}
	__syncthreads();
}

/* works but needs to apply LUT after fftshift

////////////////////////////////////////////////////////////////////////////////
//Calculate Phases and return as float
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA_PCI.h"

__global__ void getPhases(unsigned char *g_pSLMuc, float *g_pSLMf, hipfftComplex *g_cSLMcc, float *g_LUT_coeff, int LUT_on, int data_w)
{	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int N_pixels = data_w * data_w;
	if (idx<N_pixels)
	{
		float phase255;
		float N = data_w;
		int half_w = data_w/2;
		int logN = (int)log2(N);

		int X = idx&(int)(N-1);
 		int Y = (idx-X)>>logN;
 		//int Y = (idx&(int)(N_pixels-1)-X)>>logN;

		float phase2pi = M_PI + atan2f(g_cSLMcc[idx].y, g_cSLMcc[idx].x);	
		
		if (LUT_on == 1)
		{
			__shared__ float s_c[N_LUT_coeff];
			if (threadIdx.x <N_LUT_coeff)
				s_c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
			__syncthreads();
			
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5) + s_c[56]*powf(X,6) + s_c[57]*powf(X,5)*Y + s_c[58]*powf(X,5)*phase2pi + s_c[59]*powf(X,4)*powf(Y,2) + s_c[60]*powf(X,4)*Y*phase2pi + s_c[61]*powf(X,4)*powf(phase2pi,2) + s_c[62]*powf(X,3)*powf(Y,3) + s_c[63]*powf(X,3)*powf(Y,2)*phase2pi + s_c[64]*powf(X,3)*Y*powf(phase2pi,2) + s_c[65]*powf(X,3)*powf(phase2pi,3) + s_c[66]*powf(X,2)*powf(Y,4) + s_c[67]*powf(X,2)*powf(Y,3)*phase2pi + s_c[68]*powf(X,2)*powf(Y,2)*powf(phase2pi,2) + s_c[69]*powf(X,2)*Y*powf(phase2pi,3) + s_c[70]*powf(X,2)*powf(phase2pi,4) + s_c[71]*X*powf(Y,5) + s_c[72]*X*powf(Y,4)*phase2pi + s_c[73]*X*powf(Y,3)*powf(phase2pi,2) + s_c[74]*X*powf(Y,2)*powf(phase2pi,3) + s_c[75]*X*Y*powf(phase2pi,4) + s_c[76]*X*powf(phase2pi,5) + s_c[77]*powf(Y,6) + s_c[78]*powf(Y,5)*phase2pi + s_c[79]*powf(Y,4)*powf(phase2pi,2) + s_c[80]*powf(Y,3)*powf(phase2pi,3) + s_c[81]*powf(Y,2)*powf(phase2pi,4) + s_c[82]*Y*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6) + s_c[84]*powf(X,7) + s_c[85]*powf(X,6)*Y + s_c[86]*powf(X,6)*phase2pi + s_c[87]*powf(X,5)*powf(Y,2) + s_c[88]*powf(X,5)*Y*phase2pi + s_c[89]*powf(X,5)*powf(phase2pi,2) + s_c[90]*powf(X,4)*powf(Y,3) + s_c[91]*powf(X,4)*powf(Y,2)*phase2pi + s_c[92]*powf(X,4)*Y*powf(phase2pi,2) + s_c[93]*powf(X,4)*powf(phase2pi,3) + s_c[94]*powf(X,3)*powf(Y,4) + s_c[95]*powf(X,3)*powf(Y,3)*phase2pi + s_c[96]*powf(X,3)*powf(Y,2)*powf(phase2pi,2) + s_c[97]*powf(X,3)*Y*powf(phase2pi,3) + s_c[98]*powf(X,3)*powf(phase2pi,4) + s_c[99]*powf(X,2)*powf(Y,5) + s_c[100]*powf(X,2)*powf(Y,4)*phase2pi + s_c[101]*powf(X,2)*powf(Y,3)*powf(phase2pi,2) + s_c[102]*powf(X,2)*powf(Y,2)*powf(phase2pi,3) + s_c[103]*powf(X,2)*Y*powf(phase2pi,4) + s_c[104]*powf(X,2)*powf(phase2pi,5) + s_c[105]*X*powf(Y,6) + s_c[106]*X*powf(Y,5)*phase2pi + s_c[107]*X*powf(Y,4)*powf(phase2pi,2) + s_c[108]*X*powf(Y,3)*powf(phase2pi,3) + s_c[109]*X*powf(Y,2)*powf(phase2pi,4) + s_c[110]*X*Y*powf(phase2pi,5) + s_c[111]*X*powf(phase2pi,6) + s_c[112]*powf(Y,7) + s_c[113]*powf(Y,6)*phase2pi + s_c[114]*powf(Y,5)*powf(phase2pi,2) + s_c[115]*powf(Y,4)*powf(phase2pi,3) + s_c[116]*powf(Y,3)*powf(phase2pi,4) + s_c[117]*powf(Y,2)*powf(phase2pi,5) + s_c[118]*Y*powf(phase2pi,6) + s_c[119]*powf(phase2pi,7));
			if (phase255 < 0)
				phase255 = 0;	
		}
		else 
		{
			phase255 = 255.0 * phase2pi / (2.0 * M_PI);
		}
		__syncthreads();
		
		if (X < half_w)
		{	
			if (Y < half_w)
			{
				g_pSLMuc[idx + (data_w * half_w) + half_w] = (unsigned char)phase255;
			}
			else
			{
				g_pSLMuc[idx - (data_w * half_w) + half_w] = (unsigned char)phase255;
			}
		}
		else
		{
			if (Y < half_w)
			{
				g_pSLMuc[idx + (data_w * half_w) - half_w] = (unsigned char)phase255;
			}
			else
			{
				g_pSLMuc[idx - (data_w * half_w) - half_w] = (unsigned char)phase255;
			}
		}
		
	}
	__syncthreads();
}*/
#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//Convert float to unsigned char
////////////////////////////////////////////////////////////////////////////////

#include "GenerateHologramCUDA_PCI.h"

__global__ void f2uc(unsigned char *g_uc, float *g_f, int N_pixels, float *g_LUT_coeff, int LUT_on, int data_w)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (LUT_on == 1)
	{	
		float N = data_w;
		int logN = (int)log2(N);
		
		__shared__ float c[N_LUT_coeff];
		if (threadIdx.x <N_LUT_coeff)
			c[threadIdx.x] = g_LUT_coeff[threadIdx.x];
		__syncthreads();
		
		if (idx<N_pixels)
		{
				int X1 = idx&(int)(N-1);
 				int X2 = (idx&(int)(N_pixels-1)-X1)>>logN;  
 				float X3 = g_f[idx] + M_PI;								//pixel phase
 				float fas = 255.0 - (c[0] + c[1]*X1 + c[2]*X2 + c[3]*X3 + c[4]*powf(X1,2) + c[5]*X1*X2 + c[6]*X1*X3 + c[7]*powf(X2,2) + c[8]*X2*X3 + c[9]*powf(X3,2) + c[10]*powf(X1,3) + c[11]*powf(X1,2)*X2 + c[12]*powf(X1,2)*X3 + c[13]*X1*powf(X2,2) + c[14]*X1*X2*X3 + c[15]*X1*powf(X3,2) + c[16]*powf(X2,3) + c[17]*powf(X2,2)*X3 + c[18]*X2*powf(X3,2) + c[19]*powf(X3,3) + c[20]*powf(X1,4) + c[21]*powf(X1,3)*X2 + c[22]*powf(X1,3)*X3 + c[23]*powf(X1,2)*powf(X2,2) + c[24]*powf(X1,2)*X2*X3 + c[25]*powf(X1,2)*powf(X3,2) + c[26]*X1*powf(X2,3) + c[27]*X1*powf(X2,2)*X3 + c[28]*X1*X2*powf(X3,2) + c[29]*X1*powf(X3,3) + c[30]*powf(X2,4) + c[31]*powf(X2,3)*X3 + c[32]*powf(X2,2)*powf(X3,2) + c[33]*X2*powf(X3,3) + c[34]*powf(X3,4) + c[35]*powf(X1,5) + c[36]*powf(X1,4)*X2 + c[37]*powf(X1,4)*X3 + c[38]*powf(X1,3)*powf(X2,2) + c[39]*powf(X1,3)*X2*X3 + c[40]*powf(X1,3)*powf(X3,2) + c[41]*powf(X1,2)*powf(X2,3) + c[42]*powf(X1,2)*powf(X2,2)*X3 + c[43]*powf(X1,2)*X2*powf(X3,2) + c[44]*powf(X1,2)*powf(X3,3) + c[45]*X1*powf(X2,4) + c[46]*X1*powf(X2,3)*X3 + c[47]*X1*powf(X2,2)*powf(X3,2) + c[48]*X1*X2*powf(X3,3) + c[49]*X1*powf(X3,4) + c[50]*powf(X2,5) + c[51]*powf(X2,4)*X3 + c[52]*powf(X2,3)*powf(X3,2) + c[53]*powf(X2,2)*powf(X3,3) + c[54]*X2*powf(X3,4) + c[55]*powf(X3,5) + c[56]*powf(X1,6) + c[57]*powf(X1,5)*X2 + c[58]*powf(X1,5)*X3 + c[59]*powf(X1,4)*powf(X2,2) + c[60]*powf(X1,4)*X2*X3 + c[61]*powf(X1,4)*powf(X3,2) + c[62]*powf(X1,3)*powf(X2,3) + c[63]*powf(X1,3)*powf(X2,2)*X3 + c[64]*powf(X1,3)*X2*powf(X3,2) + c[65]*powf(X1,3)*powf(X3,3) + c[66]*powf(X1,2)*powf(X2,4) + c[67]*powf(X1,2)*powf(X2,3)*X3 + c[68]*powf(X1,2)*powf(X2,2)*powf(X3,2) + c[69]*powf(X1,2)*X2*powf(X3,3) + c[70]*powf(X1,2)*powf(X3,4) + c[71]*X1*powf(X2,5) + c[72]*X1*powf(X2,4)*X3 + c[73]*X1*powf(X2,3)*powf(X3,2) + c[74]*X1*powf(X2,2)*powf(X3,3) + c[75]*X1*X2*powf(X3,4) + c[76]*X1*powf(X3,5) + c[77]*powf(X2,6) + c[78]*powf(X2,5)*X3 + c[79]*powf(X2,4)*powf(X3,2) + c[80]*powf(X2,3)*powf(X3,3) + c[81]*powf(X2,2)*powf(X3,4) + c[82]*X2*powf(X3,5) + c[83]*powf(X3,6) + c[84]*powf(X1,7) + c[85]*powf(X1,6)*X2 + c[86]*powf(X1,6)*X3 + c[87]*powf(X1,5)*powf(X2,2) + c[88]*powf(X1,5)*X2*X3 + c[89]*powf(X1,5)*powf(X3,2) + c[90]*powf(X1,4)*powf(X2,3) + c[91]*powf(X1,4)*powf(X2,2)*X3 + c[92]*powf(X1,4)*X2*powf(X3,2) + c[93]*powf(X1,4)*powf(X3,3) + c[94]*powf(X1,3)*powf(X2,4) + c[95]*powf(X1,3)*powf(X2,3)*X3 + c[96]*powf(X1,3)*powf(X2,2)*powf(X3,2) + c[97]*powf(X1,3)*X2*powf(X3,3) + c[98]*powf(X1,3)*powf(X3,4) + c[99]*powf(X1,2)*powf(X2,5) + c[100]*powf(X1,2)*powf(X2,4)*X3 + c[101]*powf(X1,2)*powf(X2,3)*powf(X3,2) + c[102]*powf(X1,2)*powf(X2,2)*powf(X3,3) + c[103]*powf(X1,2)*X2*powf(X3,4) + c[104]*powf(X1,2)*powf(X3,5) + c[105]*X1*powf(X2,6) + c[106]*X1*powf(X2,5)*X3 + c[107]*X1*powf(X2,4)*powf(X3,2) + c[108]*X1*powf(X2,3)*powf(X3,3) + c[109]*X1*powf(X2,2)*powf(X3,4) + c[110]*X1*X2*powf(X3,5) + c[111]*X1*powf(X3,6) + c[112]*powf(X2,7) + c[113]*powf(X2,6)*X3 + c[114]*powf(X2,5)*powf(X3,2) + c[115]*powf(X2,4)*powf(X3,3) + c[116]*powf(X2,3)*powf(X3,4) + c[117]*powf(X2,2)*powf(X3,5) + c[118]*X2*powf(X3,6) + c[119]*powf(X3,7));
				if (fas >= 0)
					g_uc[idx] = (unsigned char)fas;
				else
					g_uc[idx] = 0;	
				//g_uc[idx] = (unsigned char)(c[0] + c[1]*X1 + c[2]*X2 + c[3]*X3 + c[4]*powf(X1,2) + c[5]*X1*X2 + c[6]*X1*X3 + c[7]*powf(X2,2) + c[8]*X2*X3 + c[9]*powf(X3,2) + c[10]*powf(X1,3) + c[11]*powf(X1,2)*X2 + c[12]*powf(X1,2)*X3 + c[13]*X1*powf(X2,2) + c[14]*X1*X2*X3 + c[15]*X1*powf(X3,2) + c[16]*powf(X2,3) + c[17]*powf(X2,2)*X3 + c[18]*X2*powf(X3,2) + c[19]*powf(X3,3) + c[20]*powf(X1,4) + c[21]*powf(X1,3)*X2 + c[22]*powf(X1,3)*X3 + c[23]*powf(X1,2)*powf(X2,2) + c[24]*powf(X1,2)*X2*X3 + c[25]*powf(X1,2)*powf(X3,2) + c[26]*X1*powf(X2,3) + c[27]*X1*powf(X2,2)*X3 + c[28]*X1*X2*powf(X3,2) + c[29]*X1*powf(X3,3) + c[30]*powf(X2,4) + c[31]*powf(X2,3)*X3 + c[32]*powf(X2,2)*powf(X3,2) + c[33]*X2*powf(X3,3) + c[34]*powf(X3,4) + c[35]*powf(X1,5) + c[36]*powf(X1,4)*X2 + c[37]*powf(X1,4)*X3 + c[38]*powf(X1,3)*powf(X2,2) + c[39]*powf(X1,3)*X2*X3 + c[40]*powf(X1,3)*powf(X3,2) + c[41]*powf(X1,2)*powf(X2,3) + c[42]*powf(X1,2)*powf(X2,2)*X3 + c[43]*powf(X1,2)*X2*powf(X3,2) + c[44]*powf(X1,2)*powf(X3,3) + c[45]*X1*powf(X2,4) + c[46]*X1*powf(X2,3)*X3 + c[47]*X1*powf(X2,2)*powf(X3,2) + c[48]*X1*X2*powf(X3,3) + c[49]*X1*powf(X3,4) + c[50]*powf(X2,5) + c[51]*powf(X2,4)*X3 + c[52]*powf(X2,3)*powf(X3,2) + c[53]*powf(X2,2)*powf(X3,3) + c[54]*X2*powf(X3,4) + c[55]*powf(X3,5) + c[56]*powf(X1,6) + c[57]*powf(X1,5)*X2 + c[58]*powf(X1,5)*X3 + c[59]*powf(X1,4)*powf(X2,2) + c[60]*powf(X1,4)*X2*X3 + c[61]*powf(X1,4)*powf(X3,2) + c[62]*powf(X1,3)*powf(X2,3) + c[63]*powf(X1,3)*powf(X2,2)*X3 + c[64]*powf(X1,3)*X2*powf(X3,2) + c[65]*powf(X1,3)*powf(X3,3) + c[66]*powf(X1,2)*powf(X2,4) + c[67]*powf(X1,2)*powf(X2,3)*X3 + c[68]*powf(X1,2)*powf(X2,2)*powf(X3,2) + c[69]*powf(X1,2)*X2*powf(X3,3) + c[70]*powf(X1,2)*powf(X3,4) + c[71]*X1*powf(X2,5) + c[72]*X1*powf(X2,4)*X3 + c[73]*X1*powf(X2,3)*powf(X3,2) + c[74]*X1*powf(X2,2)*powf(X3,3) + c[75]*X1*X2*powf(X3,4) + c[76]*X1*powf(X3,5) + c[77]*powf(X2,6) + c[78]*powf(X2,5)*X3 + c[79]*powf(X2,4)*powf(X3,2) + c[80]*powf(X2,3)*powf(X3,3) + c[81]*powf(X2,2)*powf(X3,4) + c[82]*X2*powf(X3,5) + c[83]*powf(X3,6) + c[84]*powf(X1,7) + c[85]*powf(X1,6)*X2 + c[86]*powf(X1,6)*X3 + c[87]*powf(X1,5)*powf(X2,2) + c[88]*powf(X1,5)*X2*X3 + c[89]*powf(X1,5)*powf(X3,2) + c[90]*powf(X1,4)*powf(X2,3) + c[91]*powf(X1,4)*powf(X2,2)*X3 + c[92]*powf(X1,4)*X2*powf(X3,2) + c[93]*powf(X1,4)*powf(X3,3) + c[94]*powf(X1,3)*powf(X2,4) + c[95]*powf(X1,3)*powf(X2,3)*X3 + c[96]*powf(X1,3)*powf(X2,2)*powf(X3,2) + c[97]*powf(X1,3)*X2*powf(X3,3) + c[98]*powf(X1,3)*powf(X3,4) + c[99]*powf(X1,2)*powf(X2,5) + c[100]*powf(X1,2)*powf(X2,4)*X3 + c[101]*powf(X1,2)*powf(X2,3)*powf(X3,2) + c[102]*powf(X1,2)*powf(X2,2)*powf(X3,3) + c[103]*powf(X1,2)*X2*powf(X3,4) + c[104]*powf(X1,2)*powf(X3,5) + c[105]*X1*powf(X2,6) + c[106]*X1*powf(X2,5)*X3 + c[107]*X1*powf(X2,4)*powf(X3,2) + c[108]*X1*powf(X2,3)*powf(X3,3) + c[109]*X1*powf(X2,2)*powf(X3,4) + c[110]*X1*X2*powf(X3,5) + c[111]*X1*powf(X3,6) + c[112]*powf(X2,7) + c[113]*powf(X2,6)*X3 + c[114]*powf(X2,5)*powf(X3,2) + c[115]*powf(X2,4)*powf(X3,3) + c[116]*powf(X2,3)*powf(X3,4) + c[117]*powf(X2,2)*powf(X3,5) + c[118]*X2*powf(X3,6) + c[119]*powf(X3,7));	
		}
	}
	else
	{
		if (idx<N_pixels)
		{
			g_uc[idx] = (unsigned char)(((g_f[idx] + M_PI)*255.0/(2.0*M_PI)));
		}	
	}
	__syncthreads();
}


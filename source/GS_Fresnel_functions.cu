#include "hip/hip_runtime.h"
/*   
   Hologram generating algorithms for CUDA Devices
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se


   This file is part of GenerateHologramCUDA.

    GenerateHologramCUDA is free software: you can redistribute it and/or 
    modify it under the terms of the GNU Lesser General Public License as published 
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GenerateHologramCUDA is distributed in the hope that it will be 
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Lesser General Public License for more details.

    You should have received a copy of the GNU Lesser General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "GenerateHologramCUDA.h"
////////////////////////////////////////////////////////////////////////////////////
//Common functions
///////////////////////////////////////////////////////////////////////////////////
__device__ unsigned char phase2uc(float phase2pi)
{
	return (unsigned char)floor((phase2pi + M_PI)*256.0f / (2.0f * M_PI));
}
__device__ int phase2int32(float phase2pi)
{
	return (int)floor((phase2pi + M_PI)*256.0f / (2.0f * M_PI));
}
__device__ float ApplyAberrationCorrection(float phase, float correction)
{
		phase = phase + correction;		//apply correction
		return (phase - (2.0*M_PI) * floor((phase+M_PI) / (2.0*M_PI))); //apply mod([-pi, pi], phase) 
}
/*__device__ unsigned char applyPolLUT(float phase2pi, int X, int Y, float *s_c, int N_PolCoeff)		
{
	float phase255 = 0;
	switch (N_PolCoeff)	{
		case 120:
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5) + s_c[56]*powf(X,6) + s_c[57]*powf(X,5)*Y + s_c[58]*powf(X,5)*phase2pi + s_c[59]*powf(X,4)*powf(Y,2) + s_c[60]*powf(X,4)*Y*phase2pi + s_c[61]*powf(X,4)*powf(phase2pi,2) + s_c[62]*powf(X,3)*powf(Y,3) + s_c[63]*powf(X,3)*powf(Y,2)*phase2pi + s_c[64]*powf(X,3)*Y*powf(phase2pi,2) + s_c[65]*powf(X,3)*powf(phase2pi,3) + s_c[66]*powf(X,2)*powf(Y,4) + s_c[67]*powf(X,2)*powf(Y,3)*phase2pi + s_c[68]*powf(X,2)*powf(Y,2)*powf(phase2pi,2) + s_c[69]*powf(X,2)*Y*powf(phase2pi,3) + s_c[70]*powf(X,2)*powf(phase2pi,4) + s_c[71]*X*powf(Y,5) + s_c[72]*X*powf(Y,4)*phase2pi + s_c[73]*X*powf(Y,3)*powf(phase2pi,2) + s_c[74]*X*powf(Y,2)*powf(phase2pi,3) + s_c[75]*X*Y*powf(phase2pi,4) + s_c[76]*X*powf(phase2pi,5) + s_c[77]*powf(Y,6) + s_c[78]*powf(Y,5)*phase2pi + s_c[79]*powf(Y,4)*powf(phase2pi,2) + s_c[80]*powf(Y,3)*powf(phase2pi,3) + s_c[81]*powf(Y,2)*powf(phase2pi,4) + s_c[82]*Y*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6) + s_c[84]*powf(X,7) + s_c[85]*powf(X,6)*Y + s_c[86]*powf(X,6)*phase2pi + s_c[87]*powf(X,5)*powf(Y,2) + s_c[88]*powf(X,5)*Y*phase2pi + s_c[89]*powf(X,5)*powf(phase2pi,2) + s_c[90]*powf(X,4)*powf(Y,3) + s_c[91]*powf(X,4)*powf(Y,2)*phase2pi + s_c[92]*powf(X,4)*Y*powf(phase2pi,2) + s_c[93]*powf(X,4)*powf(phase2pi,3) + s_c[94]*powf(X,3)*powf(Y,4) + s_c[95]*powf(X,3)*powf(Y,3)*phase2pi + s_c[96]*powf(X,3)*powf(Y,2)*powf(phase2pi,2) + s_c[97]*powf(X,3)*Y*powf(phase2pi,3) + s_c[98]*powf(X,3)*powf(phase2pi,4) + s_c[99]*powf(X,2)*powf(Y,5) + s_c[100]*powf(X,2)*powf(Y,4)*phase2pi + s_c[101]*powf(X,2)*powf(Y,3)*powf(phase2pi,2) + s_c[102]*powf(X,2)*powf(Y,2)*powf(phase2pi,3) + s_c[103]*powf(X,2)*Y*powf(phase2pi,4) + s_c[104]*powf(X,2)*powf(phase2pi,5) + s_c[105]*X*powf(Y,6) + s_c[106]*X*powf(Y,5)*phase2pi + s_c[107]*X*powf(Y,4)*powf(phase2pi,2) + s_c[108]*X*powf(Y,3)*powf(phase2pi,3) + s_c[109]*X*powf(Y,2)*powf(phase2pi,4) + s_c[110]*X*Y*powf(phase2pi,5) + s_c[111]*X*powf(phase2pi,6) + s_c[112]*powf(Y,7) + s_c[113]*powf(Y,6)*phase2pi + s_c[114]*powf(Y,5)*powf(phase2pi,2) + s_c[115]*powf(Y,4)*powf(phase2pi,3) + s_c[116]*powf(Y,3)*powf(phase2pi,4) + s_c[117]*powf(Y,2)*powf(phase2pi,5) + s_c[118]*Y*powf(phase2pi,6) + s_c[119]*powf(phase2pi,7));
			break;
		case 84:
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5) + s_c[56]*powf(X,6) + s_c[57]*powf(X,5)*Y + s_c[58]*powf(X,5)*phase2pi + s_c[59]*powf(X,4)*powf(Y,2) + s_c[60]*powf(X,4)*Y*phase2pi + s_c[61]*powf(X,4)*powf(phase2pi,2) + s_c[62]*powf(X,3)*powf(Y,3) + s_c[63]*powf(X,3)*powf(Y,2)*phase2pi + s_c[64]*powf(X,3)*Y*powf(phase2pi,2) + s_c[65]*powf(X,3)*powf(phase2pi,3) + s_c[66]*powf(X,2)*powf(Y,4) + s_c[67]*powf(X,2)*powf(Y,3)*phase2pi + s_c[68]*powf(X,2)*powf(Y,2)*powf(phase2pi,2) + s_c[69]*powf(X,2)*Y*powf(phase2pi,3) + s_c[70]*powf(X,2)*powf(phase2pi,4) + s_c[71]*X*powf(Y,5) + s_c[72]*X*powf(Y,4)*phase2pi + s_c[73]*X*powf(Y,3)*powf(phase2pi,2) + s_c[74]*X*powf(Y,2)*powf(phase2pi,3) + s_c[75]*X*Y*powf(phase2pi,4) + s_c[76]*X*powf(phase2pi,5) + s_c[77]*powf(Y,6) + s_c[78]*powf(Y,5)*phase2pi + s_c[79]*powf(Y,4)*powf(phase2pi,2) + s_c[80]*powf(Y,3)*powf(phase2pi,3) + s_c[81]*powf(Y,2)*powf(phase2pi,4) + s_c[82]*Y*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6));
			break;
		case 56:
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4) + s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5));
			break;
		case 35:
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3) + s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4));
			break;
		case 20:
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3));
			break;
		default:
			phase255 = 0;
			break;
	}
	if (phase255 < 0)
		phase255 = 0;	
	return (unsigned char)phase255;
}*/
__device__ unsigned char applyPolLUT(float phase2pi, int X, int Y, float *s_c, int N_PolCoeff)		
{
	float phase255 = 255.0f;
	switch (N_PolCoeff)	{
		case 120:
			phase255 = phase255 -(s_c[84]*powf(X,7) + s_c[85]*powf(X,6)*Y + s_c[86]*powf(X,6)*phase2pi + s_c[87]*powf(X,5)*powf(Y,2) + s_c[88]*powf(X,5)*Y*phase2pi + s_c[89]*powf(X,5)*powf(phase2pi,2) + s_c[90]*powf(X,4)*powf(Y,3) + s_c[91]*powf(X,4)*powf(Y,2)*phase2pi + s_c[92]*powf(X,4)*Y*powf(phase2pi,2) + s_c[93]*powf(X,4)*powf(phase2pi,3) + s_c[94]*powf(X,3)*powf(Y,4) + s_c[95]*powf(X,3)*powf(Y,3)*phase2pi + s_c[96]*powf(X,3)*powf(Y,2)*powf(phase2pi,2) + s_c[97]*powf(X,3)*Y*powf(phase2pi,3) + s_c[98]*powf(X,3)*powf(phase2pi,4) + s_c[99]*powf(X,2)*powf(Y,5) + s_c[100]*powf(X,2)*powf(Y,4)*phase2pi + s_c[101]*powf(X,2)*powf(Y,3)*powf(phase2pi,2) + s_c[102]*powf(X,2)*powf(Y,2)*powf(phase2pi,3) + s_c[103]*powf(X,2)*Y*powf(phase2pi,4) + s_c[104]*powf(X,2)*powf(phase2pi,5) + s_c[105]*X*powf(Y,6) + s_c[106]*X*powf(Y,5)*phase2pi + s_c[107]*X*powf(Y,4)*powf(phase2pi,2) + s_c[108]*X*powf(Y,3)*powf(phase2pi,3) + s_c[109]*X*powf(Y,2)*powf(phase2pi,4) + s_c[110]*X*Y*powf(phase2pi,5) + s_c[111]*X*powf(phase2pi,6) + s_c[112]*powf(Y,7) + s_c[113]*powf(Y,6)*phase2pi + s_c[114]*powf(Y,5)*powf(phase2pi,2) + s_c[115]*powf(Y,4)*powf(phase2pi,3) + s_c[116]*powf(Y,3)*powf(phase2pi,4) + s_c[117]*powf(Y,2)*powf(phase2pi,5) + s_c[118]*Y*powf(phase2pi,6) + s_c[119]*powf(phase2pi,7));
		case 84:
			phase255 = phase255 -(s_c[56]*powf(X,6) + s_c[57]*powf(X,5)*Y + s_c[58]*powf(X,5)*phase2pi + s_c[59]*powf(X,4)*powf(Y,2) + s_c[60]*powf(X,4)*Y*phase2pi + s_c[61]*powf(X,4)*powf(phase2pi,2) + s_c[62]*powf(X,3)*powf(Y,3) + s_c[63]*powf(X,3)*powf(Y,2)*phase2pi + s_c[64]*powf(X,3)*Y*powf(phase2pi,2) + s_c[65]*powf(X,3)*powf(phase2pi,3) + s_c[66]*powf(X,2)*powf(Y,4) + s_c[67]*powf(X,2)*powf(Y,3)*phase2pi + s_c[68]*powf(X,2)*powf(Y,2)*powf(phase2pi,2) + s_c[69]*powf(X,2)*Y*powf(phase2pi,3) + s_c[70]*powf(X,2)*powf(phase2pi,4) + s_c[71]*X*powf(Y,5) + s_c[72]*X*powf(Y,4)*phase2pi + s_c[73]*X*powf(Y,3)*powf(phase2pi,2) + s_c[74]*X*powf(Y,2)*powf(phase2pi,3) + s_c[75]*X*Y*powf(phase2pi,4) + s_c[76]*X*powf(phase2pi,5) + s_c[77]*powf(Y,6) + s_c[78]*powf(Y,5)*phase2pi + s_c[79]*powf(Y,4)*powf(phase2pi,2) + s_c[80]*powf(Y,3)*powf(phase2pi,3) + s_c[81]*powf(Y,2)*powf(phase2pi,4) + s_c[82]*Y*powf(phase2pi,5) + s_c[83]*powf(phase2pi,6));
		case 56:
			phase255 = phase255 -(s_c[35]*powf(X,5) + s_c[36]*powf(X,4)*Y + s_c[37]*powf(X,4)*phase2pi + s_c[38]*powf(X,3)*powf(Y,2) + s_c[39]*powf(X,3)*Y*phase2pi + s_c[40]*powf(X,3)*powf(phase2pi,2) + s_c[41]*powf(X,2)*powf(Y,3) + s_c[42]*powf(X,2)*powf(Y,2)*phase2pi + s_c[43]*powf(X,2)*Y*powf(phase2pi,2) + s_c[44]*powf(X,2)*powf(phase2pi,3) + s_c[45]*X*powf(Y,4) + s_c[46]*X*powf(Y,3)*phase2pi + s_c[47]*X*powf(Y,2)*powf(phase2pi,2) + s_c[48]*X*Y*powf(phase2pi,3) + s_c[49]*X*powf(phase2pi,4) + s_c[50]*powf(Y,5) + s_c[51]*powf(Y,4)*phase2pi + s_c[52]*powf(Y,3)*powf(phase2pi,2) + s_c[53]*powf(Y,2)*powf(phase2pi,3) + s_c[54]*Y*powf(phase2pi,4) + s_c[55]*powf(phase2pi,5));
		case 35:
			phase255 = phase255 -(s_c[20]*powf(X,4) + s_c[21]*powf(X,3)*Y + s_c[22]*powf(X,3)*phase2pi + s_c[23]*powf(X,2)*powf(Y,2) + s_c[24]*powf(X,2)*Y*phase2pi + s_c[25]*powf(X,2)*powf(phase2pi,2) + s_c[26]*X*powf(Y,3) + s_c[27]*X*powf(Y,2)*phase2pi + s_c[28]*X*Y*powf(phase2pi,2) + s_c[29]*X*powf(phase2pi,3) + s_c[30]*powf(Y,4) + s_c[31]*powf(Y,3)*phase2pi + s_c[32]*powf(Y,2)*powf(phase2pi,2) + s_c[33]*Y*powf(phase2pi,3) + s_c[34]*powf(phase2pi,4));
		case 20:
			phase255 = 255.0 - (s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*powf(X,2) + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*powf(Y,2) + s_c[8]*Y*phase2pi + s_c[9]*powf(phase2pi,2) + s_c[10]*powf(X,3) + s_c[11]*powf(X,2)*Y + s_c[12]*powf(X,2)*phase2pi + s_c[13]*X*powf(Y,2) + s_c[14]*X*Y*phase2pi + s_c[15]*X*powf(phase2pi,2) + s_c[16]*powf(Y,3) + s_c[17]*powf(Y,2)*phase2pi + s_c[18]*Y*powf(phase2pi,2) + s_c[19]*powf(phase2pi,3));
			break;
		default:
			phase255 = 0;
			break;
	}
	if (phase255 < 0)
		phase255 = 0;
	return (unsigned char)phase255;
}
__device__ void warpReduceC(volatile float *s_Vre, volatile float *s_Vim, int tid)
{
	s_Vre[tid] += s_Vre[tid + 32];
	s_Vim[tid] += s_Vim[tid + 32];

	s_Vre[tid] += s_Vre[tid + 16];
	s_Vim[tid] += s_Vim[tid + 16];

	s_Vre[tid] += s_Vre[tid + 8];
	s_Vim[tid] += s_Vim[tid + 8];

	s_Vre[tid] += s_Vre[tid + 4];
	s_Vim[tid] += s_Vim[tid + 4];

	s_Vre[tid] += s_Vre[tid + 2];
	s_Vim[tid] += s_Vim[tid + 2];

	s_Vre[tid] += s_Vre[tid + 1];
	s_Vim[tid] += s_Vim[tid + 1];
}
/////////////////////////////////////////////////////////////////////////////////////////////////
//Calculate hologram using "Lenses and Prisms"
/////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void LensesAndPrisms(float *g_x, float *g_y, float *g_z, float *g_I, unsigned char *g_SLMuc, int N_spots, unsigned char *g_LUT, bool ApplyLUT_b, int data_w, bool UseAberrationCorr_b, float *d_AberrationCorr_f, bool UseLUTPol_b, float *d_LUTPolCoeff_f, int N_PolCoeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	
	__shared__ float s_x[BLOCK_SIZE];
	__shared__ float s_y[BLOCK_SIZE];
	__shared__ float s_z[BLOCK_SIZE];
	__shared__ float s_a[BLOCK_SIZE];
	
	if (tid < N_spots)
	{
		s_x[tid] = g_x[tid];
		s_y[tid] = g_y[tid];
		s_z[tid] = g_z[tid];
		s_a[tid] = sqrtf(g_I[tid]);
	}
	__syncthreads();	
	
	if (idx < data_w*data_w)
	{
		//get pixel coordinates 
		float N = data_w;
		int logN = (int)log2(N);
		int X_int = idx&(int)(N-1);
	 	//int Y_int = (idx&(int)(N*N-1)-X_int)>>logN;
	 	int Y_int = (idx-X_int)>>logN;
	 	float X = (float)(X_int - (data_w>>1)) / N; //does not work for 768 pixels
 		float Y = (float)(Y_int - (data_w>>1)) / N;
		
		float phase2pi;  // [-pi,pi]
		float SLMre = 0;
		float SLMim = 0;
				
		for (int ii=0; ii<N_spots; ++ii)
		{
			phase2pi = M_PI * s_z[ii] * (X*X + Y*Y) + 2.0 * M_PI * (X * (s_x[ii]) + Y * (s_y[ii]) );
			SLMre = SLMre + s_a[ii] * cosf(phase2pi);
			SLMim = SLMim + s_a[ii] * sinf(phase2pi); 
		}
		phase2pi = atan2f(SLMim, SLMre);	
		
		if (UseAberrationCorr_b)
			phase2pi = ApplyAberrationCorrection(phase2pi, d_AberrationCorr_f[idx]);

		if (ApplyLUT_b) 
		{
			if (!UseLUTPol_b)
			{
				__shared__ unsigned char s_LUT[256];
				if (tid < 256)
					s_LUT[tid] = g_LUT[tid];
				__syncthreads();
				g_SLMuc[idx] = s_LUT[phase2int32(phase2pi)];
			}
			else
			{
				__shared__ float s_LUTcoeff[120];
				if (tid < N_PolCoeff)
					s_LUTcoeff[tid] = d_LUTPolCoeff_f[tid];
				__syncthreads();
				g_SLMuc[idx] = applyPolLUT(phase2pi, X, Y, s_LUTcoeff, N_PolCoeff);
			}
		}
		else
			g_SLMuc[idx] = phase2uc(phase2pi);
	}	
	__syncthreads();

}

__global__ void checkAmplitudes(float *g_x, float *g_y, float *g_z, unsigned char *g_pSLM_uc, float *g_amps, int N_spots, int N_pixels, int data_w)
{
	int blockSize = 512;
	int spot_number = blockIdx.x;
	int tid = threadIdx.x;
	int i = tid;
	
	__shared__ float s_Vre[BLOCK_SIZE];
	__shared__ float s_Vim[BLOCK_SIZE];
	__shared__ float s_xm, s_ym, s_zm;

	s_Vre[tid] = 0;
	s_Vim[tid] = 0;
		
	float N = data_w;
	int logN = (int)log2(N);
	float d = 0.001953125;	//SLM pixel size (1/512)	512!
	
	if (tid == 0)
		s_xm = g_x[spot_number];
	if (tid == 64)
		s_ym = g_y[spot_number];	
	if (tid == 128)
		s_zm = g_z[spot_number];
	
	float X1 = d * ((float)tid - 256.0);				//512!
	float Y1 = - d * 256.0;	
	__syncthreads();
		
	while (i < N_pixels) 
	{ 
		float pSLM_1 = 2*M_PI*(float)g_pSLM_uc[i] - M_PI;
		float p = pSLM_1 - M_PI * (s_zm * (X1*X1 + Y1*Y1) + 2 * (X1 * s_xm + Y1 * s_ym));
		
		s_Vre[tid] += cosf(p);
		s_Vim[tid] += sinf(p);

		i += blockSize;
		Y1 += d; 	
	}
	__syncthreads();
 
	if (tid < 256) 
	{ 
		s_Vre[tid] += s_Vre[tid + 256]; 
		s_Vim[tid] += s_Vim[tid + 256];
	} 
	__syncthreads(); 

	if (tid < 128)
	{ 
		s_Vre[tid] += s_Vre[tid + 128];
		s_Vim[tid] += s_Vim[tid + 128];  
	} 
	__syncthreads(); 

	if (tid < 64) 
	{ 
		s_Vre[tid] += s_Vre[tid + 64];
		s_Vim[tid] += s_Vim[tid + 64]; 
	} 
	__syncthreads(); 

	
	if (tid < 32) {

		s_Vre[tid] += s_Vre[tid + 32];
		s_Vim[tid] += s_Vim[tid + 32];

		s_Vre[tid] += s_Vre[tid + 16];
		s_Vim[tid] += s_Vim[tid + 16];

		s_Vre[tid] += s_Vre[tid + 8];
		s_Vim[tid] += s_Vim[tid + 8];

		s_Vre[tid] += s_Vre[tid + 4];
		s_Vim[tid] += s_Vim[tid + 4];

		s_Vre[tid] += s_Vre[tid + 2];
		s_Vim[tid] += s_Vim[tid + 2];

		s_Vre[tid] += s_Vre[tid + 1];
		s_Vim[tid] += s_Vim[tid + 1];
		
	}
	if (tid == 0) 
	{
		float Vre = s_Vre[0] / 262144.0;			//512!
		float Vim = s_Vim[0] / 262144.0;
		g_amps[spot_number] = hypotf(Vim, Vre);
	}
}
////////////////////////////////////////////////////////////////////////////////
//Functions for GS with Fresnel propagation
////////////////////////////////////////////////////////////////////////////////
//Propagate from the SLM to the spot positions using Fresnel summation
//(Works for 512x512 pixels only!)
////////////////////////////////////////////////////////////////////////////////
__global__ void PropagateToSpotPositions_Fresnel(float *g_x, float *g_y, float *g_z, float *g_pSLM2pi, float *g_Vre, float *g_Vim, int N_spots, int n, int data_w)
{
	int blockSize = 512;
	int spot_number = blockIdx.x;
	int tid = threadIdx.x;
	int i = tid;
	
	__shared__ float s_Vre[BLOCK_SIZE];
	__shared__ float s_Vim[BLOCK_SIZE];
	__shared__ float s_xm, s_ym, s_zm;

	s_Vre[tid] = 0;
	s_Vim[tid] = 0;
		
	float N = data_w;
	int logN = (int)log2(N);
	float d = 0.001953125;	//Normalized pixel pitch (1/512)
	
	if (tid == 0)
		s_xm = g_x[spot_number];
	if (tid == 64)
		s_ym = g_y[spot_number];	
	if (tid == 128)
		s_zm = g_z[spot_number];
	
	float X1 = d * ((float)tid - 256.0);
	float Y1 = - d * 256.0;	
	__syncthreads();
		
	while (i < n) 
	{ 
		float pSLM_1 = g_pSLM2pi[i];
		float p = pSLM_1 - M_PI * (s_zm * (X1*X1 + Y1*Y1) + 2 * (X1 * s_xm + Y1 * s_ym));
		
		s_Vre[tid] += cosf(p);
		s_Vim[tid] += sinf(p);

		i += blockSize;
		Y1 += d; 	
	}
	__syncthreads();
 
	if (tid < 256) 
	{ 
		s_Vre[tid] += s_Vre[tid + 256]; 
		s_Vim[tid] += s_Vim[tid + 256];
	} 
	__syncthreads(); 

	if (tid < 128)
	{ 
		s_Vre[tid] += s_Vre[tid + 128];
		s_Vim[tid] += s_Vim[tid + 128];  
	} 
	__syncthreads(); 

	if (tid < 64) 
	{ 
		s_Vre[tid] += s_Vre[tid + 64];
		s_Vim[tid] += s_Vim[tid + 64]; 
	} 
	__syncthreads(); 

	
	if (tid < 32)
		warpReduceC(s_Vre, s_Vim, tid);

	if (tid == 0) 
	{
		g_Vre[spot_number] = s_Vre[0] / 262144.0;
		g_Vim[spot_number] = s_Vim[0] / 262144.0;
	}
}


////////////////////////////////////////////////////////////////////////////////
//Obtain phases in SLM plane
////////////////////////////////////////////////////////////////////////////////
//works only for blocksize 512 and max 256 spots
////////////////////////////////////////////////////////////////////////////////
__global__ void PropagateToSLM_Fresnel(float *g_x, 
								float *g_y, 
								float *g_z, 
								float *g_I, 
								float *g_SpotsRe, 
								float *g_SpotsIm, 
								float *g_pSLM2pi, 
								int N_pixels, 
								int N_spots, 
								float *g_weights, 
								int iteration, 
								float *g_pSLMstart, 
								float RPC, 
								float *g_amps,
								bool getpSLM255,
								unsigned char *g_pSLM255_uc,
								unsigned char *g_LUT, 
								bool ApplyLUT_b, 
								bool UseAberrationCorr_b, 
								float *g_AberrationCorr_f, 
								bool UseLUTPol_b, 
								float *g_LUTPolCoeff_f, 
								int N_PolCoeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;	
	__shared__ float s_aSpot[BLOCK_SIZE], s_a_mean, s_weight[BLOCK_SIZE], s_pSpot[BLOCK_SIZE];
	__shared__ float s_xm[BLOCK_SIZE];
	__shared__ float s_ym[BLOCK_SIZE];
	__shared__ float s_zm[BLOCK_SIZE];
	float reSLM = 0, imSLM = 0, pSLM2pi_f = 0;

	if (idx<N_pixels)
	{
		//float N = 512;
		//int logN = (int)log2(N);
		float d = 0.001953125;		//Normalized pixel pitch (1/512)
		
		//load data to shared memory
		if (N_spots <= 64)
		{
			if (tid < N_spots)
			{
				float Vre = g_SpotsRe[tid];
				float Vim = g_SpotsIm[tid];
				if (iteration == 0)
					s_aSpot[tid] = 1/sqrtf(g_I[tid]);
				else
					s_aSpot[tid] = hypotf(Vim, Vre)/sqrtf(g_I[tid]);		//divide by the desired amplitude for spot m
				//s_aSpot_sum[tid] = s_aSpot[tid];
				s_pSpot[tid] = atan2f(Vim, Vre);
			}	
			else if ((tid - 64) < N_spots)
				s_weight[tid - 64] = g_weights[tid - 64 + iteration*N_spots];
			else if ((tid - 128) < N_spots)
				s_xm[tid - 128] = g_x[tid - 128];
			else if ((tid - 192) < N_spots)
				s_ym[tid - 192] = g_y[tid - 192];
			else if ((tid - 256) < N_spots)
				s_zm[tid - 256] = g_z[tid - 256];																	
		}
		else
		{	
			if (tid<N_spots)
			{
				float Vre = g_SpotsRe[tid];
				float Vim = g_SpotsIm[tid];
				s_pSpot[tid] = atan2f(Vim, Vre);
				if (iteration == 0)
					s_aSpot[tid] = 1/sqrtf(g_I[tid]);
				else
					s_aSpot[tid] = hypotf(Vim, Vre)/sqrtf(g_I[tid]);		//divide by the desired amplitude for spot m
				//s_aSpot_sum[tid] = s_aSpot[tid];
				s_weight[tid] = g_weights[tid + iteration*N_spots];
				s_xm[tid] = g_x[tid];
				s_ym[tid] = g_y[tid];
				s_zm[tid] = g_z[tid];
				
			}
		}	
		__syncthreads();		

		//compute weights 
		if  (tid==0)
		{
			float s_aSpot_sum = 0;
			for (int jj=0; jj<N_spots;jj++)
			{	
				s_aSpot_sum += s_aSpot[jj];		
			}
			s_a_mean = s_aSpot_sum / (float)N_spots; //integer division!!
		}
		__syncthreads();
	
		if (tid<N_spots)
		{
			s_weight[tid] = s_weight[tid] * s_a_mean / s_aSpot[tid];	
			g_weights[tid + N_spots*(iteration+1)] = s_weight[tid];
			g_amps[tid + N_spots*iteration] = s_aSpot[tid];		//may be excluded, used for monitoring only
		}
		__syncthreads();				
		//get pixel coordinates (change this to allow data_w!=512) 
 		float X = d * (((float)(threadIdx.x) - 256.0f));
 		float Y = d * (((float)(blockIdx.x) - 256.0f));

		//compute SLM phase by summing contribution from all spots
		for (int k=0; k<N_spots; k++)
		{
			float delta = M_PI * s_zm[k] * (X*X + Y*Y) + 2.0 * M_PI * (X * s_xm[k] + Y * s_ym[k]);
			reSLM += s_weight[k] * cosf(s_pSpot[k] + delta);
			imSLM += s_weight[k] * sinf(s_pSpot[k] + delta);
		}
		pSLM2pi_f = atan2f(imSLM, reSLM);		
		
	
		if (RPC < (2.0f*M_PI))			//Apply RPC (restricted Phase Change)
		{	
			float pSLMstart = g_pSLMstart[idx];
			if (fabs(pSLM2pi_f - pSLMstart) > RPC)
				pSLM2pi_f = pSLMstart;
			if (getpSLM255)
				g_pSLMstart[idx] = pSLM2pi_f;
		}		

		if (getpSLM255)					//Compute final SLM phases and write to global memory... 
		{								
			if (UseAberrationCorr_b)
				pSLM2pi_f = ApplyAberrationCorrection(pSLM2pi_f, g_AberrationCorr_f[idx]);
			if (ApplyLUT_b)
			{
				if (!UseLUTPol_b)
				{
					__shared__ unsigned char s_LUT[256];
					if (tid < 256)
						s_LUT[tid] = g_LUT[tid];
					__syncthreads();
					g_pSLM255_uc[idx] = s_LUT[phase2int32(pSLM2pi_f)];
				}
				else
				{
					__shared__ float s_LUTcoeff[120];
					if (tid < N_PolCoeff)
						s_LUTcoeff[tid] = g_LUTPolCoeff_f[tid];
					__syncthreads();
					g_pSLM255_uc[idx] = applyPolLUT(pSLM2pi_f, X, Y, s_LUTcoeff, N_PolCoeff);
				}
			}
			else
				g_pSLM255_uc[idx] = phase2uc(pSLM2pi_f);
		}
		else
		{
			g_pSLM2pi[idx] = pSLM2pi_f;	//...or write intermediate phase to global memory
		}
	}
}

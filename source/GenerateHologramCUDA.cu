#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
/*   
   Hologram generating algorithms for CUDA Devices
   
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se

   This file is part of GenerateHologramCUDA.

   GenerateHologramCUDA is free software: you can redistribute it and/or 
   modify it under the terms of the GNU Lesser General Public License as published 
   by the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   GenerateHologramCUDA is distributed in the hope that it will be 
   useful, but WITHOUT ANY WARRANTY; without even the implied warranty
   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU Lesser General Public License for more details.

   You should have received a copy of the GNU Lesser General Public License
   along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
///////////////////////////////////////////////////////////////////////////////////
//The function "GenerateHologram" contains two different algorithms for
//hologram generation. The last parameter in the function call selects which 
//one to use:
//0: Complex addition of "Lenses and Prisms", no optimization (3D)
//1: Weighted Gerchberg-Saxton algorithm using Fresnel propagation (3D)
//
//-(1) Is implemented in two different ways, one that uses precalculated values 
//     for the phase difference between each pixel and each spot (a), and one that
//     calculates the values in each iteration (b). (a) is faster for up 
//     to 4 spots. The selection of the two implementation is automatic and based
//	   based on the number of spots. 
//-(0) produces optimal holograms for 1 or 2 traps and is significantly faster.
//     (0) is automatically selected if the number of spots is < 3. 
////////////////////////////////////////////////////////////////////////////////
//Fresnel propagation based algorithm (1 and 3) described in:
//Roberto Di Leonardo, Francesca Ianni, and Giancarlo Ruocco
//"Computer generation of optimal holograms for optical trap arrays"
//Opt. Express 15, 1913-1922 (2007) 
//
//The original algorithm has been modified to allow variable spot amplitudes
////////////////////////////////////////////////////////////////////////////////
//Possible improvements:
//-Put all arguments for device functions and trap positions in constant memory. 
// (Requires all functions to be moved into the same file or the use of some 
// workaround found on nVidia forum)  	
//-Put pSLM_start and aLaser in texture memory
//-Use "zero-copy" to transfer pSLM to host. (will only work on 1.3 devices and higher)
//-Rename functions and variables for consistency and readability
////////////////////////////////////////////////////////////////////////////////


#include "GenerateHologramCUDA.h"

//////////////////////////////////////////////////
//Global declaration
//////////////////////////////////////////////////
float *d_x, *d_y, *d_z, *d_I;					//trap coordinates and intensity in GPU memory
float *delta, *d_delta;							//distance between pixels on SLM and trap positions in host and GPU memory
float *d_ei_dre, *d_ei_dim;						//exp(i*re(delta)), exp(i*re(delta))
float *d_Vim, *d_Vre;							//energy flux contribution from each SLM pixel to each trap 
float *d_VimR, *d_VreR;							//sum of above for each trap
float *d_pSLM;									//the optimized phase pattern, float
float *d_weights, *d_weights_start, *d_amps;	//used weights and calculated amplitudes for each spot and each iteration
float *d_pSLM_start;							//Initial phase pattern
float *d_spotRe, *d_spotIm;

int MaxSpots, n_blocks_V, n_blocks_Phi, memsize_SLM_f, memsize_SLM_uc, memsize_V_f, memsize_spots_f, data_w, N_pixels, N_iterations_last, N_spots_a;

unsigned char *d_pSLM_uc;						//The optimized phase pattern, unsigned char, the one sent to the SLM
unsigned char *h_LUT, *d_LUT;
int maxThreads_device;
int use_LUTfile;
bool bEnableSLM;

////////////////////////////////////////////////////
//Global declarations for the FFT version
////////////////////////////////////////////////////
float *d_aLaserFFT, *d_LUT_coeff;
hipfftHandle plan;
hipfftComplex *d_FFTo, *d_FFTd, *d_SLM_cc;
int *d_spot_index, memsize_SLM_cc;

////////////////////////////////////////////////////////////////////////////////
// Functions to talk to SLM Hardware
////////////////////////////////////////////////////////////////////////////////
extern "C" int InitalizeSLM(	//returns 0 if PCIe hardware is used, 1 if PCI hardware is used
	bool bRAMWriteEnable, char* LUTFile, unsigned char* LUT, unsigned short TrueFrames
);

extern "C" void LoadImg(
	unsigned char* Img
);

extern "C" void Wait(
	int DelayMs
);

extern "C" void SetPower(
	bool bPower
);

extern "C" void ShutDownSLM();


////////////////////////////////////////////////////////////////////////////////
//The main function, generates a hologram 
////////////////////////////////////////////////////////////////////////////////

extern "C" __declspec(dllexport) int GenerateHologram(float *h_test, unsigned char *h_pSLM, float *x_spots, float *y_spots, float *z_spots, float *I_spots, int N_spots, int N_iterations, float *weights, float alpha, int method)
{
	int retur = 0;
	float alpha_RPC = alpha*2.0f*M_PI;
	if (N_spots > MaxSpots)
	{
		N_spots = MaxSpots;
		retur = 1;
	}
	
	memsize_spots_f = N_spots*sizeof(float);
	hipMemcpy(d_x, x_spots, memsize_spots_f, hipMemcpyHostToDevice);	
	hipMemcpy(d_y, y_spots, memsize_spots_f, hipMemcpyHostToDevice);	
	hipMemcpy(d_z, z_spots, memsize_spots_f, hipMemcpyHostToDevice);
	hipMemcpy(d_I, I_spots, memsize_spots_f, hipMemcpyHostToDevice);
	
	if ((method == 0)||((method == 1)&&(N_spots < 3)))
	{
		//////////////////////////////////////////////////
		//Generate the hologram using "Lenses and Prisms"
		//////////////////////////////////////////////////
		LensesAndPrisms<<< n_blocks_Phi, block_size >>>(d_x, d_y, d_z, d_I, d_pSLM_uc, N_spots, d_LUT, use_LUTfile, data_w);
		hipDeviceSynchronize();
		checkAmplitudes<<< N_spots, 512>>>(d_x, d_y, d_z, d_pSLM_uc, d_amps, N_spots, N_pixels, data_w);
		hipDeviceSynchronize();
		hipMemcpy(h_pSLM, d_pSLM_uc, memsize_SLM_uc, hipMemcpyDeviceToHost);	
		hipMemcpy(weights, d_amps, N_spots*sizeof(float), hipMemcpyDeviceToHost);
		retur = 5;	
	}
	else if (method == 1)		
	{
		////////////////////////////////////////////////////
		//Genreate holgram using fresnel propagation
		////////////////////////////////////////////////////
		
		hipMemcpy(d_weights, d_weights_start, memsize_spots_f, hipMemcpyDeviceToDevice);
		hipMemcpy(d_pSLM_start, d_pSLM, memsize_SLM_f, hipMemcpyDeviceToDevice);
		
		hipDeviceSynchronize();
		for (int l=0; l<N_iterations; l++)
		{	
			////////////////////////////////////////////////////
			//Propagate to the farfield 
			////////////////////////////////////////////////////				
			transformToFarfield<<< N_spots, 512>>>(d_x, d_y, d_z, d_pSLM, d_spotRe, d_spotIm, N_spots, N_pixels, data_w);
			hipDeviceSynchronize();		
			////////////////////////////////////////////////////
			//Propagate to the SLM plane
			////////////////////////////////////////////////////
			computePhiNew<<< 512, 512 >>>(d_x, d_y, d_z, d_I, d_spotRe, d_spotIm, d_pSLM, N_pixels, N_spots, d_weights, l, d_pSLM_start, alpha_RPC, d_amps);
			hipDeviceSynchronize();
			retur = N_spots;
		}	

		f2uc<<< n_blocks_Phi, block_size >>>(d_pSLM_uc, d_pSLM, N_pixels, d_LUT, use_LUTfile, data_w);
		hipDeviceSynchronize();
		hipMemcpy(h_pSLM, d_pSLM_uc, memsize_SLM_uc, hipMemcpyDeviceToHost);			
		hipMemcpy(weights, d_amps, N_spots*(N_iterations)*sizeof(float), hipMemcpyDeviceToHost);
		//hipMemcpy(weights, d_weights, N_spots*sizeof(float), hipMemcpyDeviceToHost);
	}
	//generate hologram using fast fourier transforms 
	else if (method ==2)			
	{
		//hipMemcpy(d_pSLM_uc, h_pSLM, memsize_SLM_uc, hipMemcpyHostToDevice);
		//hipDeviceSynchronize();
		//p_uc2c_cc_shift<<< n_blocks_Phi, block_size >>>(d_SLM_cc, d_pSLM_uc, N_pixels, data_w);

		float amp_desired = N_pixels * sqrt(1.0f/(float)N_spots);
		float weight = 1.0f/(float)N_spots;
		for (int i=0; i < N_spots; ++i)
		{
			weights[i] = weight;
		} 
		hipDeviceSynchronize();		
		hipMemcpy(d_weights, weights, N_spots * sizeof(float), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		hipMemset(d_FFTd, 0, memsize_SLM_cc);
		hipDeviceSynchronize();		
		XYtoIndex <<< 1, N_spots >>>(d_x,  d_y, d_spot_index, N_spots, data_w);
		hipDeviceSynchronize();		
		for (int l=0; l<N_iterations; l++)
		{
			// Transform to trapping plane
			hipfftExecC2C(plan, d_SLM_cc, d_FFTo, HIPFFT_FORWARD);
			hipDeviceSynchronize();

			// Copy phases for spot indices in d_FFTo to d_FFTd
			ReplaceAmpsFFT <<< 1, N_spots >>> (d_FFTo, d_FFTd, d_spot_index, N_spots, l, d_amps, d_weights, amp_desired);
			hipDeviceSynchronize();
				//Transform back to SLM plane
			hipfftExecC2C(plan, d_FFTd, d_SLM_cc, HIPFFT_BACKWARD);
			hipDeviceSynchronize();

			// Set amplitudes in d_SLM to the laser amplitude profile
			ReplaceAmpsSLM <<< n_blocks_Phi, block_size >>> (d_aLaserFFT, d_SLM_cc, d_pSLM_start, N_pixels, alpha_RPC);
			hipDeviceSynchronize();
		}	
	
		// Calculate phases in the SLM plane   
		getPhases<<< n_blocks_Phi, block_size >>> (d_pSLM_uc, d_pSLM_start, d_SLM_cc, d_LUT_coeff, 0, data_w);	
		hipMemcpy(weights, d_amps, N_spots*(N_iterations)*sizeof(float), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();	
		hipMemcpy(h_pSLM, d_pSLM_uc, memsize_SLM_uc, hipMemcpyDeviceToHost);			
	}
	//load image to the PCIe hardware  SLMstuff
	if(bEnableSLM)
		LoadImg(h_pSLM);
	
	retur = hipGetLastError();
	//hipMemcpy(h_test, d_aLaserDFT, memsize_SLM_f, hipMemcpyDeviceToHost);
	return retur;	
}

////////////////////////////////////////////////////////////////////////////////
//Enable SLM
////////////////////////////////////////////////////////////////////////////////
extern "C" __declspec(dllexport) int startCUDAandSLM(int SLM_enabled, float *test, char* LUTFile, unsigned short TrueFrames, int deviceId, float *h_AberrationCorr)
{
	hipSetDevice(deviceId); 
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    maxThreads_device = deviceProp.maxThreadsPerBlock;
    
	MaxSpots = block_size;
	int MaxIterations = 1000;
	data_w = 512;
	N_pixels = data_w * data_w;
	N_spots_a = 3;
	N_iterations_last = 10;
	memsize_V_f = N_pixels * N_spots_a*sizeof(float);
	memsize_spots_f = MaxSpots * sizeof(float);
	memsize_SLM_f = N_pixels * sizeof(float);  
    memsize_SLM_uc = N_pixels * sizeof(unsigned char);
	memsize_SLM_cc = N_pixels * sizeof(hipfftComplex);
    n_blocks_Phi = (N_pixels/block_size + (N_pixels%block_size == 0 ? 0:1));
    n_blocks_V = (N_spots_a*N_pixels/block_size + ((N_spots_a*N_pixels)%block_size == 0 ? 0:1));

	float weights[10000];
	for (int i=0; i < MaxSpots; ++i)
	{
		weights[i] = 1;;
	} 

	hipMalloc((void**)&d_x, memsize_spots_f );
	hipMalloc((void**)&d_y, memsize_spots_f );
	hipMalloc((void**)&d_z, memsize_spots_f );
	hipMalloc((void**)&d_I, memsize_spots_f );
	hipMalloc((void**)&d_spotRe, memsize_spots_f );
	hipMalloc((void**)&d_spotIm, memsize_spots_f );

	hipMalloc((void**)&d_delta, memsize_V_f);
	hipMalloc((void**)&d_ei_dre, memsize_V_f);
	hipMalloc((void**)&d_ei_dim, memsize_V_f);
	hipMalloc((void**)&d_Vre, memsize_V_f);
	hipMalloc((void**)&d_Vim, memsize_V_f);
	hipMalloc((void**)&d_VreR, memsize_V_f);
	hipMalloc((void**)&d_VimR, memsize_V_f);
	hipMalloc((void**)&d_weights, MaxSpots*(MaxIterations+1)*sizeof(float));
	hipMalloc((void**)&d_weights_start, MaxSpots*(MaxIterations+1)*sizeof(float));
	hipMalloc((void**)&d_amps, MaxSpots*MaxIterations*sizeof(float));
	hipMalloc((void**)&d_pSLM, memsize_SLM_f);
	hipMalloc((void**)&d_pSLM_start, memsize_SLM_f);
	hipMalloc((void**)&d_pSLM_uc, memsize_SLM_uc);
	hipMemset(d_pSLM, 0, N_pixels*sizeof(float)); 
	hipMemcpy(d_weights_start, weights, MaxSpots*(N_iterations_last+1)*sizeof(float), hipMemcpyHostToDevice);
	
	hipMalloc((void**)&d_spot_index, MaxSpots * sizeof(int));
	hipMalloc((void**)&d_FFTd, memsize_SLM_cc);	
	hipMalloc((void**)&d_FFTo, memsize_SLM_cc);
	hipMalloc((void**)&d_SLM_cc, memsize_SLM_cc);
	hipfftPlan2d(&plan, data_w, data_w, HIPFFT_C2C);
	float *h_aLaserFFT = (float *)malloc(memsize_SLM_f);

	//Open up communication to the PCIe hardware
	bEnableSLM = SLM_enabled; //SLMstuff
	if(bEnableSLM)
	{
		bool bRAMWriteEnable = false;
		h_LUT = new unsigned char[256];
		use_LUTfile = InitalizeSLM(bRAMWriteEnable, LUTFile, h_LUT, TrueFrames);  //InitalizeSLM returns 1 if PCI version is installed, PCIe version returns 0 since it applies LUT in hardware 
		hipMalloc((void**)&d_LUT, 256);
		hipMemcpy(d_LUT, h_LUT, 256, hipMemcpyHostToDevice);
		delete []h_LUT;
		SetPower(true);
	}	
	else
	{
		use_LUTfile = 0;
	}
	return memsize_V_f;
}

extern "C" __declspec(dllexport) int stopCUDAandSLM()
{
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	hipFree(d_I);
	hipFree(d_delta);
	hipFree(d_ei_dre);
	hipFree(d_ei_dim);
	hipFree(d_Vre);
	hipFree(d_Vim);
	hipFree(d_VreR);
	hipFree(d_VimR);
	hipFree(d_weights);
	hipFree(d_amps);
	hipFree(d_pSLM);
	hipFree(d_pSLM_start);
	hipFree(d_pSLM_uc);
	
	hipFree(d_FFTd);
	hipFree(d_FFTo);
	hipFree(d_SLM_cc);
	hipfftDestroy(plan);
		
	if (use_LUTfile)
		hipFree(d_LUT);
	
	hipDeviceReset();
	
	//close out communication with the PCIe hardware SLMstuff
	if(bEnableSLM)
		ShutDownSLM();
	return 100;
}

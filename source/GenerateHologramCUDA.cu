#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
/*   
   Hologram generating algorithms for CUDA Devices
   
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se

   This file is part of GenerateHologramCUDA.

   GenerateHologramCUDA is free software: you can redistribute it and/or 
   modify it under the terms of the GNU Lesser General Public License as published 
   by the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   GenerateHologramCUDA is distributed in the hope that it will be 
   useful, but WITHOUT ANY WARRANTY; without even the implied warranty
   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU Lesser General Public License for more details.

   You should have received a copy of the GNU Lesser General Public License
   along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
///////////////////////////////////////////////////////////////////////////////////
//The function "GenerateHologram" contains two different algorithms for
//hologram generation. The last parameter in the function call selects which 
//one to use:
//0: Complex addition of "Lenses and Prisms", no optimization (3D)
//1: Weighted Gerchberg-Saxton algorithm using Fresnel propagation (3D)
//
//-(1) Is implemented in two different ways, one that uses precalculated values 
//     for the phase difference between each pixel and each spot (a), and one that
//     calculates the values in each iteration (b). (a) is faster for up 
//     to 4 spots. The selection of the two implementation is automatic and based
//	   based on the number of spots. 
//-(0) produces optimal holograms for 1 or 2 traps and is significantly faster.
//     (0) is automatically selected if the number of spots is < 3. 
////////////////////////////////////////////////////////////////////////////////
//Fresnel propagation based algorithm (1 and 3) described in:
//Roberto Di Leonardo, Francesca Ianni, and Giancarlo Ruocco
//"Computer generation of optimal holograms for optical trap arrays"
//Opt. Express 15, 1913-1922 (2007) 
//
//The original algorithm has been modified to allow variable spot amplitudes
////////////////////////////////////////////////////////////////////////////////
//Possible improvements:
//-Put all arguments for device functions and trap positions in constant memory. 
// (Requires all functions to be moved into the same file or the use of some 
// workaround found on nVidia forum)  	
//-Put pSLM_start and aLaser in texture memory
//-Use "zero-copy" to transfer pSLM to host. (will only work on 1.3 devices and higher)
//-Rename functions and variables for consistency and readability
////////////////////////////////////////////////////////////////////////////////


#include "GenerateHologramCUDA.h"

//////////////////////////////////////////////////
//Global declaration
//////////////////////////////////////////////////
float *d_x, *d_y, *d_z, *d_I;					//trap coordinates and intensity in GPU memory
float *delta, *d_delta;							//distance between pixels on SLM and trap positions in host and GPU memory
float *d_ei_dre, *d_ei_dim;						//exp(i*re(delta)), exp(i*re(delta))
float *d_Vim, *d_Vre;							//energy flux contribution from each SLM pixel to each trap 
float *d_VimR, *d_VreR;							//sum of above for each trap
float *d_pSLM;									//the optimized phase pattern, float
float *d_weights, *d_weights_start, *d_amps;	//used weights and calculated amplitudes for each spot and each iteration
float *d_pSLM_start;							//Initial phase pattern
float *d_spotRe, *d_spotIm;

int MaxSpots, n_blocks_V, n_blocks_Phi, memsize_SLM_f, memsize_SLM_uc, memsize_V_f, memsize_spots_f, data_w, N_pixels, N_iterations_last, N_spots_a;

unsigned char *d_pSLM_uc;						//The optimized phase pattern, unsigned char, the one sent to the SLM
unsigned char *h_LUT, *d_LUT;
int maxThreads_device;
int use_LUTfile;
bool bEnableSLM;

////////////////////////////////////////////////////
//Global declarations for the FFT version
////////////////////////////////////////////////////
float *d_aLaserFFT, *d_LUT_coeff;
hipfftHandle plan;
hipfftComplex *d_FFTo, *d_FFTd, *d_SLM_cc;
int *d_spot_index, memsize_SLM_cc;

////////////////////////////////////////////////////////////////////////////////
// Functions to talk to SLM Hardware
////////////////////////////////////////////////////////////////////////////////
extern "C" int InitalizeSLM(	//returns 0 if PCIe hardware is used, 1 if PCI hardware is used
	bool bRAMWriteEnable, char* LUTFile, unsigned char* LUT, unsigned short TrueFrames
);

extern "C" void LoadImg(
	unsigned char* Img
);

extern "C" void Wait(
	int DelayMs
);

extern "C" void SetPower(
	bool bPower
);

extern "C" void ShutDownSLM();


////////////////////////////////////////////////////////////////////////////////
//The main function, generates a hologram 
////////////////////////////////////////////////////////////////////////////////

extern "C" __declspec(dllexport) int GenerateHologram(float *h_test, unsigned char *h_pSLM, float *x_spots, float *y_spots, float *z_spots, float *I_spots, int N_spots, int N_iterations, float *weights, float alpha, int method)
{
	int retur = 0;
	float alpha_RPC = alpha*2.0f*M_PI;
	if (N_spots > MaxSpots)
	{
		N_spots = MaxSpots;
		retur = 1;
	}
	
	memsize_spots_f = N_spots*sizeof(float);
	hipMemcpy(d_x, x_spots, memsize_spots_f, hipMemcpyHostToDevice);	
	hipMemcpy(d_y, y_spots, memsize_spots_f, hipMemcpyHostToDevice);	
	hipMemcpy(d_z, z_spots, memsize_spots_f, hipMemcpyHostToDevice);
	hipMemcpy(d_I, I_spots, memsize_spots_f, hipMemcpyHostToDevice);
	
	if ((method == 0)||((method == 1)&&(N_spots < 3)))
	{
		//////////////////////////////////////////////////
		//Generate the hologram using "Lenses and Prisms"
		//////////////////////////////////////////////////
		LensesAndPrisms<<< n_blocks_Phi, block_size >>>(d_x, d_y, d_z, d_I, d_pSLM_uc, N_spots, d_LUT, use_LUTfile, data_w);
		hipDeviceSynchronize();
		checkAmplitudes<<< N_spots, 512>>>(d_x, d_y, d_z, d_pSLM_uc, d_amps, N_spots, N_pixels, data_w);
		hipDeviceSynchronize();
		hipMemcpy(h_pSLM, d_pSLM_uc, memsize_SLM_uc, hipMemcpyDeviceToHost);	
		hipMemcpy(weights, d_amps, N_spots*sizeof(float), hipMemcpyDeviceToHost);
		retur = 5;	
	}
	else if (method == 1)		
	{
		////////////////////////////////////////////////////
		//Genreate holgram using fresnel propagation
		////////////////////////////////////////////////////
		
		hipMemcpy(d_weights, d_weights_start, memsize_spots_f, hipMemcpyDeviceToDevice);
		hipMemcpy(d_pSLM_start, d_pSLM, memsize_SLM_f, hipMemcpyDeviceToDevice);
		
		////////////////////////////////////////////////////
		//Select implementation based on the number of spots.	
		////////////////////////////////////////////////////
		if (N_spots <= N_spots_a)    
		{
			n_blocks_V = N_spots*N_pixels/block_size;	

			//////////////////////////////////////////////////////////////////////////
			//Compute the phase difference between each pixel in the SLM and each spot
			//////////////////////////////////////////////////////////////////////////
			computeDelta <<< n_blocks_V, block_size >>> (d_x, d_y, d_z, d_delta, d_ei_dre, d_ei_dim, N_spots, N_pixels, data_w);
			hipDeviceSynchronize();
			
			/////////////////////////////////////////
			//Start the Gerchberg-Saxton iterations
			/////////////////////////////////////////
			for (int l=0; l<N_iterations; l++)
			{	
				//////////////////////////////////////////////////////////////////////////////////////////////////////////////
				// Compute complex amplitudes in spots (results will be located at spot number * N_pixels in d_Vre and d_Vim)
				//////////////////////////////////////////////////////////////////////////////////////////////////////////////
				computeV<<< n_blocks_Phi, block_size >>>(d_Vre, d_Vim, d_pSLM, d_delta, N_pixels, N_spots);
				hipDeviceSynchronize();
				
				//////////////////////////////////////////////////////////////////////////////////
				//Sum the (complex amplitude) contribution from all pixels on the SLM to each spot
				//////////////////////////////////////////////////////////////////////////////////
				for (int ll=0; ll<N_spots; ll++)
				{
					int offset = ll * N_pixels;
					//Reduce(N_pixels, maxThreads, maxBlocks, d_Vre, d_VreR, offset); //slightly faster but uses Nvidia code that does not permit LGPL licensing
					//Reduce(N_pixels, maxThreads, maxBlocks, d_Vim, d_VimR, offset);
					sumV(d_Vre, d_VreR, offset);
					sumV(d_Vim, d_VimR, offset);
					hipDeviceSynchronize();
				}
			
				computeWeights<<<1, N_spots>>>(d_VreR, d_VimR, N_spots, d_weights, l, d_amps, d_I, N_pixels);
				hipDeviceSynchronize();
				////////////////////////////////////////////////////
				//Transform to the SLM plane
				////////////////////////////////////////////////////
				computePhi<<< n_blocks_Phi, block_size >>>(d_VreR, d_VimR, d_pSLM, d_ei_dre, d_ei_dim, N_pixels, 
														N_spots, d_weights, (l+1), d_pSLM_start, alpha_RPC, d_amps);	
				hipDeviceSynchronize();
			}	
		}	
		else 
		{
			hipDeviceSynchronize();
			for (int l=0; l<N_iterations; l++)
			{	
				////////////////////////////////////////////////////
				//Propagate to the farfield 
				////////////////////////////////////////////////////				
				transformToFarfield<<< N_spots, 512>>>(d_x, d_y, d_z, d_pSLM, d_spotRe, d_spotIm, N_spots, N_pixels, data_w);
				hipDeviceSynchronize();		
				////////////////////////////////////////////////////
				//Propagate to the SLM plane
				////////////////////////////////////////////////////
				computePhiNew<<< 512, 512 >>>(d_x, d_y, d_z, d_I, d_spotRe, d_spotIm, d_pSLM, N_pixels, N_spots, d_weights, l, d_pSLM_start, alpha_RPC, d_amps);
				hipDeviceSynchronize();
				retur = N_spots;
			}	
		}
		f2uc<<< n_blocks_Phi, block_size >>>(d_pSLM_uc, d_pSLM, N_pixels, d_LUT, use_LUTfile, data_w);
		hipDeviceSynchronize();
		hipMemcpy(h_pSLM, d_pSLM_uc, memsize_SLM_uc, hipMemcpyDeviceToHost);			
		hipMemcpy(weights, d_amps, N_spots*(N_iterations)*sizeof(float), hipMemcpyDeviceToHost);
		//hipMemcpy(weights, d_weights, N_spots*sizeof(float), hipMemcpyDeviceToHost);
	}
	//generate hologram using fast fourier transforms 
	else if (method ==2)			
	{
		//hipMemcpy(d_pSLM_uc, h_pSLM, memsize_SLM_uc, hipMemcpyHostToDevice);
		//hipDeviceSynchronize();
		//p_uc2c_cc_shift<<< n_blocks_Phi, block_size >>>(d_SLM_cc, d_pSLM_uc, N_pixels, data_w);

		float amp_desired = N_pixels * sqrt(1.0f/(float)N_spots);
		float weight = 1.0f/(float)N_spots;
		for (int i=0; i < N_spots; ++i)
		{
			weights[i] = weight;
		} 
		hipDeviceSynchronize();		
		hipMemcpy(d_weights, weights, N_spots * sizeof(float), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		hipMemset(d_FFTd, 0, memsize_SLM_cc);
		hipDeviceSynchronize();		
		XYtoIndex <<< 1, N_spots >>>(d_x,  d_y, d_spot_index, N_spots, data_w);
		hipDeviceSynchronize();		
		for (int l=0; l<N_iterations; l++)
		{
			// Transform to trapping plane
			hipfftExecC2C(plan, d_SLM_cc, d_FFTo, HIPFFT_FORWARD);
			hipDeviceSynchronize();

			// Copy phases for spot indices in d_FFTo to d_FFTd
			ReplaceAmpsFFT <<< 1, N_spots >>> (d_FFTo, d_FFTd, d_spot_index, N_spots, l, d_amps, d_weights, amp_desired);
			hipDeviceSynchronize();
				//Transform back to SLM plane
			hipfftExecC2C(plan, d_FFTd, d_SLM_cc, HIPFFT_BACKWARD);
			hipDeviceSynchronize();

			// Set amplitudes in d_SLM to the laser amplitude profile
			ReplaceAmpsSLM <<< n_blocks_Phi, block_size >>> (d_aLaserFFT, d_SLM_cc, d_pSLM_start, N_pixels, alpha_RPC);
			hipDeviceSynchronize();
		}	
	
		// Calculate phases in the SLM plane   
		getPhases<<< n_blocks_Phi, block_size >>> (d_pSLM_uc, d_pSLM_start, d_SLM_cc, d_LUT_coeff, 0, data_w);	
		hipMemcpy(weights, d_amps, N_spots*(N_iterations)*sizeof(float), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();	
		hipMemcpy(h_pSLM, d_pSLM_uc, memsize_SLM_uc, hipMemcpyDeviceToHost);			
	}
	//load image to the PCIe hardware  SLMstuff
	if(bEnableSLM)
		LoadImg(h_pSLM);
	
	retur = hipGetLastError();
	//hipMemcpy(h_test, d_aLaserDFT, memsize_SLM_f, hipMemcpyDeviceToHost);
	return retur;	
}

////////////////////////////////////////////////////////////////////////////////
//Enable SLM
////////////////////////////////////////////////////////////////////////////////
extern "C" __declspec(dllexport) int startCUDAandSLM(int SLM_enabled, float *test, char* LUTFile, unsigned short TrueFrames, int deviceId, float *h_AberrationCorr)
{
	hipSetDevice(deviceId); 
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    maxThreads_device = deviceProp.maxThreadsPerBlock;
    
	MaxSpots = block_size;
	int MaxIterations = 1000;
	data_w = 512;
	N_pixels = data_w * data_w;
	N_spots_a = 3;
	N_iterations_last = 10;
	memsize_V_f = N_pixels * N_spots_a*sizeof(float);
	memsize_spots_f = MaxSpots * sizeof(float);
	memsize_SLM_f = N_pixels * sizeof(float);  
    memsize_SLM_uc = N_pixels * sizeof(unsigned char);
	memsize_SLM_cc = N_pixels * sizeof(hipfftComplex);
    n_blocks_Phi = (N_pixels/block_size + (N_pixels%block_size == 0 ? 0:1));
    n_blocks_V = (N_spots_a*N_pixels/block_size + ((N_spots_a*N_pixels)%block_size == 0 ? 0:1));

	float weights[10000];
	for (int i=0; i < MaxSpots; ++i)
	{
		weights[i] = 1;;
	} 

	hipMalloc((void**)&d_x, memsize_spots_f );
	hipMalloc((void**)&d_y, memsize_spots_f );
	hipMalloc((void**)&d_z, memsize_spots_f );
	hipMalloc((void**)&d_I, memsize_spots_f );
	hipMalloc((void**)&d_spotRe, memsize_spots_f );
	hipMalloc((void**)&d_spotIm, memsize_spots_f );

	hipMalloc((void**)&d_delta, memsize_V_f);
	hipMalloc((void**)&d_ei_dre, memsize_V_f);
	hipMalloc((void**)&d_ei_dim, memsize_V_f);
	hipMalloc((void**)&d_Vre, memsize_V_f);
	hipMalloc((void**)&d_Vim, memsize_V_f);
	hipMalloc((void**)&d_VreR, memsize_V_f);
	hipMalloc((void**)&d_VimR, memsize_V_f);
	hipMalloc((void**)&d_weights, MaxSpots*(MaxIterations+1)*sizeof(float));
	hipMalloc((void**)&d_weights_start, MaxSpots*(MaxIterations+1)*sizeof(float));
	hipMalloc((void**)&d_amps, MaxSpots*MaxIterations*sizeof(float));
	hipMalloc((void**)&d_pSLM, memsize_SLM_f);
	hipMalloc((void**)&d_pSLM_start, memsize_SLM_f);
	hipMalloc((void**)&d_pSLM_uc, memsize_SLM_uc);
	hipMemset(d_pSLM, 0, N_pixels*sizeof(float)); 
	hipMemcpy(d_weights_start, weights, MaxSpots*(N_iterations_last+1)*sizeof(float), hipMemcpyHostToDevice);
	
	hipMalloc((void**)&d_spot_index, MaxSpots * sizeof(int));
	hipMalloc((void**)&d_FFTd, memsize_SLM_cc);	
	hipMalloc((void**)&d_FFTo, memsize_SLM_cc);
	hipMalloc((void**)&d_SLM_cc, memsize_SLM_cc);
	hipfftPlan2d(&plan, data_w, data_w, HIPFFT_C2C);
	float *h_aLaserFFT = (float *)malloc(memsize_SLM_f);

	//Open up communication to the PCIe hardware
	bEnableSLM = SLM_enabled; //SLMstuff
	if(bEnableSLM)
	{
		bool bRAMWriteEnable = false;
		h_LUT = new unsigned char[256];
		use_LUTfile = InitalizeSLM(bRAMWriteEnable, LUTFile, h_LUT, TrueFrames);  //InitalizeSLM returns 1 if PCI version is installed, PCIe version returns 0 since it applies LUT in hardware 
		hipMalloc((void**)&d_LUT, 256);
		hipMemcpy(d_LUT, h_LUT, 256, hipMemcpyHostToDevice);
		delete []h_LUT;
		SetPower(true);
	}	
	else
	{
		use_LUTfile = 0;
	}
	return memsize_V_f;
}

extern "C" __declspec(dllexport) int stopCUDAandSLM()
{
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	hipFree(d_I);
	hipFree(d_delta);
	hipFree(d_ei_dre);
	hipFree(d_ei_dim);
	hipFree(d_Vre);
	hipFree(d_Vim);
	hipFree(d_VreR);
	hipFree(d_VimR);
	hipFree(d_weights);
	hipFree(d_amps);
	hipFree(d_pSLM);
	hipFree(d_pSLM_start);
	hipFree(d_pSLM_uc);
	
	hipFree(d_FFTd);
	hipFree(d_FFTo);
	hipFree(d_SLM_cc);
	hipfftDestroy(plan);
		
	if (use_LUTfile)
		hipFree(d_LUT);
	
	hipDeviceReset();
	
	//close out communication with the PCIe hardware SLMstuff
	if(bEnableSLM)
		ShutDownSLM();
	return 100;
}

/* FFT version, currently not working

//////////////////////////////////////////
//Add this to the header file
//////////////////////////////////////////
__global__ void XYtoIndex(float *d_x, float *d_y, int *spot_index, int N_spots, int data_w);
__global__ void ReplaceAmpsFFT(hipfftComplex *a, hipfftComplex *b, int *index, int N, int iteration, float *Intensity, float *weight, float amp_desired);
__global__ void getPhases(unsigned char *pSLMuc, float *d_pSLM_start, hipfftComplex *cSLMcc, unsigned char *g_LUT, int use_linLUT, int data_w);
__global__ void ReplaceAmpsSLM(float *a, hipfftComplex *b, float *p, int M, float RPC);



////////////////////////////////////////////////////
//Global declarations for the FFT version
////////////////////////////////////////////////////
float *d_aLaserFFT
hipfftHandle plan;
hipfftComplex *d_FFTo, *d_FFTd, *d_SLM_cc;
int *d_spot_index, memsize_SLM_cc;

///////////////////////////////////////////////////
//Add this to the StartCUDAandSLM function:
//////////////////////////////////////////////////
memsize_SLM_cc = N_pixels * sizeof(hipfftComplex);
float *h_aLaserFFT = (float *)malloc(memsize_SLM_f);
hipMalloc((void**)&d_aLaserFFT, memsize_SLM_f);
hipMalloc((void**)&d_spot_index, MaxSpots * sizeof(int));
hipMalloc((void**)&d_FFTd, memsize_SLM_cc);	
hipMalloc((void**)&d_FFTo, memsize_SLM_cc);
hipMalloc((void**)&d_SLM_cc, memsize_SLM_cc);
CUFFT_SAFE_CALL(hipfftPlan2d(&plan, data_w, data_w, HIPFFT_C2C));
float *h_aLaserFFT = (float *)malloc(memsize_SLM_f);
int IndexFFT, IndexDFT, row, col;
for(row = 0; row < data_w; row++)
{
	for(col = 0; col < data_w; col++)
	{
		//gaussian incident beam
		term1 = pow((float)row-beam_center_y,2.0f);
		term2 = pow((float)col-beam_center_x,2.0f);
		term3 = pow(beam_radius,2.0f);
		float Gauss = -(term1 + term2)/term3;
		Amplitude = exp(Gauss);
		//shift around my gaussian because we don't have an FFTShift equiv with CUDA
		if(row < (data_w/2) && col < (data_w/2))
			IndexFFT = ((row+(data_w/2))*data_w) + (col + (data_w/2));
		else if(row <(data_w/2) && col >= (data_w/2))
			IndexFFT = ((row+(data_w/2))*data_w) + (col - (data_w/2));
		else if(row >= (data_w/2) && col < (data_w/2))
			IndexFFT = ((row-(data_w/2))*data_w) + (col + (data_w/2));
			else
			IndexFFT = ((row-(data_w/2))*data_w) + (col - (data_w/2));
				
		//For the DFT version shifting is not needed
		IndexDFT = row*data_w + col;
			
		if (term1 + term2 < pow(aperture_radius, 2.0f)) 
		{
			h_aLaserFFT[IndexFFT] = Amplitude;
			h_aLaserDFT[IndexDFT] = Amplitude;
		}
		else
		{
			h_aLaserFFT[IndexFFT] = 0;
			h_aLaserDFT[IndexDFT] = 0;
		}
	}
hipMemcpy(d_aLaserFFT, h_aLaserFFT, memsize_SLM_f, hipMemcpyHostToDevice);
free(h_aLaserFFT);


////////////////////////////////////////////////////////////////
//Add this to the StopCUDAandSLM function:
///////////////////////////////////////////////////////////////
hipFree(d_FFTd);
hipFree(d_FFTo);
hipFree(d_SLM_cc);
hipFree(d_aLaserFFT);
hipfftDestroy(plan);

////////////////////////////////////////////////////////////////
//Add this to GenerateHologram
/////////////////////////////////////////////////////////////////
else if (method ==2)			//generate hologram using fast fourier transforms 
{
	float amp_desired = N_pixels * sqrt(1.0f/(float)N_spots);
	float weight = 1.0f/(float)N_spots;
	for (int i=0; i < N_spots; ++i)
	{
		weights[i] = weight;
	} 
		hipDeviceSynchronize();		
	hipMemcpy(d_weights, weights, N_spots * sizeof(float), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
	hipMemset(d_FFTd, 0, memsize_SLM_cc);
		hipDeviceSynchronize();		XYtoIndex <<< 1, N_spots >>> (d_x,  d_y, d_spot_index, N_spots, data_w);
		hipDeviceSynchronize();		for (int l=0; l<N_iterations; l++)
	{
		// Transform to FFT plane
		hipfftExecC2C(plan, d_SLM_cc, d_FFTo, HIPFFT_FORWARD);
		hipDeviceSynchronize();
	
		// Copy phases for spot indices in d_FFTo to d_FFTd
		ReplaceAmpsFFT <<< 1, N_spots >>> (d_FFTo, d_FFTd, d_spot_index, N_spots, l, d_amps, d_weights, amp_desired);
		hipDeviceSynchronize();
			//Transform back to SLM plane
		hipfftExecC2C(plan, d_FFTd, d_SLM_cc, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
		
		// Set amplitudes in d_SLM to the laser amplitudes
		ReplaceAmpsSLM <<< n_blocks_Phi, block_size >>> (d_aLaserFFT, d_SLM_cc, d_pSLM_start, N_pixels, alpha_RPC);
		hipDeviceSynchronize();
		
		retur = l;
	}	
	// Calculate phases in the FFT plane   
	getPhases<<< n_blocks_Phi, block_size >>> (d_pSLM_uc, d_pSLM_start, d_SLM_cc, d_LUT, use_LUTfile, data_w);		
	//getPhases<<< n_blocks_Phi, block_size >>> (d_pSLM_uc, d_pSLM_start, d_FFTd, d_LUT, use_LUTfile, data_w);		
	hipDeviceSynchronize();
	hipMemcpy(h_pSLM, d_pSLM_uc, memsize_SLM_uc, hipMemcpyDeviceToHost);			
}
*/
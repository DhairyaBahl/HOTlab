#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
/*   
   Hologram generating algorithms for CUDA Devices
   
   Copyright 2009, 2010, 2011 Martin Persson 
   martin.persson@physics.gu.se

   This file is part of GenerateHologramCUDA.

   GenerateHologramCUDA is free software: you can redistribute it and/or 
   modify it under the terms of the GNU Lesser General Public License as published 
   by the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   GenerateHologramCUDA is distributed in the hope that it will be 
   useful, but WITHOUT ANY WARRANTY; without even the implied warranty
   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU Lesser General Public License for more details.

   You should have received a copy of the GNU Lesser General Public License
   along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
///////////////////////////////////////////////////////////////////////////////////
//The function "GenerateHologram" contains two different algorithms for
//hologram generation. The last parameter in the function call selects which 
//one to use:
//0: Complex addition of "Lenses and Prisms", no optimization (3D)
//1: Weighted Gerchberg-Saxton algorithm using Fresnel propagation (3D)
//2: Weighted Gerchberg-Saxton algorithm using Fast Fourier Transforms (2D)
//-(0) produces optimal holograms for 1 or 2 traps and is significantly faster.
//     (0) is automatically selected if the number of spots is < 3. 
////////////////////////////////////////////////////////////////////////////////
//Fresnel propagation based algorithm (1) described in:
//Roberto Di Leonardo, Francesca Ianni, and Giancarlo Ruocco
//"Computer generation of optimal holograms for optical trap arrays"
//Opt. Express 15, 1913-1922 (2007) 
//
//The original algorithm has been modified to allow variable spot amplitudes
////////////////////////////////////////////////////////////////////////////////
//Naming convention for variables: 
//-The prefix indicates where data is located
//--In host functions:		h = host memory
//							d = device memory
//--In global functions:	g = global memory
//							s = shared memory 
//							no prefix = registers
//-The suffix indicates the data type
////////////////////////////////////////////////////////////////////////////////
//Possible improvements:
//-Improve convergence of the GS algorithms for 2 spots.
//-Compensate spot intensities for distance from center of field.
//-Put all arguments for device functions and trap positions in constant memory. 
// (Requires all functions to be moved into the same file or the use of some 
// workaround found on nVidia forum)  	
//-Put pSLMstart and aLaser in texture memory (may not improve performance on Fermi devices)
//-Use "zero-copy" to transfer pSLM to host.
//-Rename functions and variables for consistency and readability
//-Allow variable spot phases for Lenses and Prisms
////////////////////////////////////////////////////////////////////////////////


#include "GenerateHologramCUDA.h"

//////////////////////////////////////////////////
//Global declaration
//////////////////////////////////////////////////
float *d_x, *d_y, *d_z, *d_I;					//trap coordinates and intensity in GPU memory
float *d_pSLM_f;								//the optimized pSpot pattern, float [-pi, pi]
float *d_weights, *d_amps, *d_desiredAmp;		//used h_weights and calculated amplitudes for each spot and each iteration
float *d_pSLMstart_f;							//Initial pSpot pattern [-pi, pi]
float *d_spotRe_f, *d_spotIm_f;
float *d_AberrationCorr_f = NULL; 
float *d_LUTPolCoeff_f = NULL;
int N_LUTPolCoeff = 0;
int n_blocks_Phi, memsize_SLMf, memsize_SLMuc, memsize_spotsf, data_w, N_pixels, N_iterations_last;
float h_desiredAmp[MAX_SPOTS];
unsigned char *d_pSLM_uc;						//The optimized pSpot pattern, unsigned char, the one sent to the SLM [0, 255]
unsigned char *h_LUT_uc;
unsigned char *d_LUT_uc = NULL;
int maxThreads_device;
bool ApplyLUT_b = false, EnableSLM_b = false, UseAberrationCorr_b = false, UseLUTPol_b = false, saveAmps = false;

char CUDAmessage[100];
hipError_t status;

////////////////////////////////////////////////////
//Global declarations for the FFT version
////////////////////////////////////////////////////
float *d_aLaserFFT, *d_LUT_coeff;
hipfftHandle plan;
hipfftComplex *d_FFTo_cc, *d_FFTd_cc, *d_SLM_cc;
int *d_spot_index, memsize_SLMcc;

////////////////////////////////////////////////////////////////////////////////
// Functions to talk to SLM Hardware
////////////////////////////////////////////////////////////////////////////////
extern "C" int InitalizeSLM(	//returns 0 if PCIe hardware is used, 1 if PCI hardware is used
	bool bRAMWriteEnable, char* LUTFile, unsigned char* LUT, unsigned short TrueFrames
);

extern "C" void LoadImg(
	unsigned char* Img
);

extern "C" void Wait(
	int DelayMs
);

extern "C" void SetPower(
	bool bPower
);

extern "C" void ShutDownSLM();

void computeAmps(float *h_I, float *h_amp, int N_spots);
////////////////////////////////////////////////////////////////////////////////
//The main function, generates a hologram 
////////////////////////////////////////////////////////////////////////////////

extern "C" __declspec(dllexport) int GenerateHologram(float *h_test, unsigned char *h_pSLM_uc, float *x_spots, float *y_spots, float *z_spots, float *I_spots, int N_spots, int N_iterations, float *h_obtainedAmps, float alpha, int method)
{
	float alpha_RPC = alpha*2.0f*M_PI;
	if (N_spots > MAX_SPOTS)
	{
		N_spots = MAX_SPOTS;
	}
	else if (N_spots < 3)
		method = 0;
	computeAmps(I_spots, h_desiredAmp, N_spots);
	memsize_spotsf = N_spots*sizeof(float);
	hipMemcpy(d_x, x_spots, memsize_spotsf, hipMemcpyHostToDevice);	
	hipMemcpy(d_y, y_spots, memsize_spotsf, hipMemcpyHostToDevice);	
	hipMemcpy(d_z, z_spots, memsize_spotsf, hipMemcpyHostToDevice);


	switch (method)	{
		case 0:
			//////////////////////////////////////////////////
			//Generate the hologram using "Lenses and Prisms"
			//////////////////////////////////////////////////
			hipMemcpy(d_I, I_spots, memsize_spotsf, hipMemcpyHostToDevice);
			LensesAndPrisms<<< n_blocks_Phi, BLOCK_SIZE >>>(d_x, d_y, d_z, d_I, d_pSLM_uc, N_spots, d_LUT_uc, ApplyLUT_b, data_w, UseAberrationCorr_b, d_AberrationCorr_f, UseLUTPol_b, d_LUTPolCoeff_f, N_LUTPolCoeff);
			hipDeviceSynchronize();
			if (saveAmps)
			{
				checkAmplitudes<<< N_spots, 512>>>(d_x, d_y, d_z, d_pSLM_uc, d_amps, N_spots, N_pixels, data_w);
				hipDeviceSynchronize();
				hipMemcpy(h_obtainedAmps, d_amps, N_spots*sizeof(float), hipMemcpyDeviceToHost);
			}
			hipMemcpy(h_pSLM_uc, d_pSLM_uc, memsize_SLMuc, hipMemcpyDeviceToHost);	
			
			break;
		case 1:
			////////////////////////////////////////////////////////////////////////////
			//Genreate holgram using fresnel propagation
			////////////////////////////////////////////////////////////////////////////
			//Uncomment this to start with pre-calculated hologram:
			//hipMemcpy(d_pSLM_uc, h_pSLM_uc, memsize_SLMuc, hipMemcpyHostToDevice);
			//hipDeviceSynchronize();
			//uc2f<<< n_blocks_Phi, BLOCK_SIZE >>>(d_pSLM_f, d_pSLM_uc, N_pixels);
			////////////////////////////////////////////////////////////////////////////
			computeAmps(I_spots, h_desiredAmp, N_spots);
			hipMemcpy(d_desiredAmp, h_desiredAmp, memsize_spotsf, hipMemcpyHostToDevice);
			for (int l=0; l<N_iterations; l++)
			{	
				////////////////////////////////////////////////////
				//Propagate to the farfield 
				////////////////////////////////////////////////////				
				PropagateToSpotPositions_Fresnel<<< N_spots, 512>>>(d_x, d_y, d_z, d_pSLM_f, d_spotRe_f, d_spotIm_f, N_spots, N_pixels, data_w);
				hipDeviceSynchronize();		
				////////////////////////////////////////////////////
				//Propagate to the SLM plane
				////////////////////////////////////////////////////
				PropagateToSLM_Fresnel<<< 512, 512 >>>(d_x, d_y, d_z, d_desiredAmp, d_spotRe_f, d_spotIm_f, d_pSLM_f, N_pixels, N_spots, d_weights, l, d_pSLMstart_f, alpha_RPC, 
					d_amps, (l==(N_iterations-1)), d_pSLM_uc, d_LUT_uc, ApplyLUT_b, UseAberrationCorr_b, d_AberrationCorr_f, UseLUTPol_b, d_LUTPolCoeff_f, N_LUTPolCoeff, saveAmps);
				hipDeviceSynchronize();
			}	
			hipMemcpy(h_pSLM_uc, d_pSLM_uc, memsize_SLMuc, hipMemcpyDeviceToHost);			
			if (saveAmps)
				hipMemcpy(h_obtainedAmps, d_amps, N_spots*(N_iterations)*sizeof(float), hipMemcpyDeviceToHost);
			//hipMemcpy(h_obtainedAmps, d_weights, N_spots*(N_iterations)*sizeof(float), hipMemcpyDeviceToHost);
			break;
		case 2: 
			////////////////////////////////////////////////////////////////////////////////////////////
			//generate hologram using fast fourier transforms 
			////////////////////////////////////////////////////////////////////////////////////////////
			//Uncomment this to start with pre-calculated hologram:
			//hipMemcpy(d_pSLM_uc, h_pSLM_uc, memsize_SLMuc, hipMemcpyHostToDevice);
			//hipDeviceSynchronize();
			//p_uc2c_cc_shift<<< n_blocks_Phi, BLOCK_SIZE >>>(d_SLM_cc, d_pSLM_uc, N_pixels, data_w);
			////////////////////////////////////////////////////////////////////////////////////////////
			computeAmps(I_spots, h_desiredAmp, N_spots);
			hipMemcpy(d_desiredAmp, h_desiredAmp, memsize_spotsf, hipMemcpyHostToDevice);
			hipMemset(d_FFTd_cc, 0, memsize_SLMcc);		
			XYtoIndex <<< 1, N_spots >>>(d_x,  d_y, d_spot_index, N_spots, data_w);
			hipDeviceSynchronize();		
			for (int l=0; l<N_iterations; l++)
			{
				//////////////////////////////////////////////////////////
				// Transform to trapping plane
				//////////////////////////////////////////////////////////
				hipfftExecC2C(plan, d_SLM_cc, d_FFTo_cc, HIPFFT_FORWARD);
				hipDeviceSynchronize();
				//////////////////////////////////////////////////////////
				// Copy phases for spot indices in d_FFTo_cc to d_FFTd_cc
				//////////////////////////////////////////////////////////
				ReplaceAmpsSpots_FFT <<< 1, N_spots >>> (d_FFTo_cc, d_FFTd_cc, d_spot_index, N_spots, l, d_amps, d_weights, d_desiredAmp, (l==(N_iterations-1)), saveAmps);
				hipDeviceSynchronize();
				//////////////////////////////////////////////////////////
				//Transform back to SLM plane
				//////////////////////////////////////////////////////////
				hipfftExecC2C(plan, d_FFTd_cc, d_SLM_cc, HIPFFT_BACKWARD);
				hipDeviceSynchronize();
				//////////////////////////////////////////////////////////
				// Set amplitudes in d_SLM to the laser amplitude profile
				//////////////////////////////////////////////////////////
				ReplaceAmpsSLM_FFT <<< n_blocks_Phi, BLOCK_SIZE >>> (d_aLaserFFT, d_SLM_cc, d_pSLMstart_f, N_pixels, alpha_RPC, (l==(N_iterations-1)), d_pSLM_uc, d_LUT_uc, 
									ApplyLUT_b, UseAberrationCorr_b, d_AberrationCorr_f, UseLUTPol_b, d_LUTPolCoeff_f, N_LUTPolCoeff);
				hipDeviceSynchronize();
			}		
			if (saveAmps)
				hipMemcpy(h_obtainedAmps, d_amps, N_spots*(N_iterations)*sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(h_pSLM_uc, d_pSLM_uc, memsize_SLMuc, hipMemcpyDeviceToHost);			
			break;
	//case 3: Apply corrections on h_pSLM_uc (yet to be implemented)
	}

	//load image to the PCIe hardware  SLMstuff
	if(EnableSLM_b)
		LoadImg(h_pSLM_uc);

	//Handle CUDA errors
	status = hipGetLastError();
	if(status)
	{
		strcat(CUDAmessage, "CUDA says: ");
		strcat(CUDAmessage,	hipGetErrorString(status));
		strcat(CUDAmessage,	" in function 'GenerateHologram'\n");
		AfxMessageBox(CUDAmessage);
	}
	return status;
}

////////////////////////////////////////////////////////////////////////////////
//Set correction parameters
////////////////////////////////////////////////////////////////////////////////
extern "C" __declspec(dllexport) int Corrections(int UseAberrationCorr, float *h_AberrationCorr, int UseLUTPol, int PolOrder, float *h_LUTPolCoeff, int saveAmplitudes)
{
	UseAberrationCorr_b = (bool)UseAberrationCorr;
	UseLUTPol_b = (bool)UseLUTPol;
	saveAmps = (bool)saveAmplitudes;
	int Ncoeff[5] = {20, 35, 56, 84, 120};
	if ((3<=PolOrder)&&(PolOrder<=7))
		N_LUTPolCoeff = Ncoeff[PolOrder - 3];
	else
	{
		AfxMessageBox("Polynomial order out of range\n -coerced to 3");
		N_LUTPolCoeff = Ncoeff[0];
	}

	if(UseAberrationCorr_b)
	{
		if (d_AberrationCorr_f == NULL)		//Allocate memory only if not already allocated
			hipMalloc((void**)&d_AberrationCorr_f, memsize_SLMf);
		UseAberrationCorr_b = !hipMemcpy(d_AberrationCorr_f, h_AberrationCorr, memsize_SLMf, hipMemcpyHostToDevice);
	}
	else if (d_AberrationCorr_f != NULL)	//If memory is allocated: free memory and reset pointer to NULL
	{	
		hipFree(d_AberrationCorr_f); 
		d_AberrationCorr_f = NULL;
	}
	if(UseLUTPol_b)
	{
		if (d_LUTPolCoeff_f == NULL)		      //Allocate memory only if not already allocated
			hipMalloc((void**)&d_LUTPolCoeff_f, 120*sizeof(float));
		UseLUTPol_b = !hipMemcpy(d_LUTPolCoeff_f, h_LUTPolCoeff, N_LUTPolCoeff*sizeof(float), hipMemcpyHostToDevice);
	}
	else if (d_LUTPolCoeff_f!=NULL)	//If memory is allocated: free memory and reset pointer to NULL
	{
		hipFree(d_LUTPolCoeff_f);	
		d_LUTPolCoeff_f = NULL;	
	}
	
	//Handle CUDA errors
	status = hipGetLastError();
	if(status)
	{
		strcat(CUDAmessage, "CUDA says: ");
		strcat(CUDAmessage,	hipGetErrorString(status));
		strcat(CUDAmessage,	" in function 'Corrections'\n");
		AfxMessageBox(CUDAmessage);
	}
	return status;
}

////////////////////////////////////////////////////////////////////////////////
//Allocate GPU memory and start up SLM
////////////////////////////////////////////////////////////////////////////////
extern "C" __declspec(dllexport) int startCUDAandSLM(int EnableSLM, float *h_pSLMstart, char* LUTFile, unsigned short TrueFrames, int deviceId)
{
	//Make sure GPU with desired deviceId exists, set deviceId to 0 if not
	int deviceCount=0;
	if (hipGetDeviceCount(&deviceCount)!=0)
		AfxMessageBox("No CUDA compatible hardware found");
	if (deviceId>=deviceCount)
	{
		AfxMessageBox("Invalid deviceId, GPU with deviceId 0 used");
		deviceId=0;
	}
	hipSetDevice(deviceId);
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);
    maxThreads_device = deviceProp.maxThreadsPerBlock;
    
	int MaxIterations = 1000;
	data_w = SLM_SIZE;
	N_pixels = data_w * data_w;
	N_iterations_last = 10;
	memsize_spotsf = MAX_SPOTS * sizeof(float);
	memsize_SLMf = N_pixels * sizeof(float);  
    memsize_SLMuc = N_pixels * sizeof(unsigned char);
	memsize_SLMcc = N_pixels * sizeof(hipfftComplex);
    n_blocks_Phi = (N_pixels/BLOCK_SIZE + (N_pixels%BLOCK_SIZE == 0 ? 0:1));

	//memory allocations for all methods
	hipMalloc((void**)&d_x, memsize_spotsf );
	hipMalloc((void**)&d_y, memsize_spotsf );
	hipMalloc((void**)&d_z, memsize_spotsf );
	hipMalloc((void**)&d_I, memsize_spotsf );
	hipMalloc((void**)&d_desiredAmp, memsize_spotsf );
	hipMalloc((void**)&d_weights, MAX_SPOTS*(MaxIterations+1)*sizeof(float));
	hipMalloc((void**)&d_amps, MAX_SPOTS*MaxIterations*sizeof(float));
	
	hipMalloc((void**)&d_spotRe_f, memsize_spotsf );
	hipMalloc((void**)&d_spotIm_f, memsize_spotsf );
	hipMalloc((void**)&d_pSLM_f, memsize_SLMf);
	hipMalloc((void**)&d_pSLMstart_f, memsize_SLMf);
	hipMalloc((void**)&d_pSLM_uc, memsize_SLMuc);
	hipMemset(d_pSLMstart_f, 0, N_pixels*sizeof(float));

	hipMemcpy(d_pSLM_f, h_pSLMstart, N_pixels*sizeof(float), hipMemcpyHostToDevice);
	
	//memory allocations etc. for all FFT based Gerchberg-Saxton
	hipMalloc((void**)&d_spot_index, MAX_SPOTS * sizeof(int));
	hipMalloc((void**)&d_FFTd_cc, memsize_SLMcc);	
	hipMalloc((void**)&d_FFTo_cc, memsize_SLMcc);
	hipMalloc((void**)&d_SLM_cc, memsize_SLMcc);
	hipDeviceSynchronize();
	p2c <<< n_blocks_Phi, BLOCK_SIZE >>>(d_SLM_cc, d_pSLM_f, N_pixels);
	hipDeviceSynchronize();
	hipfftPlan2d(&plan, data_w, data_w, HIPFFT_C2C);
	
	float *h_aLaserFFT = (float *)malloc(memsize_SLMf);

	//Open up communication to the PCIe hardware
	EnableSLM_b = EnableSLM; //SLMstuff
	if(EnableSLM_b)
	{
		bool bRAMWriteEnable = false;
		h_LUT_uc = new unsigned char[256]; //change this for use with 16-bit interfaces
		ApplyLUT_b = (bool)InitalizeSLM(bRAMWriteEnable, LUTFile, h_LUT_uc, TrueFrames);  //InitalizeSLM returns 1 if PCI version is installed, PCIe version returns 0 since it applies LUT in hardware 
		hipMalloc((void**)&d_LUT_uc, 256);
		hipMemcpy(d_LUT_uc, h_LUT_uc, 256, hipMemcpyHostToDevice);
		delete []h_LUT_uc;
		SetPower(true);
	}	
	
	//Display CUDA errors
	status = hipGetLastError();
	if(status)
	{
		strcat(CUDAmessage, "CUDA says: ");
		strcat(CUDAmessage,	hipGetErrorString(status));
		strcat(CUDAmessage,	" in function 'startCUDAandSLM'\n");
		AfxMessageBox(CUDAmessage);
	}
	return status;
}

extern "C" __declspec(dllexport) int stopCUDAandSLM()
{
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	hipFree(d_I);

	hipFree(d_weights);
	hipFree(d_amps);
	hipFree(d_pSLM_f);
	hipFree(d_pSLMstart_f);
	hipFree(d_pSLM_uc);
	
	hipFree(d_FFTd_cc);
	hipFree(d_FFTo_cc);
	hipFree(d_SLM_cc);
	hipfftDestroy(plan);
		
	if (ApplyLUT_b)
	{	
		hipFree(d_LUT_uc); 
		d_LUT_uc = NULL;
	}
	
	if (UseAberrationCorr_b)
	{
		hipFree(d_AberrationCorr_f); 
		d_AberrationCorr_f = NULL;
	}
	
	if (UseLUTPol_b)
	{	
		hipFree(d_LUTPolCoeff_f); 
		d_LUTPolCoeff_f = NULL;
	}

	status = hipGetLastError();
	if(status)
	{
		
		strcat(CUDAmessage, "CUDA says: ");
		strcat(CUDAmessage,	hipGetErrorString(status));
		strcat(CUDAmessage,	" in function 'stopCUDAandSLM'\n");
		AfxMessageBox(CUDAmessage);
	}

	hipDeviceReset();
	
	//close out communication with the PCIe hardware SLMstuff
	if(EnableSLM_b)
		ShutDownSLM();

	return status;
}
////////////////////////////////////////////////////////////////////////////////
//Calculate amplitudes in positions given by x, y, and z from a given hologram
////////////////////////////////////////////////////////////////////////////////
extern "C" __declspec(dllexport) int GetAmps(float *x_spots, float *y_spots, float *z_spots, float *h_pSLM_uc, int N_spots_all, int data_w, float *h_amps)
{
	float *d_xall, *d_yall, *d_zall, *d_amps_all;
	hipMalloc((void**)&d_xall, N_spots_all*sizeof(float) );
	hipMalloc((void**)&d_yall, N_spots_all*sizeof(float) );
	hipMalloc((void**)&d_zall, N_spots_all*sizeof(float) );
	hipMalloc((void**)&d_amps_all, N_spots_all*sizeof(float) );
	hipMemcpy(d_xall, x_spots, N_spots_all*sizeof(float), hipMemcpyHostToDevice);	
	hipMemcpy(d_yall, y_spots, N_spots_all*sizeof(float), hipMemcpyHostToDevice);	
	hipMemcpy(d_zall, z_spots, N_spots_all*sizeof(float), hipMemcpyHostToDevice);
	
	int N_pixels = data_w*data_w;
	hipMemcpy(d_pSLM_uc, h_pSLM_uc, memsize_SLMuc, hipMemcpyHostToDevice);
	int offset = 0;
	int N_spots_rem = N_spots_all;
	int N_spots_this;
	while (N_spots_rem > 0)
	{
		N_spots_this = (N_spots_rem > 512) ? 512 : N_spots_rem;
		checkAmplitudes<<< N_spots_this, 512>>>(d_xall+offset, d_yall+offset, d_zall+offset, d_pSLM_uc, d_amps_all+offset, N_spots_this, N_pixels, data_w);
		hipDeviceSynchronize();
		
		N_spots_rem -= 512;
		offset += 512;
	}
	hipMemcpy(h_amps, d_amps_all, N_spots_all*sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(d_xall);
	hipFree(d_yall);
	hipFree(d_zall);
	hipFree(d_amps_all);
	
	status = hipGetLastError();
	if(status)
	{
		strcat(CUDAmessage, "CUDA says: ");
		strcat(CUDAmessage,	hipGetErrorString(status));
		strcat(CUDAmessage,	" in function 'GetAmps'\n");
		AfxMessageBox(CUDAmessage);
	}
	return status;
}

void computeAmps(float *h_I, float *h_desiredAmp, int N_spots)
{
	float Isum = 0;
	for (int i = 0; i<N_spots; i++)
		Isum += h_I[i];
	for (int j = 0; j<N_spots; j++)
		h_desiredAmp[j] = (h_I[j] <= 0) ? 0.01f:sqrtf(h_I[j]/Isum);
}


__global__ void testfunc(float *testdata)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	testdata[idx] = idx;
}